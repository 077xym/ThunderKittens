#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <pybind11/pybind11.h>
#include <torch/extension.h>

#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>
#include <vector>

#include "kittens.cuh"
#include "pyutils/torch_helpers.cuh"

constexpr int NUM_DEVICES = 8;

using namespace kittens;

using q_pgl = pgl<gl<bf16, -1, -1, -1, -1>, NUM_DEVICES, true>; 
using k_pgl = pgl<gl<bf16, -1, -1, -1, -1>, NUM_DEVICES, true>; 
using v_pgl = pgl<gl<bf16, -1, -1, -1, -1>, NUM_DEVICES, true>; 

#ifdef TORCH_COMPILE

template <int I, int SIZE> struct CHECK_INPUTS {
    static inline void apply(const int64_t B,
                             const int64_t H_qo,
                             const int64_t H_kv,
                             const int64_t N,
                             const int64_t D_h,
                             const std::vector<torch::Tensor>& Qs,
                             const std::vector<torch::Tensor>& Ks,
                             const std::vector<torch::Tensor>& Vs) {
        CHECK_INPUT(Qs[I]);
        CHECK_INPUT(Ks[I]);
        CHECK_INPUT(Vs[I]);

        TORCH_CHECK(Qs[I].size(0) == B, "Q batch dimension (device ", I, ") does not match with other inputs");
        TORCH_CHECK(Ks[I].size(0) == B, "K batch dimension (device ", I, ") does not match with other inputs");
        TORCH_CHECK(Vs[I].size(0) == B, "V batch dimension (device ", I, ") does not match with other inputs");

        TORCH_CHECK(Qs[I].size(1) == H_qo, "QO head dimension (device ", I, ") does not match with other inputs");
        TORCH_CHECK(Ks[I].size(1) == H_kv, "KV head dimension (device ", I, ") does not match with other inputs");
        TORCH_CHECK(Vs[I].size(1) == H_kv, "KV head dimension (device ", I, ") does not match with other inputs");

        TORCH_CHECK(Qs[I].size(2) == N, "Q sequence length dimension (device ", I, ") does not match with other inputs");
        TORCH_CHECK(Ks[I].size(2) == N, "K sequence length dimension (device ", I, ") does not match with other inputs");
        TORCH_CHECK(Vs[I].size(2) == N, "V sequence length dimension (device ", I, ") does not match with other inputs");

        TORCH_CHECK(Qs[I].size(3) == D_h, "Q head dimension (device ", I, ") does not match with other inputs");
        TORCH_CHECK(Ks[I].size(3) == D_h, "K head dimension (device ", I, ") does not match with other inputs");
        TORCH_CHECK(Vs[I].size(3) == D_h, "V head dimension (device ", I, ") does not match with other inputs");
        
        CHECK_INPUTS<I + 1, SIZE>::apply(B, H_qo, H_kv, N, D_h, Qs, Ks, Vs);  
    }
};
template <int SIZE> struct CHECK_INPUTS<SIZE, SIZE> {
    static inline void apply(const int64_t B,
                             const int64_t H_qo,
                             const int64_t H_kv,
                             const int64_t N,
                             const int64_t D_h,
                             const std::vector<torch::Tensor>&, 
                             const std::vector<torch::Tensor>&, 
                             const std::vector<torch::Tensor>&) {}
};

std::vector<torch::Tensor> ring_attention_forward(
    const std::vector<torch::Tensor> &Qs, 
    const std::vector<torch::Tensor> &Ks, 
    const std::vector<torch::Tensor> &Vs, 
    bool causal
) {
    // Input checking (up to CHECK_INPUTS) takes about 3us 
    TORCH_CHECK(Qs.size() == NUM_DEVICES, "Qs must be of size ", NUM_DEVICES);
    TORCH_CHECK(Ks.size() == NUM_DEVICES, "Ks must be of size ", NUM_DEVICES);
    TORCH_CHECK(Vs.size() == NUM_DEVICES, "Vs must be of size ", NUM_DEVICES);

    int64_t B    = Qs[0].size(0);
    int64_t H_qo = Qs[0].size(1);
    int64_t H_kv = Ks[0].size(1);
    int64_t N    = Qs[0].size(2);
    int64_t D_h  = Qs[0].size(3);

    TORCH_CHECK(H_qo >= H_kv, "QO heads must be greater than or equal to KV heads");
    TORCH_CHECK(H_qo % H_kv == 0, "QO heads must be divisible by KV heads");

    CHECK_INPUTS<0, NUM_DEVICES>::apply(B, H_qo, H_kv, N, D_h, Qs, Ks, Vs);

    return Qs;
}

std::vector<torch::Tensor> ring_attention_backward(
    torch::Tensor q, torch::Tensor k, torch::Tensor v, torch::Tensor o, 
    torch::Tensor l_vec, torch::Tensor og, bool causal
) {
    TORCH_CHECK(false, "Backward ring attention not implemented");
    return {q, k, v, o, l_vec, og};
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.doc() = "ThunderKittens Ring Attention Kernels";
    m.def(
        "ring_mha_forward",  
        torch::wrap_pybind_function(ring_attention_forward),
        "Forward ring MHA"
    );
    m.def(
        "ring_mha_backward", 
        torch::wrap_pybind_function(ring_attention_backward), 
        "Backward ring MHA"
    );
}

#else

#endif
