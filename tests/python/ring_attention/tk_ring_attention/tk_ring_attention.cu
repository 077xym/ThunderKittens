#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>
#include <pybind11/pybind11.h>
#include <torch/extension.h>
namespace py = pybind11;

#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_runtime.h>

#include <chrono>
#include <iostream>
#include <vector>

#include "kittens.cuh"
#include "pyutils/torch_helpers.cuh"

constexpr int NUM_DEVICES         = 8;
constexpr int CONSUMER_WARPGROUPS = 3; 
constexpr int PRODUCER_WARPGROUPS = 1; 
constexpr int NUM_WARPGROUPS      = CONSUMER_WARPGROUPS + PRODUCER_WARPGROUPS; 
constexpr int NUM_WORKERS         = NUM_WARPGROUPS * kittens::WARPGROUP_WARPS;

using namespace kittens;

template<int D> struct fwd_tile_dims {};
template<> struct fwd_tile_dims<64> {
    constexpr static int tile_width = 64;
    constexpr static int QO_height  = 4 * 16;
    constexpr static int KV_height  = 8 * 16;
    constexpr static int stages     = 4; 
};
template<> struct fwd_tile_dims<128> {
    constexpr static int tile_width = 128;
    constexpr static int QO_height  = 4 * 16;
    constexpr static int KV_height  = 8 * 16;
    constexpr static int stages     = 2;
};
template<int D> struct fwd_pglobals {
    using Q_tile = st_bf<fwd_tile_dims<D>::QO_height, fwd_tile_dims<D>::tile_width>;
    using K_tile = st_bf<fwd_tile_dims<D>::KV_height, fwd_tile_dims<D>::tile_width>;
    using V_tile = st_bf<fwd_tile_dims<D>::KV_height, fwd_tile_dims<D>::tile_width>;
    using O_tile = st_bf<fwd_tile_dims<D>::QO_height, fwd_tile_dims<D>::tile_width>;

    using Q_pgl = pgl<gl<bf16, -1, -1, -1, -1, Q_tile>, NUM_DEVICES, true>; 
    using K_pgl = pgl<gl<bf16, -1, -1, -1, -1, K_tile>, NUM_DEVICES, true>; 
    using V_pgl = pgl<gl<bf16, -1, -1, -1, -1, V_tile>, NUM_DEVICES, true>; 
    using O_pgl = pgl<gl<bf16, -1, -1, -1, -1, O_tile>, NUM_DEVICES, true>;

    Q_pgl Q;
    K_pgl K;
    V_pgl V;
    O_pgl O;

    const int N;
};

template<int D, bool is_causal>
__global__  __launch_bounds__(NUM_WORKERS * kittens::WARP_THREADS, 1)
void blockwise_attn_ker(const __grid_constant__ fwd_pglobals<D> p_G, const __grid_constant__ int dev_idx) {
    extern __shared__ int __shm[]; 
    tma_swizzle_allocator al((int*)&__shm[0]);
    int warpid = kittens::warpid(), warpgroupid = warpid/kittens::WARPGROUP_WARPS;

    using K = fwd_tile_dims<D>;

    using q_tile    =         st_bf<K::QO_height, K::tile_width>;
    using k_tile    =         st_bf<K::KV_height, K::tile_width>;
    using v_tile    =         st_bf<K::KV_height, K::tile_width>;
    using l_col_vec = col_vec<st_fl<K::QO_height, K::tile_width>>;
    using o_tile    =         st_bf<K::QO_height, K::tile_width>;
    
    q_tile    (&q_smem)[CONSUMER_WARPGROUPS] = al.allocate<q_tile, CONSUMER_WARPGROUPS>();
    k_tile    (&k_smem)[K::stages]           = al.allocate<k_tile, K::stages          >();
    v_tile    (&v_smem)[K::stages]           = al.allocate<v_tile, K::stages          >();
    l_col_vec (&l_smem)[CONSUMER_WARPGROUPS] = al.allocate<l_col_vec, CONSUMER_WARPGROUPS>();
    auto      (*o_smem)                      = reinterpret_cast<o_tile(*)>(q_smem);
    
    int kv_blocks   = p_G.N / (K::KV_height);
    // int kv_head_idx = blockIdx.y / g.hr;
    int kv_head_idx = blockIdx.y;
    int seq_idx     = blockIdx.x * CONSUMER_WARPGROUPS; 

    __shared__ kittens::semaphore qsmem_semaphore, k_smem_arrived[K::stages], v_smem_arrived[K::stages], compute_done[K::stages];
    if (threadIdx.x == 0) { 
        init_semaphore(qsmem_semaphore, 0, 1); 
        for(int j = 0; j < K::stages; j++) {
            init_semaphore(k_smem_arrived[j], 0, 1); 
            init_semaphore(v_smem_arrived[j], 0, 1); 
            init_semaphore(compute_done[j], CONSUMER_WARPGROUPS, 0); 
        }

        tma::expect_bytes(qsmem_semaphore, sizeof(q_smem));

        for (int wg = 0; wg < CONSUMER_WARPGROUPS; wg++) {
            coord<q_tile> q_tile_idx = {blockIdx.z, blockIdx.y, (seq_idx) + wg, 0};
            tma::load_async(q_smem[wg], p_G.Q[dev_idx], q_tile_idx, qsmem_semaphore);
        }

        for (int j = 0; j < K::stages - 1; j++) {
            coord<k_tile> kv_tile_idx = {blockIdx.z, kv_head_idx, j, 0};
            tma::expect_bytes(k_smem_arrived[j], sizeof(k_tile));
            tma::load_async(k_smem[j], p_G.K[dev_idx], kv_tile_idx, k_smem_arrived[j]);
            tma::expect_bytes(v_smem_arrived[j], sizeof(v_tile));
            tma::load_async(v_smem[j], p_G.V[dev_idx], kv_tile_idx, v_smem_arrived[j]);
        }
    }
    __syncthreads(); 

    int pipe_idx = K::stages - 1; 
    
    if(warpgroupid == NUM_WARPGROUPS-1) {
        warpgroup::decrease_registers<32>();      
        
        int kv_iters; 
        if constexpr (is_causal) {
            kv_iters = (seq_idx * (K::QO_height/kittens::TILE_ROW_DIM<bf16>)) - 1 + (CONSUMER_WARPGROUPS * (K::QO_height/kittens::TILE_ROW_DIM<bf16>)); 
            kv_iters = ((kv_iters / (K::KV_height/kittens::TILE_ROW_DIM<bf16>)) == 0) ? (0) : ((kv_iters / (K::KV_height/kittens::TILE_ROW_DIM<bf16>)) - 1);
        }
        else { kv_iters = kv_blocks-2; }

        if(warpid == NUM_WORKERS-4) {
            for (auto kv_idx = pipe_idx - 1; kv_idx <= kv_iters; kv_idx++) {
                coord<k_tile> kv_tile_idx = {blockIdx.z, kv_head_idx, kv_idx + 1, 0};
                tma::expect_bytes(k_smem_arrived[(kv_idx+1)%K::stages], sizeof(k_tile));
                tma::load_async(k_smem[(kv_idx+1)%K::stages], p_G.K[dev_idx], kv_tile_idx, k_smem_arrived[(kv_idx+1)%K::stages]);
                tma::expect_bytes(v_smem_arrived[(kv_idx+1)%K::stages], sizeof(v_tile));
                tma::load_async(v_smem[(kv_idx+1)%K::stages], p_G.V[dev_idx], kv_tile_idx, v_smem_arrived[(kv_idx+1)%K::stages]);
                
                wait(compute_done[(kv_idx)%K::stages], (kv_idx/K::stages)%2);
            }
        }
    }
    else {
        warpgroup::increase_registers<160>();

        rt_fl<16, K::KV_height>  att_block;
        rt_bf<16, K::KV_height>  att_block_mma;
        rt_fl<16, K::tile_width> o_reg;
        
        col_vec<rt_fl<16, K::KV_height>> max_vec, norm_vec, max_vec_last_scaled, max_vec_scaled;
        
        neg_infty(max_vec);
        zero(norm_vec);
        zero(o_reg);

        int kv_iters; 
        if constexpr (is_causal) {
            kv_iters = (seq_idx * 4) - 1 + (CONSUMER_WARPGROUPS * 4);
            kv_iters = (kv_iters/8);
        }
        else { kv_iters = kv_blocks - 1; }

        wait(qsmem_semaphore, 0);

        for (auto kv_idx = 0; kv_idx <= kv_iters; kv_idx++) {
        
            wait(k_smem_arrived[(kv_idx)%K::stages], (kv_idx/K::stages)%2);
            warpgroup::mm_ABt(att_block, q_smem[warpgroupid], k_smem[(kv_idx)%K::stages]);
            
            copy(max_vec_last_scaled, max_vec);
            if constexpr (D == 64) { mul(max_vec_last_scaled, max_vec_last_scaled, 1.44269504089f*0.125f); }
            else                   { mul(max_vec_last_scaled, max_vec_last_scaled, 1.44269504089f*0.08838834764f); }
            
            warpgroup::mma_async_wait();

            if constexpr (is_causal) {
                const int q_blk = (seq_idx * (K::QO_height/kittens::TILE_ROW_DIM<bf16>)) + warpid; 
                      int k_blk = (kv_idx * (K::KV_height/kittens::TILE_ROW_DIM<bf16>)); 

                #pragma unroll
                for(int _ = 0; k_blk == (kv_iters-1)*(K::KV_height/kittens::TILE_ROW_DIM<bf16>) || k_blk == (kv_iters)*(K::KV_height/kittens::TILE_ROW_DIM<bf16>); k_blk+=10000) {
                    #pragma unroll
                    for (auto j = 0; j < (K::KV_height/kittens::TILE_ROW_DIM<bf16>); j++) {
                        auto k_idx = k_blk + j;
                        auto &attn_subtile = reinterpret_cast<rt_fl<16, 16>&>(att_block.tiles[0][j]);

                        if      (k_idx >  q_blk) { neg_infty  (attn_subtile); }
                        else if (k_idx == q_blk) { make_causal(attn_subtile, attn_subtile, kittens::base_types::constants<float>::neg_infty()); }
                        __syncwarp();
                    }
                }
            }

            row_max(max_vec, att_block, max_vec);
            
            if constexpr (D == 64) { 
                mul(att_block, att_block,    1.44269504089f*0.125f); 
                mul(max_vec_scaled, max_vec, 1.44269504089f*0.125f);
            }
            else                   { 
                mul(att_block, att_block,    1.44269504089f*0.08838834764f); 
                mul(max_vec_scaled, max_vec, 1.44269504089f*0.08838834764f);
            }

            sub_row(att_block, att_block, max_vec_scaled);
            exp2(att_block, att_block);
            sub(max_vec_last_scaled, max_vec_last_scaled, max_vec_scaled);
            exp2(max_vec_last_scaled,       max_vec_last_scaled);
            mul(norm_vec,            norm_vec,     max_vec_last_scaled);
            row_sum(norm_vec,  att_block, norm_vec);
            add(att_block, att_block, 0.f);
            copy(att_block_mma, att_block); 
            mul_row(o_reg, o_reg, max_vec_last_scaled); 

            wait(v_smem_arrived[(kv_idx)%K::stages], (kv_idx/K::stages)%2); 

            warpgroup::mma_AB(o_reg, att_block_mma, v_smem[(kv_idx)%K::stages]);
            warpgroup::mma_async_wait();

            if(warpgroup::laneid() == 0) arrive(compute_done[(kv_idx)%K::stages], 1);
        }

        div_row(o_reg, o_reg, norm_vec);
        warpgroup::store(o_smem[warpgroupid], o_reg); 
        warpgroup::sync(warpgroupid+4);

        if (warpid % 4 == 0) {
            coord<o_tile> o_tile_idx = {blockIdx.z, blockIdx.y, (seq_idx) + warpgroupid, 0};
            tma::store_async(p_G.O[dev_idx], o_smem[warpgroupid], o_tile_idx);
        }

        mul(max_vec_scaled,   max_vec_scaled, 0.69314718056f);
        log(norm_vec, norm_vec);
        add(norm_vec, norm_vec, max_vec_scaled);

        if constexpr (D == 64) { mul(norm_vec, norm_vec, -8.0f); }
        else                   { mul(norm_vec, norm_vec, -11.313708499f); }
    
        warpgroup::store(l_smem[warpgroupid], norm_vec);
        warpgroup::sync(warpgroupid+4);

        tma::store_async_wait();
    }
}

#ifdef TORCH_COMPILE

template <int I, int SIZE> struct CHECK_INPUTS {
    static inline void apply(const int64_t B,
                             const int64_t H_qo,
                             const int64_t H_kv,
                             const int64_t N,
                             const int64_t D_h,
                             const std::vector<torch::Tensor>& Qs,
                             const std::vector<torch::Tensor>& Ks,
                             const std::vector<torch::Tensor>& Vs) {
        CHECK_INPUT(Qs[I]);
        CHECK_INPUT(Ks[I]);
        CHECK_INPUT(Vs[I]);

        TORCH_CHECK(Qs[I].size(0) == B, "Q batch dimension (device ", I, ") does not match with other inputs");
        TORCH_CHECK(Ks[I].size(0) == B, "K batch dimension (device ", I, ") does not match with other inputs");
        TORCH_CHECK(Vs[I].size(0) == B, "V batch dimension (device ", I, ") does not match with other inputs");

        TORCH_CHECK(Qs[I].size(1) == H_qo, "QO head dimension (device ", I, ") does not match with other inputs");
        TORCH_CHECK(Ks[I].size(1) == H_kv, "KV head dimension (device ", I, ") does not match with other inputs");
        TORCH_CHECK(Vs[I].size(1) == H_kv, "KV head dimension (device ", I, ") does not match with other inputs");

        TORCH_CHECK(Qs[I].size(2) == N, "Q sequence length dimension (device ", I, ") does not match with other inputs");
        TORCH_CHECK(Ks[I].size(2) == N, "K sequence length dimension (device ", I, ") does not match with other inputs");
        TORCH_CHECK(Vs[I].size(2) == N, "V sequence length dimension (device ", I, ") does not match with other inputs");

        TORCH_CHECK(Qs[I].size(3) == D_h, "Q head dimension (device ", I, ") does not match with other inputs");
        TORCH_CHECK(Ks[I].size(3) == D_h, "K head dimension (device ", I, ") does not match with other inputs");
        TORCH_CHECK(Vs[I].size(3) == D_h, "V head dimension (device ", I, ") does not match with other inputs");
        
        CHECK_INPUTS<I + 1, SIZE>::apply(B, H_qo, H_kv, N, D_h, Qs, Ks, Vs);  
    }
};
template <int SIZE> struct CHECK_INPUTS<SIZE, SIZE> {
    static inline void apply(const int64_t B,
                             const int64_t H_qo,
                             const int64_t H_kv,
                             const int64_t N,
                             const int64_t D_h,
                             const std::vector<torch::Tensor>&, 
                             const std::vector<torch::Tensor>&, 
                             const std::vector<torch::Tensor>&) {}
};

torch::Tensor pgl_tensor(
    const std::vector<int64_t> &sizes,
    const at::ScalarType dtype,
    const std::vector<int> &device_ids,
    const int device_id,
    const bool requires_grad
);
torch::Tensor pgl_tensor(
    const std::vector<int64_t> &sizes,
    const at::ScalarType dtype,
    const int *device_ids,
    const int device_id,
    const bool requires_grad
);
torch::Tensor pgl_tensor(
    const torch::Tensor &other, 
    const std::vector<int> &device_ids, 
    const int device_id
);

// TODO: combine outputs before returning
std::vector<torch::Tensor> ring_attention_forward(
    const std::vector<torch::Tensor> &Qs, 
    const std::vector<torch::Tensor> &Ks, 
    const std::vector<torch::Tensor> &Vs, 
    bool causal
) {
    // Input checking (up to CHECK_INPUTS<...>) takes about 3us 
    TORCH_CHECK(Qs.size() == NUM_DEVICES, "Qs must be of size ", NUM_DEVICES);
    TORCH_CHECK(Ks.size() == NUM_DEVICES, "Ks must be of size ", NUM_DEVICES);
    TORCH_CHECK(Vs.size() == NUM_DEVICES, "Vs must be of size ", NUM_DEVICES);

    int64_t B    = Qs[0].size(0);
    int64_t H_qo = Qs[0].size(1);
    int64_t H_kv = Ks[0].size(1);
    int64_t N    = Qs[0].size(2); // per-block sequence length
    int64_t D_h  = Qs[0].size(3);

    TORCH_CHECK(H_qo >= H_kv, "QO heads must be greater than or equal to KV heads");
    TORCH_CHECK(H_qo % H_kv == 0, "QO heads must be divisible by KV heads");

    CHECK_INPUTS<0, NUM_DEVICES>::apply(B, H_qo, H_kv, N, D_h, Qs, Ks, Vs);

    // TODO: support different head sizes
    TORCH_CHECK(H_qo == H_kv, "For now, different head sizes not supported");
    // TODO: support different head dims
    TORCH_CHECK(D_h == 64, "For now, head dim must be 64");
    // TODO: support causal attention
    TORCH_CHECK(!causal, "Causal attention not supported yet");

    // Initialize the KC threadpool
    int device_ids[NUM_DEVICES];
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) device_ids[dev_idx] = dev_idx;
    KittensClub club(device_ids, NUM_DEVICES);

    // Initialize output tensor, device pointers, and streams
    std::vector<torch::Tensor> Os(NUM_DEVICES);
    bf16 *d_Q[NUM_DEVICES];
    bf16 *d_K[NUM_DEVICES];
    bf16 *d_V[NUM_DEVICES];
    bf16 *d_O[NUM_DEVICES];
    hipStream_t streams[NUM_DEVICES];
    club.execute([&](int i) {
        Os[i] = pgl_tensor({B, H_qo, N, D_h}, at::kBFloat16, device_ids, i, true);
        d_Q[i] = reinterpret_cast<bf16*>(Qs[i].data_ptr<c10::BFloat16>());
        d_K[i] = reinterpret_cast<bf16*>(Ks[i].data_ptr<c10::BFloat16>());
        d_V[i] = reinterpret_cast<bf16*>(Vs[i].data_ptr<c10::BFloat16>());
        d_O[i] = reinterpret_cast<bf16*>(Os[i].data_ptr<c10::BFloat16>());
        streams[i] = at::cuda::getCurrentCUDAStream().stream();
        hipStreamSynchronize(streams[i]);
        CHECK_CUDA_ERROR(hipGetLastError());
    });

    // Initialize the parallel global layouts
    using pglobals = fwd_pglobals<64>;

    pglobals::Q_pgl p_Q(device_ids, d_Q, B, H_qo, N, D_h);
    pglobals::K_pgl p_K(device_ids, d_K, B, H_kv, N, D_h);
    pglobals::V_pgl p_V(device_ids, d_V, B, H_kv, N, D_h);
    pglobals::O_pgl p_O(device_ids, d_O, B, H_qo, N, D_h);
    pglobals p_G{p_Q, p_K, p_V, p_O, static_cast<int>(N)};

    // Initialize and run the kernel
    TORCH_CHECK(N % (CONSUMER_WARPGROUPS * kittens::TILE_ROW_DIM<bf16> * 4) == 0, "sequence length must be divisible by 192");
    dim3 grid(N / (CONSUMER_WARPGROUPS * kittens::TILE_ROW_DIM<bf16> * 4), H_qo, B);
    constexpr int smem = kittens::MAX_SHARED_MEMORY;

    club.execute([&](int i) {
        hipFuncSetAttribute(reinterpret_cast<const void*>(blockwise_attn_ker<64), false>, hipFuncAttributeMaxDynamicSharedMemorySize, smem);
        blockwise_attn_ker<64, false><<<grid, NUM_WORKERS * kittens::WARP_THREADS, smem, streams[i]>>>(p_G, i);
        hipStreamSynchronize(streams[i]);
        CHECK_CUDA_ERROR(hipGetLastError());
    });

    return Os;
}

std::vector<torch::Tensor> ring_attention_backward(
    torch::Tensor q, torch::Tensor k, torch::Tensor v, torch::Tensor o, 
    torch::Tensor l_vec, torch::Tensor og, bool causal
) {
    TORCH_CHECK(false, "Backward ring attention not implemented");
    return {q, k, v, o, l_vec, og};
}

struct pgl_tensor_context {
    int device_id;
    void *raw_ptr;
    size_t size;
};

void _pgl_tensor_deleter(void* ptr) {
    pgl_tensor_context *ctx = static_cast<pgl_tensor_context*>(ptr);
    pglCudaFree(ctx->device_id, ctx->raw_ptr, ctx->size);
    free(ctx);
}

torch::Tensor pgl_tensor(
    const std::vector<int64_t> &sizes,
    const at::ScalarType dtype,
    const int *device_ids,
    const int device_id,
    const bool requires_grad
) {
    TORCH_CHECK(device_id >= 0 && device_id < NUM_DEVICES, "Invalid device ID");

    // Calculate number of elements and bytes
    int64_t numel = 1;
    for (auto s : sizes) {
        TORCH_CHECK(s > 0, "Size dimensions must be positive");
        numel *= s;
    }

    // Allocate CUDA memory
    pgl_tensor_context *ctx = new pgl_tensor_context;
    ctx->device_id = device_id;
    ctx->raw_ptr = nullptr;
    ctx->size = numel * c10::elementSize(dtype);
    pglCudaMalloc<true>(NUM_DEVICES, const_cast<int*>(device_ids), device_id, &ctx->raw_ptr, ctx->size);

    // Construct Tensor
    c10::DataPtr data_ptr(ctx->raw_ptr, ctx, _pgl_tensor_deleter,
        c10::Device(c10::DeviceType::CUDA, device_id));
    at::TensorOptions options = at::TensorOptions().dtype(dtype).device(torch::kCUDA, device_id);
    at::Storage storage = at::Storage({}, ctx->size, std::move(data_ptr), nullptr, false);
    torch::Tensor tensor = at::empty(0, options).set_(storage, 0, at::IntArrayRef(sizes.data(), sizes.size()), {});
    tensor.set_requires_grad(requires_grad);

    // Sanity check. Can be removed in production code
    TORCH_CHECK(tensor.is_contiguous(), "Tensor must be contiguous");

    return tensor;
}

torch::Tensor pgl_tensor(
    const std::vector<int64_t> &sizes,
    const at::ScalarType dtype,
    const std::vector<int> &device_ids,
    const int device_id,
    const bool requires_grad
) {
    TORCH_CHECK(device_id >= 0 && device_id < static_cast<int>(device_ids.size()), "Invalid device ID");
    return pgl_tensor(sizes, dtype, device_ids.data(), device_id, requires_grad);
}

torch::Tensor pgl_tensor(
    const torch::Tensor &other, 
    const std::vector<int> &device_ids, 
    const int device_id
) {
    TORCH_CHECK(device_id >= 0 && device_id < static_cast<int>(device_ids.size()), "Invalid device ID");

    bool on_gpu = other.device().is_cuda();
    if (on_gpu) {
        std::cerr << "WARNING (pgl_tensor): the given tensor is already on GPU. "
                  << "This will result in a redundant memory allocation and copy.\n";
    }
    
    // Allocate CUDA memory
    pgl_tensor_context *ctx = new pgl_tensor_context;
    ctx->device_id = device_id;
    ctx->raw_ptr = nullptr;
    ctx->size = other.nbytes();
    pglCudaMalloc<true>(NUM_DEVICES, const_cast<int*>(device_ids.data()), device_id, &ctx->raw_ptr, ctx->size);

    // Copy data
    hipMemcpyKind copy_kind = on_gpu ? hipMemcpyDeviceToDevice : hipMemcpyHostToDevice;
    hipMemcpy(ctx->raw_ptr, other.data_ptr(), ctx->size, copy_kind);

    // Construct Tensor (this is required because data_ptr is a smart pointer)
    c10::DataPtr data_ptr(ctx->raw_ptr, ctx, _pgl_tensor_deleter,
        c10::Device(c10::DeviceType::CUDA, device_id));
    at::TensorOptions options = other.options().device(torch::kCUDA, device_id); // includes dtype, device, layout
    at::Storage storage = at::Storage({}, ctx->size, std::move(data_ptr), nullptr, false);
    torch::Tensor tensor = at::empty(0, options).set_(storage, 0, other.sizes(), {});
    if (other.requires_grad()) tensor.set_requires_grad(true);

    // Sanity check. Can be removed in production code
    TORCH_CHECK(tensor.is_contiguous(), "Tensor must be contiguous");

    return tensor;
}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.doc() = "ThunderKittens Ring Attention Kernels";
    m.def(
        "ring_mha_forward",  
        torch::wrap_pybind_function(ring_attention_forward),
        "Forward ring MHA"
    );
    m.def(
        "ring_mha_backward", 
        torch::wrap_pybind_function(ring_attention_backward), 
        "Backward ring MHA"
    );
    m.def(
        "pgl_tensor", 
        static_cast<torch::Tensor(*)(const torch::Tensor&, const std::vector<int>&, const int)>(&pgl_tensor),
        "Create a PGL tensor from existing tensor"
    );
    m.def(
        "pgl_tensor", 
        static_cast<torch::Tensor(*)(const std::vector<int64_t>&, const at::ScalarType, const std::vector<int>&, const int, const bool)>(&pgl_tensor),
        "Create a new PGL tensor from sizes and dtype"
    );
}

#else

#endif
