#include "hip/hip_runtime.h"
#include "conversions.cuh"

#ifdef TEST_WARP_REGISTER_CONVERSIONS

void warp::reg::conversions::tests(test_data &results) {
    std::cout << " ----- Starting ops/warp/register/conversions tests! -----" << std::endl;
    constexpr int SIZE = INTENSITY_1 ? 2  :
                         INTENSITY_2 ? 4  : 
                         INTENSITY_3 ? 8  :
                         INTENSITY_4 ? 16 : -1;
    sweep_size_2d_warp<warp::reg::conversions::swap_layout, SIZE, SIZE, ducks::rt_layout::row>::run(results);
    sweep_size_2d_warp<warp::reg::conversions::swap_layout, SIZE, SIZE, ducks::rt_layout::col>::run(results);
    sweep_size_2d_warp<warp::reg::conversions::transpose, SIZE, SIZE, ducks::rt_layout::row>::run(results);
    sweep_size_2d_warp<warp::reg::conversions::transpose, SIZE, SIZE, ducks::rt_layout::col>::run(results);
    sweep_size_2d_warp<warp::reg::conversions::type_convert, SIZE, SIZE, float2, bf16_2>::run(results);
    sweep_size_2d_warp<warp::reg::conversions::type_convert, SIZE, SIZE, bf16_2, float2>::run(results);
    sweep_size_2d_warp<warp::reg::conversions::subtile, SIZE, SIZE, std::integral_constant<int, 1>>::run(results);
    sweep_size_2d_warp<warp::reg::conversions::subtile, SIZE, SIZE, std::integral_constant<int, 2>>::run(results);
    sweep_size_2d_warp<warp::reg::conversions::subtile, SIZE, SIZE, std::integral_constant<int, 3>>::run(results);
    sweep_size_2d_warp<warp::reg::conversions::subtile, SIZE, SIZE, std::integral_constant<int, 4>>::run(results);
}

#endif