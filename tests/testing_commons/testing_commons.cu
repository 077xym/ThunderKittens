#include "hip/hip_runtime.h"
#include "testing_commons.cuh"

// Explicit specializations

template<> std::string layout_name<kittens::ducks::st_layout::naive          >() { return "naive";           }
template<> std::string layout_name<kittens::ducks::st_layout::tma_swizzle    >() { return "tma_swizzle";     }
template<> std::string layout_name<kittens::ducks::st_layout::xor_swizzle    >() { return "xor_swizzle";     }
template<> std::string layout_name<kittens::ducks::st_layout::wgmma_row_0b   >() { return "wgmma_row_0b";    }
template<> std::string layout_name<kittens::ducks::st_layout::wgmma_row_32b  >() { return "wgmma_row_32b";   }
template<> std::string layout_name<kittens::ducks::st_layout::wgmma_col_t_0b >() { return "wgmma_col_t_0b";  }
template<> std::string layout_name<kittens::ducks::st_layout::wgmma_col_t_32b>() { return "wgmma_col_t_32b"; }

int should_write_outputs;
test_result validate(kittens::bf16 *d_i, kittens::bf16 *d_o, const std::vector<float> &i_ref, std::vector<float> &o_ref, std::string test_name, int cols, float eps) {
    using namespace kittens;
    const int input_size  = i_ref.size();
    const int output_size = o_ref.size();
    // copy back
    bf16* o_bf = new bf16[output_size];
    float *o = new float[output_size];
    hipDeviceSynchronize();
    CudaCheckError();
    hipMemcpy(o_bf, d_o, output_size * sizeof(bf16), hipMemcpyDeviceToHost);
    CudaCheckError();
    for(int idx = 0; idx < output_size; idx++) {
        o[idx] = __bfloat162float(o_bf[idx]);
        o_ref[idx] = __bfloat162float(__float2bfloat16(o_ref[idx]));
    }
    // check
    std::cout << "test `" << test_name << "`";
    bool good = true;
    for(int i = 0; i < output_size; i++) {
        if(abs(o_ref[i] - o[i]) > eps) {
            good = false;
            break;
        }
    }
    if(good) std::cout << " -- PASSED" << std::endl;
    else std::cout << " ----- ALERT! FAILED test `" << test_name << "` -----" << std::endl;
    if(should_write_outputs && !good) {
        std::ofstream reffile("outputs/"+test_name+"_ref.txt");
        std::ofstream outfile("outputs/"+test_name+"_out.txt");
        for(int i = 0; i < output_size; i++) {
            reffile << o_ref[i] << ' ';
            outfile << o[i] << ' ';
            if(i%cols == cols-1) {
                reffile << '\n';
                outfile << '\n';
            }
        }
        reffile << "\n\n\nINPUTS:\n\n";
        for(int i = 0; i < input_size; i++) {
            reffile << i_ref[i] << ' ';
            if(i%cols == cols-1) {
                reffile << '\n';
            }
        }
        reffile.close();
        outfile.close();
    }
    hipFree(d_i);
    hipFree(d_o);
    delete[] o_bf, o;
    CudaCheckError();
    return good ? test_result::PASSED : test_result::FAILED;
}