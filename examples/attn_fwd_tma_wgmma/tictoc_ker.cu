#include "hip/hip_runtime.h"
#include <hip/hip_cooperative_groups.h>

#include "../../src/kittens.cuh"


#define NUM_WORKERS 16 // to reduce __syncwarp() stalls.
#define WARPGROUP_SIZE 4
#define NUM_WARPGROUPS (NUM_WORKERS/WARPGROUP_SIZE)

#define QO_BLOCKS 1 // 4 is also close? 

// shared tile
#define qo_height 4
#define kv_height 4
#define NUM_WORKERS_KV (NUM_WORKERS/kv_height)

// register tile
#define width  (ATTN_D/16)

#define ATTN_B 16
#define ATTN_H 16
#define ATTN_N 4096
#define ATTN_D 64 // hardcoded into this kernel
#define BLOCK_SIZE (32*NUM_WORKERS)

#define KITTENS_HOPPER

using namespace kittens;

using layout_row = ducks::st_layout::wgmma_row_0b;
using layout_col = ducks::st_layout::wgmma_col_t_0b;

template<int N>
__global__ void attend_ker(int d, CUtensorMap* tma_q, CUtensorMap* tma_k, CUtensorMap* tma_v, CUtensorMap* tma_o, 
                            const bf16* __restrict__ __q__, const bf16* __restrict__ __k__, const bf16* __restrict__ __v__, bf16* __o__)
{

    extern __shared__ alignment_dummy __shm[]; // this is the CUDA shared memory
    shared_allocator al = shared_allocator::create_allocator((int*)&__shm[0]);

    st_bf<qo_height, width, layout_row> (&q_smem)[2][NUM_WARPGROUPS] = al.allocate<st_bf<qo_height, width, layout_row>, NUM_WARPGROUPS, 2>();
    st_bf<kv_height, width, layout_row> (&k_smem)[2][NUM_WORKERS_KV] = al.allocate<st_bf<kv_height, width, layout_row>, NUM_WORKERS_KV, 2>();
    st_bf<kv_height, width, layout_col> (&v_smem)[2][NUM_WORKERS_KV] = al.allocate<st_bf<kv_height, width, layout_col>, NUM_WORKERS_KV, 2>();

    int tic = 0; 
    int toc = 1; 

    int sic = 0; 
    int soc = 1; 
 
    rt_fl<1, kv_height> att_block;
    rt_bf<1, kv_height> att_block_mma;
    rt_fl<1, width> o_prev;
    rt_fl<1, kv_height>::col_vec max_vec_last, max_vec;
    rt_fl<1, kv_height>::col_vec norm_vec_last, norm_vec;

    int warpid      = kittens::warpid();
    int warpgroupid = warpid/WARPGROUP_SIZE; 

    constexpr int qo_tiles  = N / q_smem[0][0].rows; 
    constexpr int kv_blocks = N / (NUM_WORKERS_KV*k_smem[0][0].rows);
    auto block = cooperative_groups::this_thread_block();

    // __shared__ cuda::barrier<cuda::thread_scope::thread_scope_block> q_barrier;
    // if (threadIdx.x == 0) {init(&q_barrier, block.size());}

    __shared__ uint64_t qsmem_barrier[1]; 
    __shared__ uint64_t ksmem_barrier[1];
    __shared__ uint64_t vsmem_barrier[1];

    constexpr int tile_bytes = sizeof(bf16) * k_smem[0][0].num_elements * NUM_WORKERS_KV;
    int tile_idx;

    if (warpid == 0) {
        tma::init_barrier(qsmem_barrier[0], block.size()); 
        tma::set_barrier_bytes(qsmem_barrier[0], tile_bytes); 

        tma::init_barrier(ksmem_barrier[0], block.size());
        tma::set_barrier_bytes(ksmem_barrier[0], tile_bytes);

        tma::init_barrier(vsmem_barrier[0], block.size());
        tma::set_barrier_bytes(vsmem_barrier[0], tile_bytes);
    }

    constexpr int kPhaseBit_q = 1;
    constexpr int kPhaseBit_k = 1; 
    constexpr int kPhaseBit_v = 1;

    block.sync();

    // warpgroup::load_async(q_smem[sic][warpgroupid], __q__ + (blockIdx.y*N*d) + ((blockIdx.x)*NUM_WARPGROUPS + warpgroupid) * q_smem[0][0].num_elements, d, q_barrier); 

    if (warpid == 0) {
        for (int wg = 0; wg < 4; wg++) {
            if constexpr (QO_BLOCKS == 1) { 
                tile_idx = (blockIdx.y * NUM_WARPGROUPS * blockDim.x) + 
                            (blockIdx.x * NUM_WARPGROUPS) + warpgroupid + wg; 
            }
            else {
                tile_idx = (blockIdx.y * qo_tiles) + 
                            (blockIdx.x * QO_BLOCKS * NUM_WARPGROUPS) + 
                            (0 * NUM_WARPGROUPS) + (wg);
            }

            tma::load_async(q_smem[sic][wg], tma_q, tile_idx, qsmem_barrier[0]); 
        }
        
        for (int w = 0; w < NUM_WORKERS_KV; w++) {        
            tile_idx = (blockIdx.y * NUM_WORKERS_KV * kv_blocks) + (0 * NUM_WORKERS_KV) + warpid + w; 
            tma::load_async(k_smem[tic][w], tma_k, tile_idx, ksmem_barrier[0]); 
            tma::load_async(v_smem[tic][w], tma_v, tile_idx, vsmem_barrier[0]); 
        }
    }

    for (auto q_blk = 0; q_blk < QO_BLOCKS; q_blk++) {

        neg_infty(max_vec); // zero registers for the Q chunk
        zero(norm_vec);
        zero(o_prev);

        tma::arrive_and_wait(qsmem_barrier[0], kPhaseBit_q); 

        if (threadIdx.x == 0 && QO_BLOCKS > 1) {
            tma::init_barrier(qsmem_barrier[0], block.size()); 
            tma::set_barrier_bytes(qsmem_barrier[0], tile_bytes);
        }
        __syncthreads(); 

        if ((q_blk + 1 < QO_BLOCKS) && (warpid == 0)) {
            for (int wg = 0; wg < 4; wg++) {
                tile_idx = (blockIdx.y * qo_tiles) + 
                            (blockIdx.x * QO_BLOCKS * NUM_WARPGROUPS) + 
                            (((q_blk + 1) % QO_BLOCKS) * NUM_WARPGROUPS) + (wg);
                
                tma::load_async(q_smem[soc][wg], tma_q, tile_idx, qsmem_barrier[0]); 
            }
        }

        warpgroup::mul(q_smem[sic][warpgroupid], q_smem[sic][warpgroupid], __float2bfloat16(0.125f));

        for(auto kv_idx = 0; kv_idx < kv_blocks; kv_idx++) {

            tma::arrive_and_wait(ksmem_barrier[0], kPhaseBit_k); 
            tma::arrive_and_wait(vsmem_barrier[0], kPhaseBit_v); 

            if ((threadIdx.x == 0) && kv_blocks > 1) {
                tma::init_barrier(ksmem_barrier[0], block.size()); 
                tma::set_barrier_bytes(ksmem_barrier[0], tile_bytes); 

                tma::init_barrier(vsmem_barrier[0], block.size()); 
                tma::set_barrier_bytes(vsmem_barrier[0], tile_bytes); 
            }
            __syncthreads();

            if ((kv_idx + 1 < kv_blocks) && (warpid == 0)) {
                for (int w = 0; w < NUM_WORKERS_KV; w++) {        
                    tile_idx = (blockIdx.y * NUM_WORKERS_KV * kv_blocks) + ((kv_idx + 1) * NUM_WORKERS_KV) + warpid + w; 
                    tma::load_async(k_smem[toc][w], tma_k, tile_idx, ksmem_barrier[0]); 
                    tma::load_async(v_smem[toc][w], tma_v, tile_idx, vsmem_barrier[0]); 
                }
            }
            else if ((q_blk + 1 < QO_BLOCKS) && (warpid == 0)) {
                for (int w = 0; w < NUM_WORKERS_KV; w++) {        
                    tile_idx = (blockIdx.y * NUM_WORKERS_KV * kv_blocks) + (0 * NUM_WORKERS_KV) + warpid + w; 
                    tma::load_async(k_smem[toc][w], tma_k, tile_idx, ksmem_barrier[0]); 
                    tma::load_async(v_smem[toc][w], tma_v, tile_idx, vsmem_barrier[0]); 
                }
            }

            for(int subtile = 0; subtile < NUM_WORKERS_KV; subtile++) {
                warpgroup::fence(att_block);
                warpgroup::dot_reset(att_block, q_smem[sic][warpgroupid], k_smem[tic][subtile]);
                warpgroup::mma_commit_group();

                copy(norm_vec_last, norm_vec);
                copy(max_vec_last,  max_vec);

                warpgroup::mma_async_wait();

                row_max(max_vec, att_block, max_vec); // accumulate onto the max_vec
                sub_row(att_block, att_block, max_vec);
                exp(att_block, att_block);

                sub(max_vec_last, max_vec_last, max_vec);
                exp(max_vec_last, max_vec_last);
                mul(norm_vec, norm_vec, max_vec_last);

                row_sum(norm_vec, att_block, norm_vec); // accumulate onto the norm_vec
                div_row(att_block, att_block, norm_vec);

                mul(norm_vec_last, norm_vec_last, max_vec_last);
                div(norm_vec_last, norm_vec_last, norm_vec);

                copy(att_block_mma, att_block); // convert to bf16 for mma
                mul_row(o_prev, o_prev, norm_vec_last); // normalize o_prev in advance of mma'ing onto it

                warpgroup::fence(o_prev);
                warpgroup::mma_accum(o_prev, att_block_mma, v_smem[tic][subtile]);
                warpgroup::mma_commit_group();
            }

            tic ^= 1;
            toc ^= 1;
        }

        if constexpr (QO_BLOCKS == 1) {
            store(__o__ + (blockIdx.y*N*d) + ((blockIdx.x)*NUM_WORKERS + warpid) * (q_smem[0][0].num_elements/WARPGROUP_SIZE), o_prev, d);
            // warpgroup::store(__o__ + (blockIdx.y*N*d) + ((blockIdx.x)*NUM_WARPGROUPS + warpgroupid) * q_smem[0][0].num_elements, o_prev, d);
        }
        else {
            tma::store_async_wait();
            warpgroup::store(q_smem[sic][warpgroupid], o_prev); 

            if (warpid == 0) {
                for (int wg = 0; wg < 4; wg++) {
                    if constexpr (QO_BLOCKS == 1) {
                        tile_idx = (blockIdx.y * NUM_WARPGROUPS * blockDim.x) + 
                                    (blockIdx.x * NUM_WARPGROUPS) + warpgroupid + wg; 
                    }
                    else {
                        tile_idx = (blockIdx.y * qo_tiles) + 
                                    (blockIdx.x * QO_BLOCKS * NUM_WARPGROUPS) + 
                                    (q_blk * NUM_WARPGROUPS) + (wg);
                    }
                    
                    tma::store_async(tma_o, q_smem[sic][wg], tile_idx); 
                    tma::store_commit_group(); 
                }
            }
        }

        sic ^= 1; 
        soc ^= 1; 
    }

    if constexpr (QO_BLOCKS != 1) {
        tma::store_async_wait();
    }
}

#include "harness.impl"