#include "hip/hip_runtime.h"
#include "src/kittens.cuh"

#define NUM_WORKERS (4) // this comes from the fact that we want a 64-long sliding window
using namespace kittens;

#define WINDOW_WIDTH (64)
static_assert(WINDOW_WIDTH%64==0 && WINDOW_WIDTH<=256);
#define WINDOW_TILES ((WINDOW_WIDTH/64)+1)
#define WINDOW_MINI_TILES ((WINDOW_WIDTH/16)+1)

__global__ __launch_bounds__(NUM_WORKERS*kittens::WARP_THREADS, 2)
void sliding_window(int n, int d, const bf16* __restrict__ __q__, const bf16* __restrict__ __k__, const bf16* __restrict__ __v__, bf16* __o__) {

    using G = kittens::group<NUM_WORKERS>;

    auto warpid        = kittens::warpid();
    auto block_start   = blockIdx.x*(n*64);
    const bf16 *_q = __q__ + block_start, *_k = __k__ + block_start, *_v = __v__ + block_start;
          bf16 *_o = __o__ + block_start;

    extern __shared__ alignment_dummy __shm[]; // this is the CUDA shared memory
    shared_allocator al((int*)&__shm[0]);
    
    st_bf_1x4<ducks::st_layout::xor_swizzle> (&k_smem)[WINDOW_TILES][NUM_WORKERS] = al.allocate<st_bf_1x4<ducks::st_layout::xor_swizzle>, WINDOW_TILES, NUM_WORKERS>();
    st_bf_1x4<ducks::st_layout::xor_swizzle> (&v_smem)[WINDOW_TILES][NUM_WORKERS] = al.allocate<st_bf_1x4<ducks::st_layout::xor_swizzle>, WINDOW_TILES, NUM_WORKERS>();

    rt_bf_1x4<> q_reg, k_reg, v_reg;
    rt_fl_1x1<> att_block[WINDOW_MINI_TILES];
    rt_bf_1x1<> att_block_bf;
    rt_fl_1x4<> o_reg;
    rt_fl_1x1<>::col_vec max_vec, norm_vec;
    
    int qo_blocks = n / (q_reg.rows*NUM_WORKERS), kv_blocks = n / (q_reg.rows*NUM_WORKERS);

    int start_block = 0, last_block = WINDOW_TILES-1;
    for(auto qo_blk = 0; qo_blk < qo_blocks; qo_blk++, start_block=(start_block+1)%WINDOW_TILES, last_block=(last_block+1)%WINDOW_TILES) {

        __syncthreads(); // we need to make sure all warps are done before we can start loading the next kv chunk

        // load the curent k, v blocks into last_block. If qo_blk > 0, then the previous tiles stick around.
        load(k_smem[last_block][warpid], _k + (qo_blk*NUM_WORKERS + warpid)*q_reg.num_elements, q_reg.cols);
        load(v_smem[last_block][warpid], _v + (qo_blk*NUM_WORKERS + warpid)*q_reg.num_elements, q_reg.cols);

        // load q registers
        load(q_reg, _q + (qo_blk*NUM_WORKERS + warpid)*q_reg.num_elements, q_reg.cols);
        mul(q_reg, q_reg, __float2bfloat16(0.125f)); // temperature adjustment

        neg_infty(max_vec); // zero registers for the Q chunk
        zero(norm_vec);
        zero(o_reg);

        __syncthreads(); // we need to make sure all memory is loaded before we can begin the compute phase

        for(int subtile = 0; subtile < WINDOW_MINI_TILES; subtile++) {
            int src_idx = warpid+subtile;
            if (4*qo_blk + src_idx >= 4*(WINDOW_TILES-1)) {

                load(k_reg, k_smem[(start_block+(src_idx/4))%WINDOW_TILES][src_idx%4]);

                zero(att_block[subtile]);
                dot(att_block[subtile], q_reg, k_reg, att_block[subtile]);
                if(subtile == WINDOW_MINI_TILES-1) {
                    // last tile becomes causal
                    make_causal(att_block[subtile], att_block[subtile], base_types::constants<float>::neg_infty());
                }
            }
            else {
                neg_infty(att_block[subtile]); // initial blocks must be zero
            }
        }
        // now do the softmax. first we subtract max for numerical stability. then exp.
        #pragma unroll
        for(int subtile = 0; subtile < WINDOW_MINI_TILES; subtile++) {
            row_max(max_vec, att_block[subtile], max_vec); // accumulate onto the max_vec
        }
        #pragma unroll
        for(int subtile = 0; subtile < WINDOW_MINI_TILES; subtile++) {
            sub_row(att_block[subtile], att_block[subtile], max_vec);
            exp(att_block[subtile], att_block[subtile]);
        }
        // now we sum so that we can divide.
        #pragma unroll
        for(int subtile = 0; subtile < WINDOW_MINI_TILES; subtile++) {
            row_sum(norm_vec, att_block[subtile], norm_vec);
        }
        #pragma unroll
        for(int subtile = 0; subtile < WINDOW_MINI_TILES; subtile++) {
            div_row(att_block[subtile], att_block[subtile], norm_vec);
        }
        for(int subtile = 0; subtile < WINDOW_MINI_TILES; subtile++) {
            int src_idx = warpid+subtile;
            load(v_reg, v_smem[(start_block+(src_idx/4))%WINDOW_TILES][src_idx%4]);
            rt_bf_1x4<ducks::rt_layout::col> &v_reg_col = swap_layout_inplace(v_reg); // this is a reference and the call has invalidated v_reg

            copy(att_block_bf, att_block[subtile]);
            mma(o_reg, att_block_bf, v_reg_col, o_reg); // accumulate
        }

        store(_o + (qo_blk*NUM_WORKERS + warpid)*q_reg.num_elements, o_reg, d); // write out o. compiler has an issue with register usage if d is made constexpr q_reg.rows :/
    }
}

#include "harness.impl"