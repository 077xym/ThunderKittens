#include "hip/hip_runtime.h"
#include "../../../src/kittens.cuh"
#include <hip/hip_cooperative_groups.h>
using namespace kittens;

constexpr int NUM_WORKERS = 8, NUM_WARPGROUPS = (NUM_WORKERS/WARPGROUP_WARPS);
using layout_q = ducks::st_layout::wgmma_swizzle;
using layout_k = ducks::st_layout::wgmma_swizzle;
using layout_v = ducks::st_layout::wgmma_interleave; // must support imm-trans-b
using layout_o = ducks::st_layout::swizzle; // fastest write out
template<int D> struct fwd_attend_ker_tile_dims {
    static_assert(D==64 || D==128);
    constexpr static int tile_width = D/kittens::TILE_DIM;
    constexpr static int qo_height  = 4;
    constexpr static int kv_height  = 512/D;
};

template<int D, int N> __global__  __launch_bounds__(NUM_WORKERS*kittens::WARP_THREADS, 2)
void fwd_attend_ker(CUtensorMap* tma_q, CUtensorMap* tma_k, CUtensorMap* tma_v, CUtensorMap* tma_o) {
    extern __shared__ int __shm[]; // dynamic shared memory
    tma_swizzle_allocator al((int*)&__shm[0]); // lightweight allocator enforces alignments.

    constexpr int tile_width = fwd_attend_ker_tile_dims<D>::tile_width; // constants
    constexpr int qo_height  = fwd_attend_ker_tile_dims<D>::qo_height;
    constexpr int kv_height  = fwd_attend_ker_tile_dims<D>::kv_height;
    constexpr int kv_blocks  = N / (kv_height*TILE_DIM);

    st_bf<qo_height, tile_width, layout_q> (&q_smem)[NUM_WARPGROUPS] = al.allocate<st_bf<qo_height, tile_width, layout_q>, NUM_WARPGROUPS>(); // shared tiles
    st_bf<kv_height, tile_width, layout_k> (&k_smem)[2]              = al.allocate<st_bf<kv_height, tile_width, layout_k>, 2>();
    st_bf<kv_height, tile_width, layout_v> (&v_smem)[2]              = al.allocate<st_bf<kv_height, tile_width, layout_v>, 2>();

    rt_fl<1, kv_height> att_block; // declare registers
    rt_bf<1, kv_height> att_block_mma;
    rt_fl<1, tile_width> o_accum;
    col_vec<rt_fl<1, kv_height>> max_vec_last, max_vec;
    col_vec<rt_fl<1, kv_height>> norm_vec_last, norm_vec;

    int warpid      = kittens::warpid(); // who am i? when am i?
    int warpgroupid = warpid/kittens::WARPGROUP_WARPS; 
    int tic = 0, toc = 1, phase = 0; // since we have two barriers, we need a half-rate tic as the phase bit

    __shared__ uint64_t qsmem_barrier, ksmem_barrier[2], vsmem_barrier[2]; // initialize barriers
    if      (warpid == 0) tma::init_barrier<typeof(q_smem[0]), NUM_WARPGROUPS>(qsmem_barrier);
    else if (warpid == 1) tma::init_barrier<typeof(k_smem[0])>(ksmem_barrier[tic]);
    else if (warpid == 2) tma::init_barrier<typeof(v_smem[0])>(vsmem_barrier[tic]);
    else if (warpid == 3) tma::init_barrier(ksmem_barrier[toc]);
    else if (warpid == 4) tma::init_barrier(vsmem_barrier[toc]);
    __syncthreads();

    if (warpid%4 == 0) { // load q from HBM
        int tile_idx = (blockIdx.y * NUM_WARPGROUPS * blockDim.x) + (blockIdx.x * NUM_WARPGROUPS) + warpgroupid;
        tma::load_async(q_smem[warpgroupid], tma_q, qsmem_barrier, tile_idx);
    }
    if      (warpid == 0) tma::load_async(k_smem[tic], tma_k, ksmem_barrier[tic], blockIdx.y*kv_blocks); // load initial k, v from HBM
    else if (warpid == 1) tma::load_async(v_smem[tic], tma_v, vsmem_barrier[tic], blockIdx.y*kv_blocks);

    neg_infty(max_vec); // zero registers, while we wait
    zero(norm_vec);
    zero(o_accum);

    tma::arrive_and_wait(qsmem_barrier, 0); // wait for memory to arrive
    if constexpr (D==64)  warpgroup::mul(q_smem[warpgroupid], q_smem[warpgroupid], __float2bfloat16(0.125f)); // temperature adjustment
    if constexpr (D==128) warpgroup::mul(q_smem[warpgroupid], q_smem[warpgroupid], __float2bfloat16(0.08838834764f)); // temperature adjustment

    for(auto kv_idx = 0; kv_idx < kv_blocks; kv_idx++, tic ^= 1, toc ^= 1) {

        tma::arrive_and_wait(ksmem_barrier[tic], phase); // wait for kv memory to arrive
        __syncthreads(); // everybody on the same page?
        if (warpid == 0) { // go get the next K from HBM
            tma::set_bytes(ksmem_barrier[toc], detail::transfer_bytes<typeof(k_smem[0])>::bytes);
            if (kv_idx+1 < kv_blocks) tma::load_async(k_smem[toc], tma_k, ksmem_barrier[toc], (blockIdx.y * kv_blocks) + kv_idx + 1);
        }
        else if (warpid == 1) { // go get the next V from HBM
            tma::set_bytes(vsmem_barrier[toc], detail::transfer_bytes<typeof(v_smem[0])>::bytes);
            if (kv_idx+1 < kv_blocks) tma::load_async(v_smem[toc], tma_v, vsmem_barrier[toc], (blockIdx.y * kv_blocks) + kv_idx + 1);
        }

        warpgroup::mma_fence(att_block); // qk matmul fence
        warpgroup::mm_ABt(att_block, q_smem[warpgroupid], k_smem[tic]); // clear registers -- note mm_ABt, not mma_ABt.
        warpgroup::mma_commit_group(); // dew it

        copy(norm_vec_last, norm_vec); // copy registers, while we wait
        copy(max_vec_last,  max_vec);

        warpgroup::mma_async_wait(); // ding dong! matmuls arrived.

        row_max(max_vec, att_block, max_vec); // accumulate new max onto the max_vec
        sub_row(att_block, att_block, max_vec); // ensure all <=0 for numerics
        exp(att_block, att_block); // exponentiate attention block for softmax in FP32
        sub(max_vec_last, max_vec_last, max_vec); // how do we need to normalize previous O's due to new max, in log?
        exp(max_vec_last, max_vec_last); // how do we need to norm previous O's due to new max, actually?
        mul(norm_vec, norm_vec, max_vec_last); // norm previous exp sum using new max
        row_sum(norm_vec, att_block, norm_vec); // accumulate new exp sum onto the norm_vec
        div_row(att_block, att_block, norm_vec); // softmax normalization of existing attention block
        mul(norm_vec_last, norm_vec_last, max_vec_last); // incorporate previous max into norm for o
        div(norm_vec_last, norm_vec_last, norm_vec); // incorporate current norm into new norm for o
        copy(att_block_mma, att_block); // convert to bf16 for mma
        mul_row(o_accum, o_accum, norm_vec_last); // normalize o in advance of mma'ing onto it

        tma::arrive_and_wait(vsmem_barrier[tic], phase); // wait for kv memory to arrive

        warpgroup::mma_fence(o_accum);  // av matmul fence
        warpgroup::mma_AB(o_accum, att_block_mma, v_smem[tic]); // mm accumulate next attention chunk onto o
        warpgroup::mma_commit_group(); // dew it.

        if(tic) phase^=1;
    }

    auto *o_smem = reinterpret_cast<st_bf<qo_height, tile_width, layout_o>*>(&q_smem[0].data[0]); // reuse q memory for store
    warpgroup::store(o_smem[warpgroupid], o_accum); // store from registers to shared mem
    __syncthreads(); // everyone done?
    if (warpid%4 == 0) { // store o to HBM
        int tile_idx = (blockIdx.y * NUM_WARPGROUPS * blockDim.x) + (blockIdx.x * NUM_WARPGROUPS) + warpgroupid; 
        tma::store_async(tma_o, (o_smem[warpgroupid]), tile_idx); 
        tma::store_commit_group(); // dew it
    }
    tma::store_async_wait(); // done it.
}

#include "harness.impl"