#include "hip/hip_runtime.h"
#include "../../src/kittens.cuh" // for harness_h100_fwd.impl
#include <cuda/pipeline>
#include <hip/hip_cooperative_groups.h>

#define NUM_WORKERS (8)
#define NUM_WARPGROUPS (NUM_WORKERS/(kittens::WARPGROUP_WARPS))
#define NUM_WORKERS_KV (1)

using namespace kittens;

template<ducks::rt::row_layout RT>
__device__ static inline void wg_make_causal(RT &dst, const RT &src, const typename base_types::packing<typename RT::dtype>::unpacked_type &val=0) {
    const typename RT::dtype packed_val = base_types::packing<typename RT::dtype>::pack(val);
    #pragma unroll
    for(int i = 0; i < dst.height; i++) {
        #pragma unroll
        for(int j = 0; j < dst.width; j++) {

            if(j < ((warpid() % kittens::WARPGROUP_WARPS) * dst.height) + i) { // below the diagonal, copy
                #pragma unroll
                for(int k = 0; k < dst.packed_per_tile; k++) {
                    dst.tiles[i][j].data[k] = src.tiles[i][j].data[k];
                }
            }
            else if(j > ((warpid() % kittens::WARPGROUP_WARPS) * dst.height) + i) { // above the diagonal, zero
                #pragma unroll
                for(int k = 0; k < dst.packed_per_tile; k++) {
                    dst.tiles[i][j].data[k] = packed_val;
                }
            }
            else { // on the diagonal, interesting!
                constexpr uint32_t MASK_X = 0xFF773311, MASK_Y = 0xF7733110; // magic numbers for on-diagonal core matrices
                dst.tiles[i][j].data[1] = src.tiles[i][j].data[1]; // below diagonal, copy
                dst.tiles[i][j].data[2] = packed_val; // above diagonal, zero
                if((MASK_X >> laneid()) & 1) {
                    dst.tiles[i][j].data[0].x = src.tiles[i][j].data[0].x;
                    dst.tiles[i][j].data[3].x = src.tiles[i][j].data[3].x;
                }
                else {
                    dst.tiles[i][j].data[0].x = val;
                    dst.tiles[i][j].data[3].x = val;
                }
                if((MASK_Y >> laneid()) & 1) {
                    dst.tiles[i][j].data[0].y = src.tiles[i][j].data[0].y;
                    dst.tiles[i][j].data[3].y = src.tiles[i][j].data[3].y;
                }
                else {
                    dst.tiles[i][j].data[0].y = val;
                    dst.tiles[i][j].data[3].y = val;
                }
            }
        }
    }
}

using layout_q = kittens::ducks::st_layout::wgmma_swizzle; 
using layout_k = kittens::ducks::st_layout::wgmma_swizzle; 
using layout_v = kittens::ducks::st_layout::wgmma_interleave; 
using layout_o = kittens::ducks::st_layout::swizzle;

__global__  __launch_bounds__((NUM_WORKERS)*kittens::WARP_THREADS, 2)
void fwd_attend_ker_dim64(int N, const CUtensorMap* tma_q, const CUtensorMap* tma_k, const CUtensorMap* tma_v, CUtensorMap* tma_o) {
    extern __shared__ int __shm[]; // this is the CUDA shared memory
    tma_swizzle_allocator al((int*)&__shm[0]);

    st_bf<4, 4, layout_q>          (&q_smem)   [NUM_WARPGROUPS] = al.allocate<st_bf<4, 4, layout_q>,          NUM_WARPGROUPS>();
    st_bf<4, 4, layout_k>          (&k_smem)[2][NUM_WORKERS_KV] = al.allocate<st_bf<4, 4, layout_k>, 2,       NUM_WORKERS_KV>();
    st_bf<4, 4, layout_v>          (&v_smem)[2][NUM_WORKERS_KV] = al.allocate<st_bf<4, 4, layout_v>, 2,       NUM_WORKERS_KV>();

    int tic = 0, toc = 1;
 
    rt_fl<1, 4> att_block;
    rt_bf<1, 4> att_block_mma;
    rt_fl<1, 4> o_prev;
    rt_fl<1, 4>::col_vec max_vec_last, max_vec;
    rt_fl<1, 4>::col_vec norm_vec_last, norm_vec;

    int warpid      = kittens::warpid();
    int warpgroupid = warpid/kittens::WARPGROUP_WARPS;

    int qo_index    = (blockIdx.x * NUM_WARPGROUPS) + warpgroupid;

    int kv_blocks = N / (NUM_WORKERS_KV*k_smem[0][0].rows);

    __shared__ uint64_t qsmem_barrier, kvsmem_barrier;//, vsmem_barrier;

    int q_phasebit = 0;
    int kv_phasebit = 0;

    if (threadIdx.x == 0) {
        tma::init_barrier<st_bf<4, 4, layout_q>, NUM_WARPGROUPS>(qsmem_barrier, 1);
        tma::init_barrier<st_bf<4, 4, layout_k>, NUM_WORKERS_KV*2>(kvsmem_barrier, 1); 
    }

    if (warpid == 0) {
        for (int wg = 0; wg < NUM_WORKERS/kittens::WARPGROUP_WARPS; wg++) { // load q
            int tile_idx = (blockIdx.y * NUM_WARPGROUPS * gridDim.x) + (blockIdx.x * NUM_WARPGROUPS) + wg;
            tma::load_async((q_smem[wg]), tma_q, qsmem_barrier, tile_idx); 
        }
        for (int w = 0; w < NUM_WORKERS_KV; w++) { // load k, v      
            int tile_idx = (blockIdx.y * NUM_WORKERS_KV * kv_blocks) + (0 * NUM_WORKERS_KV) + w; 
            tma::load_async((k_smem[tic][w]), tma_k, kvsmem_barrier, tile_idx); 
            tma::load_async((v_smem[tic][w]), tma_v, kvsmem_barrier, tile_idx); 
        }
    }

    neg_infty(max_vec); // zero registers for the Q chunk
    zero(norm_vec);
    zero(o_prev);
    __syncthreads();

    tma::arrive_and_wait(qsmem_barrier, q_phasebit);
    q_phasebit ^= 1;

    warpgroup::mul(q_smem[warpgroupid], q_smem[warpgroupid], __float2bfloat16(0.125f));

    for (auto kv_idx = 0; kv_idx <= qo_index; kv_idx++, tic ^= 1, toc ^= 1) {
        tma::arrive_and_wait(kvsmem_barrier, kv_phasebit);
        kv_phasebit ^= 1;

        __syncthreads();
        if (warpid == 0) {
            if (kv_idx + 1 < kv_blocks) {
                tma::set_bytes(kvsmem_barrier, 2 * NUM_WORKERS_KV * k_smem[0][0].num_elements * sizeof(bf16));
                
                for (int w = 0; w < NUM_WORKERS_KV; w++) {        
                    int tile_idx = (blockIdx.y * NUM_WORKERS_KV * kv_blocks) + ((kv_idx + 1) * NUM_WORKERS_KV) + w; 
                    tma::load_async((k_smem[toc][w]), tma_k, kvsmem_barrier, tile_idx); 
                    tma::load_async((v_smem[toc][w]), tma_v, kvsmem_barrier, tile_idx);
                }
            }
        }

        warpgroup::mma_fence(att_block);
        warpgroup::mm_ABt(att_block, q_smem[warpgroupid], k_smem[tic][0]);
        warpgroup::mma_commit_group();

        copy(norm_vec_last, norm_vec);
        copy(max_vec_last,  max_vec);

        warpgroup::mma_async_wait();

        if (kv_idx == qo_index) {
            wg_make_causal(att_block, att_block, -INFINITY); 
        }

        row_max(max_vec, att_block, max_vec); // accumulate onto the max_vec
        sub_row(att_block, att_block, max_vec);
        exp(att_block, att_block);

        sub(max_vec_last, max_vec_last, max_vec);
        exp(max_vec_last, max_vec_last);
        mul(norm_vec, norm_vec, max_vec_last);

        row_sum(norm_vec, att_block, norm_vec); // accumulate onto the norm_vec
        div_row(att_block, att_block, norm_vec);

        mul(norm_vec_last, norm_vec_last, max_vec_last);
        div(norm_vec_last, norm_vec_last, norm_vec);

        copy(att_block_mma, att_block); // convert to bf16 for mma
        mul_row(o_prev, o_prev, norm_vec_last); // normalize o_prev in advance of mma'ing onto it

        warpgroup::mma_fence(o_prev);
        warpgroup::mma_AB(o_prev, att_block_mma, v_smem[tic][0]);
        warpgroup::mma_commit_group();
    }

    auto (*o_smem) = reinterpret_cast<st_bf<4, 4, layout_o>(*)>(q_smem); // reuse q memory
    warpgroup::store(o_smem[warpgroupid], o_prev); 
    __syncthreads();
    
    if (warpid % 4 == 0) { // store o
        int tile_idx = (blockIdx.y * NUM_WARPGROUPS * gridDim.x) + (blockIdx.x * NUM_WARPGROUPS) + warpgroupid;
        tma::store_async(tma_o, (o_smem[warpgroupid]), tile_idx); 
        tma::store_commit_group(); 
    }

    tma::store_async_wait();
}

#include "harness_h100_fwd.impl"