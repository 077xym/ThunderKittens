#include "hip/hip_runtime.h"

// #include "src/kittens.cuh"
#include "../../src/kittens.cuh" // for harness_h100_fwd.impl
#include <cuda/pipeline>
#include <hip/hip_cooperative_groups.h>

#define ATTN_B 16
#define ATTN_H 16
#define ATTN_N 4096
#define ATTN_D 64

#define NUM_WORKERS 8
#define NUM_WARPGROUPS (NUM_WORKERS/(kittens::WARPGROUP_WARPS))

#define qo_height 4
#define kv_height 8
#define NUM_WORKERS_KV 1
#define tile_width 64/16

using namespace kittens;

using layout_q = ducks::st_layout::wgmma_swizzle; // need to make this 128b
using layout_k = ducks::st_layout::wgmma_swizzle; // need to make this 128b
using layout_v = ducks::st_layout::wgmma_interleave; // need to make this 128b
using layout_o = ducks::st_layout::swizzle;

__global__  __launch_bounds__((NUM_WORKERS)*kittens::WARP_THREADS, 2)
void attend_ker_fwd_train(CUtensorMap* tma_q, CUtensorMap* tma_k, CUtensorMap* tma_v, CUtensorMap* tma_o, CUtensorMap* tma_l) {
    extern __shared__ int __shm[]; // this is the CUDA shared memory
    tma_swizzle_allocator al((int*)&__shm[0]);

    st_bf<qo_height, tile_width, layout_q>          (&q_smem)   [NUM_WARPGROUPS] = al.allocate<st_bf<qo_height, tile_width, layout_q>,          NUM_WARPGROUPS>();
    st_bf<kv_height, tile_width, layout_k>          (&k_smem)[2][NUM_WORKERS_KV] = al.allocate<st_bf<kv_height, tile_width, layout_k>, 2,       NUM_WORKERS_KV>();
    st_bf<kv_height, tile_width, layout_v>          (&v_smem)[2][NUM_WORKERS_KV] = al.allocate<st_bf<kv_height, tile_width, layout_v>, 2,       NUM_WORKERS_KV>();
    st_bf<qo_height, tile_width, layout_o>::col_vec (&l_smem)   [NUM_WARPGROUPS] = al.allocate<st_bf<qo_height, tile_width, layout_o>::col_vec, NUM_WARPGROUPS>();

    int tic = 0, toc = 1;
 
    rt_fl<1, kv_height> att_block;
    rt_bf<1, kv_height> att_block_mma;
    rt_fl<1, tile_width> o_prev;
    rt_fl<1, kv_height>::col_vec max_vec_last, max_vec;
    rt_fl<1, kv_height>::col_vec norm_vec_last, norm_vec;

    int warpid      = kittens::warpid();
    int warpgroupid = warpid/kittens::WARPGROUP_WARPS;

    constexpr int kv_blocks = ATTN_N / (NUM_WORKERS_KV*k_smem[0][0].rows);

    __shared__ uint64_t qsmem_barrier, kvsmem_barrier;//, vsmem_barrier;

    int q_phasebit = 0;
    int kv_phasebit = 0;

    if (threadIdx.x == 0) {
        tma::init_barrier<st_bf<qo_height, tile_width, layout_q>, NUM_WARPGROUPS>(qsmem_barrier, 1);
        tma::init_barrier<st_bf<kv_height, tile_width, layout_k>, NUM_WORKERS_KV*2>(kvsmem_barrier, 1); 
    }

    if (warpid == 0) {
        for (int wg = 0; wg < NUM_WORKERS/kittens::WARPGROUP_WARPS; wg++) { // load q
            int tile_idx = (blockIdx.y * NUM_WARPGROUPS * gridDim.x) + (blockIdx.x * NUM_WARPGROUPS) + wg;
            tma::load_async((q_smem[wg]), tma_q, qsmem_barrier, tile_idx); 
        }
        for (int w = 0; w < NUM_WORKERS_KV; w++) { // load k, v      
            int tile_idx = (blockIdx.y * NUM_WORKERS_KV * kv_blocks) + (0 * NUM_WORKERS_KV) + w; 
            tma::load_async((k_smem[tic][w]), tma_k, kvsmem_barrier, tile_idx); 
            tma::load_async((v_smem[tic][w]), tma_v, kvsmem_barrier, tile_idx); 
        }
    }

    neg_infty(max_vec); // zero registers for the Q chunk
    zero(norm_vec);
    zero(o_prev);
    __syncthreads();

    tma::arrive_and_wait(qsmem_barrier, q_phasebit);
    q_phasebit ^= 1;

    warpgroup::mul(q_smem[warpgroupid], q_smem[warpgroupid], __float2bfloat16(0.125f));

    for (auto kv_idx = 0; kv_idx < kv_blocks; kv_idx++, tic ^= 1, toc ^= 1) {
        tma::arrive_and_wait(kvsmem_barrier, kv_phasebit);
        kv_phasebit ^= 1;

        __syncthreads();
        if (warpid == 0) {
            tma::set_bytes(kvsmem_barrier, 2 * NUM_WORKERS_KV * k_smem[0][0].num_elements * sizeof(bf16));

            if (kv_idx + 1 < kv_blocks) {
                for (int w = 0; w < NUM_WORKERS_KV; w++) {        
                    int tile_idx = (blockIdx.y * NUM_WORKERS_KV * kv_blocks) + ((kv_idx + 1) * NUM_WORKERS_KV) + w; 
                    tma::load_async((k_smem[toc][w]), tma_k, kvsmem_barrier, tile_idx); 
                    tma::load_async((v_smem[toc][w]), tma_v, kvsmem_barrier, tile_idx);
                }
            }
        }

        warpgroup::mma_fence(att_block);
        warpgroup::mm_ABt(att_block, q_smem[warpgroupid], k_smem[tic][0]);
        warpgroup::mma_commit_group();

        copy(norm_vec_last, norm_vec);
        copy(max_vec_last,  max_vec);

        warpgroup::mma_async_wait();

        row_max(max_vec, att_block, max_vec); // accumulate onto the max_vec
        sub_row(att_block, att_block, max_vec);
        exp(att_block, att_block);

        sub(max_vec_last, max_vec_last, max_vec);
        exp(max_vec_last, max_vec_last);
        mul(norm_vec, norm_vec, max_vec_last);

        row_sum(norm_vec, att_block, norm_vec); // accumulate onto the norm_vec
        div_row(att_block, att_block, norm_vec);

        mul(norm_vec_last, norm_vec_last, max_vec_last);
        div(norm_vec_last, norm_vec_last, norm_vec);

        copy(att_block_mma, att_block); // convert to bf16 for mma
        mul_row(o_prev, o_prev, norm_vec_last); // normalize o_prev in advance of mma'ing onto it

        warpgroup::mma_fence(o_prev);
        warpgroup::mma_AB(o_prev, att_block_mma, v_smem[tic][0]);
        warpgroup::mma_commit_group();
    }

    auto (*o_smem) = reinterpret_cast<st_bf<qo_height, tile_width, layout_o>(*)>(q_smem); // reuse q memory
    warpgroup::store(o_smem[warpgroupid], o_prev); 
    __syncthreads();
    
    if (warpid % 4 == 0) { // store o
        int tile_idx = (blockIdx.y * NUM_WARPGROUPS * gridDim.x) + (blockIdx.x * NUM_WARPGROUPS) + warpgroupid;
        tma::store_async(tma_o, (o_smem[warpgroupid]), tile_idx); 
        tma::store_commit_group(); 
    }

    log(norm_vec, norm_vec);
    add(norm_vec, norm_vec, max_vec);
    __syncthreads();

    warpgroup::store(l_smem[warpgroupid], norm_vec);
    __syncthreads();
    if (warpid % 4 == 0) {
        int tile_idx = (blockIdx.y * NUM_WARPGROUPS * gridDim.x) + (blockIdx.x * NUM_WARPGROUPS) + warpgroupid; 
        tma::store_async(tma_l, (l_smem[warpgroupid]), tile_idx); 
        tma::store_commit_group(); 
    }

    tma::store_async_wait();
}

#define WORKERS 4
using layout_nrow = ducks::st_layout::swizzle;

__global__  __launch_bounds__(WORKERS*kittens::WARP_THREADS, 2)
void attend_ker_prep_train(CUtensorMap* tma_o, CUtensorMap* tma_d, CUtensorMap* tma_o_grad) {
    extern __shared__ int __shm[]; // this is the CUDA shared memory
    tma_swizzle_allocator al((int*)&__shm[0]);

    int warpid = kittens::warpid();

    st_bf<4, 4, layout_nrow>          (&og_smem)[WORKERS] = al.allocate<st_bf<4, 4, layout_nrow>, WORKERS>();
    st_bf<4, 4, layout_nrow>          (&o_smem) [WORKERS] = al.allocate<st_bf<4, 4, layout_nrow>, WORKERS>();
    st_bf<4, 4, layout_nrow>::col_vec (&d_smem) [WORKERS] = al.allocate<st_bf<4, 4, layout_nrow>::col_vec, WORKERS>();

    rt_fl<4, 4> og_reg;
    rt_fl<4, 4> o_reg; 
    rt_fl<4, 4>::col_vec d_reg;

    __shared__ uint64_t smem_barrier;
    int o_phasebit = 0; 

    if (threadIdx.x == 0) {
        tma::init_barrier<st_bf<4, 4, layout_o>, WORKERS * 2>(smem_barrier, 1);
    }

    if (warpid == 0) {
        for (int w = 0; w < WORKERS; w++) { // load o, o_grad
            int tile_idx = (blockIdx.y * WORKERS * gridDim.x) + (blockIdx.x * WORKERS) + w; 
            tma::load_async((o_smem[w]),  tma_o,      smem_barrier, tile_idx); 
            tma::load_async((og_smem[w]), tma_o_grad, smem_barrier, tile_idx); 
        }
    }
    __syncthreads();

    tma::arrive_and_wait(smem_barrier, o_phasebit);
    o_phasebit ^= 1;

    load(o_reg, o_smem[warpid]);
    load(og_reg, og_smem[warpid]);

    mul(og_reg, og_reg, o_reg);
    row_sum(d_reg, og_reg);
    
    store(d_smem[warpid], d_reg);
    __syncthreads(); 

    if (warpid == 0) {
        for (int w = 0; w < WORKERS; w++) {
            int tile_idx = (blockIdx.y * WORKERS * gridDim.x) + (blockIdx.x * WORKERS) + w; 
            tma::store_async(tma_d, (d_smem[w]), tile_idx); 
        }
        tma::store_commit_group();
    }

    tma::store_async_wait();
}

#define WORKERS_BWD 8
#define WORKERS_BWD_QO 8 

#define NUM_WARPGROUPS_BWD    (WORKERS_BWD/(kittens::WARPGROUP_WARPS))
#define NUM_WARPGROUPS_BWD_QO (WORKERS_BWD_QO/(kittens::WARPGROUP_WARPS))

#define tile_h 4
#define tile_h_qo 4
#define tile_w 64/16

using layout_wgmma     = ducks::st_layout::wgmma_swizzle;
using layout_wgmma_itl = ducks::st_layout::wgmma_interleave;
using layout_tma_swi   = ducks::st_layout::swizzle; 

#define k_smem_tile  st_bf<tile_h, tile_w, layout_wgmma_itl>
#define v_smem_tile  st_bf<tile_h, tile_w, layout_wgmma>

#define q_smem_tile  st_bf<tile_h_qo, tile_w, layout_wgmma_itl>
#define og_smem_tile st_bf<tile_h_qo, tile_w, layout_wgmma_itl>
#define qg_smem_tile st_bf<tile_h_qo, tile_w, layout_tma_swi>
#define l_smem_tile  st_bf<tile_h_qo, tile_w, layout_tma_swi>::col_vec
#define d_smem_tile  st_bf<tile_h_qo, tile_w, layout_tma_swi>::col_vec

using namespace cooperative_groups;
namespace cg = cooperative_groups;

#define KV_BLOCKS 2

__global__ __launch_bounds__(WORKERS_BWD*kittens::WARP_THREADS, 1)
void attend_ker_bwd_train(CUtensorMap* tma_q, CUtensorMap* tma_k, CUtensorMap* tma_v, 
                            CUtensorMap* tma_l_vec, CUtensorMap* tma_d_vec, 
                            CUtensorMap* tma_og, CUtensorMap* tma_qg, CUtensorMap* tma_kg, CUtensorMap* tma_vg)
{
    extern __shared__ int __shm[]; // this is the CUDA shared memory
    tma_swizzle_allocator al((int*)&__shm[0]);

    k_smem_tile  (&k_smem) [NUM_WARPGROUPS_BWD] = al.allocate<k_smem_tile, NUM_WARPGROUPS_BWD>();
    v_smem_tile  (&v_smem) [NUM_WARPGROUPS_BWD] = al.allocate<v_smem_tile, NUM_WARPGROUPS_BWD>();

    q_smem_tile  (&q_smem) [2][NUM_WARPGROUPS_BWD_QO]                     = al.allocate<q_smem_tile,  2, NUM_WARPGROUPS_BWD_QO>();
    og_smem_tile (&og_smem)[2][NUM_WARPGROUPS_BWD_QO]                     = al.allocate<og_smem_tile, 2, NUM_WARPGROUPS_BWD_QO>();
    qg_smem_tile (&qg_smem)[2][NUM_WARPGROUPS_BWD_QO][NUM_WARPGROUPS_BWD] = al.allocate<qg_smem_tile, 2, NUM_WARPGROUPS_BWD_QO, NUM_WARPGROUPS_BWD>();
    
    l_smem_tile  (&l_smem) [2][NUM_WARPGROUPS_BWD_QO]                     = al.allocate<l_smem_tile,  2, NUM_WARPGROUPS_BWD_QO>();
    d_smem_tile  (&d_smem) [2][NUM_WARPGROUPS_BWD_QO]                     = al.allocate<d_smem_tile,  2, NUM_WARPGROUPS_BWD_QO>();

    rt_fl<tile_h/kittens::WARPGROUP_WARPS, tile_w> kg_reg;
    rt_fl<tile_h/kittens::WARPGROUP_WARPS, tile_w> vg_reg;

    rt_fl<tile_h_qo/kittens::WARPGROUP_WARPS, tile_w> qg_reg;

    rt_bf<tile_h_qo/kittens::WARPGROUP_WARPS, tile_h>::col_vec l_reg_bf; 
    rt_bf<tile_h_qo/kittens::WARPGROUP_WARPS, tile_h>::col_vec d_reg_bf;
    rt_fl<tile_h_qo/kittens::WARPGROUP_WARPS, tile_h>::col_vec l_reg_fl; 
    rt_fl<tile_h_qo/kittens::WARPGROUP_WARPS, tile_h>::col_vec d_reg_fl;

    rt_fl<tile_h_qo/kittens::WARPGROUP_WARPS, tile_h> att_block; 
    rt_bf<tile_h_qo/kittens::WARPGROUP_WARPS, tile_h> att_block_mma;
    rt_fl<tile_h_qo/kittens::WARPGROUP_WARPS, tile_h> temp_block;

    int warpid = kittens::warpid();
    int warpgroupid = warpid/kittens::WARPGROUP_WARPS;

    constexpr int qo_blocks = ATTN_N / (tile_h_qo * kittens::TILE_DIM * NUM_WARPGROUPS_BWD_QO);

    __shared__ uint64_t kv_b, qo_b, vec_b;

    int tic = 0, toc = 1;

    int kv_phasebit  = 0;
    int qo_phasebit  = 0;
    int vec_phasebit = 0;

    if (threadIdx.x == 0) {
        tma::init_barrier<q_smem_tile,  NUM_WARPGROUPS_BWD_QO * 2>(qo_b,  1); // q, og
        tma::init_barrier<k_smem_tile , NUM_WARPGROUPS_BWD    * 2>(kv_b,  1); // k, v
        tma::init_barrier<l_smem_tile , NUM_WARPGROUPS_BWD_QO * 2>(vec_b, 1); // l, d
    } 

    if (warpid == 0) {
        for (int w = 0; w < NUM_WARPGROUPS_BWD_QO; w++) {
            int tile_idx = (blockIdx.y * NUM_WARPGROUPS_BWD_QO * qo_blocks) + (0 * NUM_WARPGROUPS_BWD_QO) + w;

            tma::load_async((q_smem [tic][w]),    tma_q,     qo_b, tile_idx); 
            tma::load_async((og_smem[tic][w]),    tma_og,    qo_b, tile_idx);

            tma::load_async((l_smem[tic][w]),     tma_l_vec, vec_b, tile_idx);
            tma::load_async((d_smem[tic][w]),     tma_d_vec, vec_b, tile_idx);
        } 
    }

    for (int kv_idx = 0; kv_idx < KV_BLOCKS; kv_idx++) {
        
        if (warpid == 0) {
            // load k and v
            for (int w = 0; w < NUM_WARPGROUPS_BWD; w++) {
                int tile_idx = (blockIdx.y * NUM_WARPGROUPS_BWD * KV_BLOCKS * gridDim.x) + (blockIdx.x * NUM_WARPGROUPS_BWD * KV_BLOCKS) + (kv_idx * NUM_WARPGROUPS_BWD) + w;
                tma::load_async((k_smem[w]), tma_k, kv_b, tile_idx); 
                tma::load_async((v_smem[w]), tma_v, kv_b, tile_idx); 
            }
        }

        zero(kg_reg);
        zero(vg_reg);

        for (int qo_idx = 0; qo_idx < qo_blocks; qo_idx++, tic ^= 1, toc ^= 1) {
            
            tma::arrive_and_wait(vec_b, vec_phasebit);
            tma::arrive_and_wait(qo_b,  qo_phasebit);
            vec_phasebit ^= 1;
            qo_phasebit  ^= 1;

            if (qo_idx + 1 < qo_blocks) {
                if (threadIdx.x == 0) {
                    tma::set_bytes(qo_b,  NUM_WARPGROUPS_BWD_QO * sizeof(bf16) * q_smem[0][0].num_elements * 2); 
                    tma::set_bytes(vec_b, NUM_WARPGROUPS_BWD_QO * sizeof(bf16) * l_smem[0][0].length       * 2);
                }

                if (warpid == 0) {
                    for (int w = 0; w < NUM_WARPGROUPS_BWD_QO; w++) {
                        int tile_idx = (blockIdx.y * NUM_WARPGROUPS_BWD_QO * qo_blocks) + ((qo_idx + 1) * NUM_WARPGROUPS_BWD_QO) + w;
                        tma::load_async((q_smem [toc][w]),    tma_q,   qo_b, tile_idx); 
                        tma::load_async((og_smem[toc][w]),    tma_og,  qo_b, tile_idx);

                        tma::load_async((l_smem[toc][w]),     tma_l_vec, vec_b, tile_idx);
                        tma::load_async((d_smem[toc][w]),     tma_d_vec, vec_b, tile_idx);
                    }
                }
            }
            else if (kv_idx + 1 < KV_BLOCKS) {
                if (threadIdx.x == 0) {
                    tma::set_bytes(qo_b,  NUM_WARPGROUPS_BWD_QO * sizeof(bf16) * q_smem[0][0].num_elements * 2); 
                    tma::set_bytes(vec_b, NUM_WARPGROUPS_BWD_QO * sizeof(bf16) * l_smem[0][0].length       * 2);
                }

                if (warpid == 0) {
                    for (int w = 0; w < NUM_WARPGROUPS_BWD_QO; w++) {
                        int tile_idx = (blockIdx.y * NUM_WARPGROUPS_BWD_QO * qo_blocks) + (0 * NUM_WARPGROUPS_BWD_QO) + w;
                        tma::load_async((q_smem [toc][w]),    tma_q,   qo_b, tile_idx); 
                        tma::load_async((og_smem[toc][w]),    tma_og,  qo_b, tile_idx); 

                        tma::load_async((l_smem[toc][w]),     tma_l_vec, vec_b, tile_idx);
                        tma::load_async((d_smem[toc][w]),     tma_d_vec, vec_b, tile_idx);
                    }
                }
            } 

            if (qo_idx == 0) {
                tma::arrive_and_wait(kv_b, kv_phasebit);
                kv_phasebit ^= 1;

                if (KV_BLOCKS > 1) {
                    if (threadIdx.x == 0) {
                        tma::set_bytes(kv_b, NUM_WARPGROUPS_BWD * sizeof(bf16) * k_smem[0].num_elements * 2);
                    }
                }
            }

            if (qo_idx > 0 || kv_idx > 0) {
                tma::store_async_wait(); 
            }


            for (int subtile = 0; subtile < NUM_WARPGROUPS_BWD_QO; subtile++) {
                warpgroup::mma_fence(att_block);
                warpgroup::mm_ABt(att_block, q_smem[tic][subtile], k_smem[warpgroupid]);
                warpgroup::mma_commit_group();

                warpgroup::load(l_reg_bf, l_smem[tic][subtile]);
                copy(l_reg_fl, l_reg_bf);
                
                warpgroup::mma_async_wait();
                mul(att_block, att_block, __bfloat162float(__float2bfloat16(0.125f)));
                sub_row(att_block, att_block, l_reg_fl);
                exp(att_block, att_block);
                copy(temp_block, att_block);
                copy(att_block_mma, att_block);

                auto (*att_smem)[NUM_WARPGROUPS_BWD_QO][NUM_WARPGROUPS_BWD] = reinterpret_cast<st_bf<tile_h_qo, tile_w, layout_wgmma_itl> (*)[NUM_WARPGROUPS_BWD_QO][NUM_WARPGROUPS_BWD]>(qg_smem); 

                warpgroup::store(att_smem[tic][subtile][warpgroupid], att_block_mma);
                __syncthreads(); 
        
                warpgroup::mma_fence(att_block);
                warpgroup::mm_ABt(att_block, og_smem[tic][subtile], v_smem[warpgroupid]);
                warpgroup::mma_commit_group();

                warpgroup::load(d_reg_bf, d_smem[tic][subtile]);
                copy(d_reg_fl, d_reg_bf);

                warpgroup::mma_fence(vg_reg);
                warpgroup::mma_AtB(vg_reg, att_smem[tic][subtile][warpgroupid], og_smem[tic][subtile]);
                warpgroup::mma_commit_group();

                warpgroup::mma_async_wait<1>();
                sub_row(att_block, att_block, d_reg_fl);
                mul(temp_block, temp_block, att_block);
                mul(temp_block, temp_block, __bfloat162float(__float2bfloat16(0.125f)));
                copy(att_block_mma, temp_block);

                warpgroup::mma_async_wait(); 
                warpgroup::store(att_smem[tic][subtile][warpgroupid], att_block_mma);
                __syncthreads();

                zero(qg_reg);
                warpgroup::mma_fence(qg_reg);
                warpgroup::mma_AB(qg_reg, att_block_mma, k_smem[warpgroupid]);
                warpgroup::mma_commit_group(); 

                warpgroup::mma_fence(kg_reg);
                warpgroup::mma_AtB(kg_reg, att_smem[tic][subtile][warpgroupid], q_smem[tic][subtile]);
                warpgroup::mma_commit_group();
                
                warpgroup::mma_async_wait();
                warpgroup::store(qg_smem[tic][subtile][warpgroupid], qg_reg);
            }

            if (warpid % 4 == 0) {
                int tile_idx = (blockIdx.y * NUM_WARPGROUPS_BWD_QO * qo_blocks) + (qo_idx * NUM_WARPGROUPS_BWD_QO) + warpgroupid; 
                for (int idx = 0; idx < NUM_WARPGROUPS_BWD; idx++) {
                    tma::store_sum_async(tma_qg, (qg_smem[tic][warpgroupid][idx]), tile_idx); 
                }
                tma::store_commit_group();
            }
        }

        warpgroup::store(k_smem[warpgroupid], kg_reg);
        warpgroup::store(v_smem[warpgroupid], vg_reg);
        __syncthreads();

        if (warpid % 4 == 0) {
            int tile_idx = (blockIdx.y * NUM_WARPGROUPS_BWD * KV_BLOCKS * gridDim.x) + (blockIdx.x * NUM_WARPGROUPS_BWD * KV_BLOCKS) + (kv_idx * NUM_WARPGROUPS_BWD) + warpgroupid; 
            tma::store_async(tma_kg, (k_smem[warpgroupid]), tile_idx);
            tma::store_async(tma_vg, (v_smem[warpgroupid]), tile_idx);
            tma::store_commit_group();
        }
    }
    tma::store_async_wait();
}

// #include "harness_h100_bwd.impl" // (comment out when using the code below)

#include "src/common/pyutils/torch_helpers.cuh"
#include <iostream>

void fwd_train_attend_ker_tk(torch::Tensor q, torch::Tensor k, torch::Tensor v, torch::Tensor o, torch::Tensor l) {
    std::cout << "Entered forward attention kernel handler" << std::endl;

    CHECK_INPUT(q);
    CHECK_INPUT(k);
    CHECK_INPUT(v);
    CHECK_INPUT(o);
    CHECK_INPUT(l);

    auto batch = q.size(0);
    auto heads = q.size(1);
    auto threads = NUM_WORKERS * kittens::WARP_THREADS;
    auto n     = q.size(2);
    auto d     = q.size(3);

    TORCH_CHECK(batch == ATTN_B, "Batch size is hard coded - if you change in PyTorch, change in h100_train.cu too");
    TORCH_CHECK(heads == ATTN_H, "Num heads is hard coded - if you change in PyTorch, change in h100_train.cu too");
    TORCH_CHECK(n == ATTN_N, "Num elements is hard coded - if you change in PyTorch, change in h100_train.cu too");
    TORCH_CHECK(d == ATTN_D, "Num elements is hard coded - if you change in PyTorch, change in h100_train.cu too");

    TORCH_CHECK(n % (NUM_WORKERS * kittens::TILE_DIM) == 0, "The number of elements should be divisible the number of workers times the tile dimension");

    // convert to bf16
    c10::BFloat16 *q_ptr = q.data_ptr<c10::BFloat16>();
    c10::BFloat16 *k_ptr = k.data_ptr<c10::BFloat16>();
    c10::BFloat16 *v_ptr = v.data_ptr<c10::BFloat16>();
    c10::BFloat16 *o_ptr = o.data_ptr<c10::BFloat16>();
    c10::BFloat16 *l_ptr = l.data_ptr<c10::BFloat16>();

    bf16* q_bf = reinterpret_cast<bf16*>(q_ptr);
    bf16* k_bf = reinterpret_cast<bf16*>(k_ptr);
    bf16* v_bf = reinterpret_cast<bf16*>(v_ptr);
    bf16* o_bf = reinterpret_cast<bf16*>(o_ptr);
    bf16* l_bf = reinterpret_cast<bf16*>(l_ptr);

    CUtensorMap* tma_q_d = tma::allocate_and_create_tensor_map<kittens::st_bf<qo_height, tile_width, layout_q>,          (ATTN_B*ATTN_H*ATTN_N)/(qo_height * 16)>(q_bf);
    CUtensorMap* tma_k_d = tma::allocate_and_create_tensor_map<kittens::st_bf<kv_height, tile_width, layout_k>,          (ATTN_B*ATTN_H*ATTN_N)/(kv_height * 16)>(k_bf);
    CUtensorMap* tma_v_d = tma::allocate_and_create_tensor_map<kittens::st_bf<kv_height, tile_width, layout_v>,          (ATTN_B*ATTN_H*ATTN_N)/(kv_height * 16)>(v_bf);
    CUtensorMap* tma_o_d = tma::allocate_and_create_tensor_map<kittens::st_bf<qo_height, tile_width, layout_o>,          (ATTN_B*ATTN_H*ATTN_N)/(qo_height * 16)>(o_bf);
    CUtensorMap* tma_l_d = tma::allocate_and_create_tensor_map<kittens::st_bf<qo_height, tile_width, layout_q>::col_vec, (ATTN_B*ATTN_H*ATTN_N)/(qo_height * 16)>(l_bf);

    std::cout << "Check and casts" << std::endl;
    unsigned long mem_size = 227000;
    hipFuncSetAttribute(reinterpret_cast<const void*>(attend_ker_fwd_train), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);

    std::cout << "Launching kernel" << std::endl;

    dim3 grid(n/(NUM_WORKERS*kittens::TILE_DIM), batch*heads, 1);
    attend_ker_fwd_train<<<grid, threads, mem_size>>>(tma_q_d, tma_k_d, tma_v_d, tma_o_d, tma_l_d);

    std::cout << "Kernel launched" << std::endl;
    CHECK_CUDA_ERROR(hipGetLastError());
    std::cout << "Exiting forward train attention kernel handler" << std::endl;
}

void prep_train_attend_ker_tk(torch::Tensor o, torch::Tensor og, torch::Tensor d_vec) { 
    std::cout << "Entered prep train attention kernel handler" << std::endl;

    CHECK_INPUT(o);
    CHECK_INPUT(og);
    CHECK_INPUT(d_vec);

    auto batch = o.size(0);
    auto heads = o.size(1);
    auto n     = o.size(2);
    auto d     = o.size(3);

    auto threads = WORKERS * kittens::WARP_THREADS;

    TORCH_CHECK(batch == ATTN_B, "Batch size is hard coded - if you change in PyTorch, change in h100_train.cu too");
    TORCH_CHECK(heads == ATTN_H, "Num heads is hard coded - if you change in PyTorch, change in h100_train.cu too");
    TORCH_CHECK(n == ATTN_N, "Num elements is hard coded - if you change in PyTorch, change in h100_train.cu too");
    TORCH_CHECK(d == ATTN_D, "Num elements is hard coded - if you change in PyTorch, change in h100_train.cu too");

    TORCH_CHECK(n % (WORKERS * kittens::TILE_DIM * 4) == 0, "The number of elements should be divisible the number of workers times the tile dimension");

    // convert to bf16
    c10::BFloat16 *o_ptr = o.data_ptr<c10::BFloat16>();
    c10::BFloat16 *og_ptr = og.data_ptr<c10::BFloat16>();
    c10::BFloat16 *d_vec_ptr = d_vec.data_ptr<c10::BFloat16>();

    bf16* o_bf = reinterpret_cast<bf16*>(o_ptr);
    bf16* og_bf = reinterpret_cast<bf16*>(og_ptr);
    bf16* d_vec_bf = reinterpret_cast<bf16*>(d_vec_ptr);

    CUtensorMap* tma_o_d_pre  = tma::allocate_and_create_tensor_map<kittens::st_bf<4, 4, layout_nrow>,          (ATTN_B*ATTN_H*ATTN_N)/(4*16)>(o_bf);
    CUtensorMap* tma_d_d_pre  = tma::allocate_and_create_tensor_map<kittens::st_bf<4, 4, layout_nrow>::col_vec, (ATTN_B*ATTN_H*ATTN_N)/(4*16)>(d_vec_bf);
    CUtensorMap* tma_og_d_pre = tma::allocate_and_create_tensor_map<kittens::st_bf<4, 4, layout_nrow>,          (ATTN_B*ATTN_H*ATTN_N)/(4*16)>(og_bf);

    std::cout << "Check and casts" << std::endl;
    unsigned long mem_size = 227000;
    hipFuncSetAttribute(reinterpret_cast<const void*>(attend_ker_prep_train), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);

    std::cout << "Launching kernel" << std::endl;

    dim3 grid_1(n/(WORKERS*kittens::TILE_DIM), batch*heads, 1);
    auto threads_1 = WORKERS * kittens::WARP_THREADS;

    attend_ker_prep_train<<<grid_1, threads_1, mem_size>>>(tma_o_d_pre, tma_d_d_pre, tma_og_d_pre);

    std::cout << "Kernel launched" << std::endl;
    CHECK_CUDA_ERROR(hipGetLastError());
    std::cout << "Exiting prep train attention kernel handler" << std::endl;
}

void bwd_train_attend_ker_tk(torch::Tensor q, torch::Tensor k, torch::Tensor v, torch::Tensor l_vec, torch::Tensor d_vec, torch::Tensor og, torch::Tensor qg, torch::Tensor kg, torch::Tensor vg) {
    std::cout << "Entered backward train attention kernel handler" << std::endl;

    CHECK_INPUT(q);
    CHECK_INPUT(k);
    CHECK_INPUT(v);
    CHECK_INPUT(l_vec);
    CHECK_INPUT(d_vec);
    CHECK_INPUT(og);
    CHECK_INPUT(qg);
    CHECK_INPUT(kg);
    CHECK_INPUT(vg);

    auto batch = q.size(0);
    auto heads = q.size(1);
    auto n     = q.size(2);
    auto d     = q.size(3);

    auto threads = WORKERS_BWD * kittens::WARP_THREADS;

    TORCH_CHECK(batch == ATTN_B, "Batch size is hard coded - if you change in PyTorch, change in h100_train.cu too");
    TORCH_CHECK(heads == ATTN_H, "Num heads is hard coded - if you change in PyTorch, change in h100_train.cu too");
    TORCH_CHECK(n == ATTN_N, "Num elements is hard coded - if you change in PyTorch, change in h100_train.cu too");
    TORCH_CHECK(d == ATTN_D, "Num elements is hard coded - if you change in PyTorch, change in h100_train.cu too");

    TORCH_CHECK(n % (WORKERS_BWD * kittens::TILE_DIM * 4) == 0, "The number of elements should be divisible the number of workers times the tile dimension");

    // convert to bf16
    c10::BFloat16 *q_ptr = q.data_ptr<c10::BFloat16>();
    c10::BFloat16 *k_ptr = k.data_ptr<c10::BFloat16>();
    c10::BFloat16 *v_ptr = v.data_ptr<c10::BFloat16>();
    c10::BFloat16 *l_ptr = l_vec.data_ptr<c10::BFloat16>();
    c10::BFloat16 *d_ptr = d_vec.data_ptr<c10::BFloat16>();
    c10::BFloat16 *og_ptr = og.data_ptr<c10::BFloat16>();
    c10::BFloat16 *qg_ptr = qg.data_ptr<c10::BFloat16>();
    c10::BFloat16 *kg_ptr = kg.data_ptr<c10::BFloat16>();
    c10::BFloat16 *vg_ptr = vg.data_ptr<c10::BFloat16>();

    bf16* q_bf = reinterpret_cast<bf16*>(q_ptr);
    bf16* k_bf = reinterpret_cast<bf16*>(k_ptr);
    bf16* v_bf = reinterpret_cast<bf16*>(v_ptr);
    bf16* l_bf = reinterpret_cast<bf16*>(l_ptr);
    bf16* d_bf = reinterpret_cast<bf16*>(d_ptr);
    bf16* og_bf = reinterpret_cast<bf16*>(og_ptr);
    bf16* qg_bf = reinterpret_cast<bf16*>(qg_ptr);
    bf16* kg_bf = reinterpret_cast<bf16*>(kg_ptr);
    bf16* vg_bf = reinterpret_cast<bf16*>(vg_ptr);

    CUtensorMap* tma_q_d_bwd = tma::allocate_and_create_tensor_map<q_smem_tile, (ATTN_B*ATTN_H*ATTN_N)/(tile_h_qo * 16)>(q_bf);
    CUtensorMap* tma_k_d_bwd = tma::allocate_and_create_tensor_map<k_smem_tile, (ATTN_B*ATTN_H*ATTN_N)/(tile_h * 16)>(k_bf);
    CUtensorMap* tma_v_d_bwd = tma::allocate_and_create_tensor_map<v_smem_tile, (ATTN_B*ATTN_H*ATTN_N)/(tile_h * 16)>(v_bf);

    CUtensorMap* tma_l_d_bwd = tma::allocate_and_create_tensor_map<l_smem_tile, (ATTN_B*ATTN_H*ATTN_N)/(tile_h_qo * 16)>(l_bf);
    CUtensorMap* tma_d_d_bwd = tma::allocate_and_create_tensor_map<d_smem_tile, (ATTN_B*ATTN_H*ATTN_N)/(tile_h_qo * 16)>(d_bf);

    CUtensorMap* tma_og_d_bwd = tma::allocate_and_create_tensor_map<og_smem_tile, (ATTN_B*ATTN_H*ATTN_N)/(tile_h_qo * 16)>(og_bf);
    CUtensorMap* tma_qg_d_bwd = tma::allocate_and_create_tensor_map<qg_smem_tile, (ATTN_B*ATTN_H*ATTN_N)/(tile_h_qo * 16)>(qg_bf);
    CUtensorMap* tma_kg_d_bwd = tma::allocate_and_create_tensor_map<k_smem_tile, (ATTN_B*ATTN_H*ATTN_N)/(tile_h * 16)>(kg_bf);
    CUtensorMap* tma_vg_d_bwd = tma::allocate_and_create_tensor_map<v_smem_tile, (ATTN_B*ATTN_H*ATTN_N)/(tile_h * 16)>(vg_bf);

    std::cout << "Check and casts" << std::endl;
    unsigned long mem_size = 227000;
    hipFuncSetAttribute(reinterpret_cast<const void*>(attend_ker_bwd_train), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);

    std::cout << "Launching kernel" << std::endl;

    dim3 grid_2(n/(KV_BLOCKS*WORKERS_BWD*kittens::TILE_DIM), batch*heads, 1);
    auto threads_2 = WORKERS_BWD * kittens::WARP_THREADS;

    attend_ker_bwd_train<<<grid_2, threads_2, mem_size>>>(tma_q_d_bwd, tma_k_d_bwd, tma_v_d_bwd, tma_l_d_bwd, tma_d_d_bwd, tma_og_d_bwd, tma_qg_d_bwd, tma_kg_d_bwd, tma_vg_d_bwd);

    std::cout << "Kernel launched" << std::endl;
    CHECK_CUDA_ERROR(hipGetLastError());
    std::cout << "Exiting backward train attention kernel handler" << std::endl;
}