#include "hip/hip_runtime.h"


#define KITTENS_HOPPER // we are on an H100
#include "../../src/kittens.cuh"
#include <hip/hip_cooperative_groups.h>

constexpr int NUM_WORKERS = 16;
constexpr int NUM_WARPGROUPS = (NUM_WORKERS/(kittens::WARPGROUP_WARPS));

constexpr int qo_height = 4, kv_height = 4;
constexpr int NUM_WORKERS_KV = 4;
constexpr int tile_width = 64/16;

using namespace kittens;

using layout_q = ducks::st_layout::wgmma_swizzle; // need to make this 128b
using layout_k = ducks::st_layout::wgmma_swizzle; // need to make this 128b
using layout_v = ducks::st_layout::wgmma_interleave; // need to make this 128b
using layout_o = ducks::st_layout::swizzle; 

template<int N> __global__  __launch_bounds__(NUM_WORKERS*kittens::WARP_THREADS, 1)
void attend_ker_fwd_train(CUtensorMap* tma_q, CUtensorMap* tma_k, CUtensorMap* tma_v, CUtensorMap* tma_o, CUtensorMap* tma_l) {
    extern __shared__ int __shm[]; // this is the CUDA shared memory
    tma_swizzle_allocator al((int*)&__shm[0]);

    st_bf<qo_height, tile_width, layout_q>           (&q_smem)   [NUM_WARPGROUPS] = al.allocate<st_bf<qo_height, tile_width, layout_q>,          NUM_WARPGROUPS>();
    st_bf<kv_height, tile_width, layout_k>           (&k_smem)[2][NUM_WORKERS_KV] = al.allocate<st_bf<kv_height, tile_width, layout_k>, 2,       NUM_WORKERS_KV>();
    st_bf<kv_height, tile_width, layout_v>           (&v_smem)[2][NUM_WORKERS_KV] = al.allocate<st_bf<kv_height, tile_width, layout_v>, 2,       NUM_WORKERS_KV>();
    st_bf<qo_height, tile_width, layout_q>::col_vec  (&l_smem)   [NUM_WARPGROUPS] = al.allocate<st_bf<qo_height, tile_width, layout_q>::col_vec, NUM_WARPGROUPS>();

    int tic = 0, toc = 1;
 
    rt_fl<1, kv_height> att_block;
    rt_bf<1, kv_height> att_block_mma;
    rt_fl<1, tile_width> o_prev;
    rt_fl<1, kv_height>::col_vec max_vec_last, max_vec;
    rt_fl<1, kv_height>::col_vec norm_vec_last, norm_vec;

    int warpid      = kittens::warpid();
    int warpgroupid = warpid/kittens::WARPGROUP_WARPS; 

    auto block = cooperative_groups::this_thread_block();

    constexpr int qo_tiles  = N / q_smem[0].rows; 
    constexpr int kv_blocks = N / (NUM_WORKERS_KV*k_smem[0][0].rows);

    __shared__ uint64_t qsmem_barrier, ksmem_barrier, vsmem_barrier;

    int q_phasebit = 0;
    int k_phasebit = 0;
    int v_phasebit = 0; 

    if (threadIdx.x == 0) {
        tma::init_barrier<st_bf<qo_height, tile_width, layout_q>, NUM_WARPGROUPS>(qsmem_barrier, 1);
        tma::init_barrier<st_bf<kv_height, tile_width, layout_k>, NUM_WORKERS_KV>(ksmem_barrier, 1); 
        tma::init_barrier<st_bf<kv_height, tile_width, layout_v>, NUM_WORKERS_KV>(vsmem_barrier, 1);
    }
    __syncthreads();

    if (warpid == 0) {
        for (int wg = 0; wg < NUM_WORKERS/kittens::WARPGROUP_WARPS; wg++) { // load q
            int tile_idx = (blockIdx.y * NUM_WARPGROUPS * gridDim.x) + (blockIdx.x * NUM_WARPGROUPS) + wg;
            tma::load_async((q_smem[wg]), tma_q, qsmem_barrier, tile_idx); 
        }
        for (int w = 0; w < NUM_WORKERS_KV; w++) { // load k, v      
            int tile_idx = (blockIdx.y * NUM_WORKERS_KV * kv_blocks) + (0 * NUM_WORKERS_KV) + w; 
            tma::load_async((k_smem[tic][w]), tma_k, ksmem_barrier, tile_idx); 
            tma::load_async((v_smem[tic][w]), tma_v, vsmem_barrier, tile_idx); 
        }
    }

    neg_infty(max_vec); // zero registers for the Q chunk
    zero(norm_vec);
    zero(o_prev);

    tma::arrive_and_wait(qsmem_barrier, q_phasebit);
    q_phasebit ^= 1;
    __syncthreads();

    warpgroup::mul(q_smem[warpgroupid], q_smem[warpgroupid], __float2bfloat16(0.125f));

    for(auto kv_idx = 0; kv_idx < kv_blocks; kv_idx++, tic ^= 1, toc ^= 1) {

        tma::arrive_and_wait(ksmem_barrier, k_phasebit);
        tma::arrive_and_wait(vsmem_barrier, v_phasebit);
        k_phasebit ^= 1;
        v_phasebit ^= 1;

        if ((threadIdx.x == 0)) {
            tma::set_bytes(ksmem_barrier, NUM_WORKERS_KV * sizeof(bf16) * k_smem[0][0].num_elements);
            tma::set_bytes(vsmem_barrier, NUM_WORKERS_KV * sizeof(bf16) * v_smem[0][0].num_elements);
        }
        __syncthreads();

        if ((kv_idx + 1 < kv_blocks) && (warpid == 0)) {
            for (int w = 0; w < NUM_WORKERS_KV; w++) {        
                int tile_idx = (blockIdx.y * NUM_WORKERS_KV * kv_blocks) + ((kv_idx + 1) * NUM_WORKERS_KV) + w; 
                tma::load_async((k_smem[toc][w]), tma_k, ksmem_barrier, tile_idx); 
                tma::load_async((v_smem[toc][w]), tma_v, vsmem_barrier, tile_idx); 
            }
        }

        for(int subtile = 0; subtile < NUM_WORKERS_KV; subtile++) {
            warpgroup::mma_fence(att_block);
            warpgroup::mm_ABt(att_block, q_smem[warpgroupid], k_smem[tic][subtile]);
            warpgroup::mma_commit_group();

            copy(norm_vec_last, norm_vec);
            copy(max_vec_last,  max_vec);

            warpgroup::mma_async_wait();

            row_max(max_vec, att_block, max_vec); // accumulate onto the max_vec
            sub_row(att_block, att_block, max_vec);
            exp(att_block, att_block);

            sub(max_vec_last, max_vec_last, max_vec);
            exp(max_vec_last, max_vec_last);
            mul(norm_vec, norm_vec, max_vec_last);

            row_sum(norm_vec, att_block, norm_vec); // accumulate onto the norm_vec
            div_row(att_block, att_block, norm_vec);

            mul(norm_vec_last, norm_vec_last, max_vec_last);
            div(norm_vec_last, norm_vec_last, norm_vec);

            copy(att_block_mma, att_block); // convert to bf16 for mma
            mul_row(o_prev, o_prev, norm_vec_last); // normalize o_prev in advance of mma'ing onto it

            warpgroup::mma_fence(o_prev);
            warpgroup::mma_AB(o_prev, att_block_mma, v_smem[tic][subtile]);
            warpgroup::mma_commit_group();
        }
    }

    auto *o_smem = reinterpret_cast<st_bf<qo_height, tile_width, layout_o>*>(&q_smem[0].data[0]); // reuse q memory
    warpgroup::store(o_smem[warpgroupid], o_prev); 
    __syncthreads();
    if (warpid % 4 == 0) { // store o
        int tile_idx = (blockIdx.y * NUM_WARPGROUPS * gridDim.x) + (blockIdx.x * NUM_WARPGROUPS) + warpgroupid; 
        tma::store_async(tma_o, (o_smem[warpgroupid]), tile_idx); 
        tma::store_commit_group(); 
    }

    log(norm_vec, norm_vec);
    add(norm_vec, norm_vec, max_vec);
    __syncthreads();

    warpgroup::store(l_smem[warpgroupid], norm_vec);
    __syncthreads();
    if (warpid % 4 == 0) { // store l
        int tile_idx = (blockIdx.y * NUM_WARPGROUPS * gridDim.x) + (blockIdx.x * NUM_WARPGROUPS) + warpgroupid; 
        tma::store_async(tma_l, (l_smem[warpgroupid]), tile_idx); 
        tma::store_commit_group(); 
    }

    tma::store_async_wait();
}

constexpr int WORKERS = 8;

constexpr int th = 4; 
constexpr int tw = 64/16;

using layout_nrow = ducks::st_layout::swizzle;

template<int N> __global__  __launch_bounds__(WORKERS*kittens::WARP_THREADS, 1)
void attend_ker_prep_train(CUtensorMap* tma_o, CUtensorMap* tma_d, CUtensorMap* tma_o_grad) {
    extern __shared__ int __shm[]; // this is the CUDA shared memory
    tma_swizzle_allocator al((int*)&__shm[0]);

    int warpid = kittens::warpid();

    st_bf<th, tw, layout_nrow>          (&og_smem)[WORKERS] = al.allocate<st_bf<th, tw, layout_nrow>, WORKERS>();
    st_bf<th, tw, layout_nrow>          (&o_smem) [WORKERS] = al.allocate<st_bf<th, tw, layout_nrow>, WORKERS>();
    st_bf<th, tw, layout_nrow>::col_vec (&d_smem) [WORKERS] = al.allocate<st_bf<th, tw, layout_nrow>::col_vec, WORKERS>();

    rt_fl<th, tw> og_reg;
    rt_fl<th, tw> o_reg; 
    rt_fl<th, tw>::col_vec d_reg;

    __shared__ uint64_t ograd_smem_barrier, o_smem_barrier;
    int o_phasebit = 0; 
    int og_phasebit = 0;

    if (threadIdx.x == 0) {
        tma::init_barrier<st_bf<th, tw, layout_o>, WORKERS>(ograd_smem_barrier, 1);
        tma::init_barrier<st_bf<th, tw, layout_o>, WORKERS>(o_smem_barrier, 1);
    }
    __syncthreads();

    if (warpid == 0) {
        for (int w = 0; w < WORKERS; w++) { // load o, o_grad
            int tile_idx = (blockIdx.y * WORKERS * gridDim.x) + (blockIdx.x * WORKERS) + w; 
            tma::load_async((o_smem[w]), tma_o, o_smem_barrier, tile_idx); 
            tma::load_async((og_smem[w]), tma_o_grad, ograd_smem_barrier, tile_idx); 
        }
    }

    tma::arrive_and_wait(ograd_smem_barrier, og_phasebit);
    tma::arrive_and_wait(o_smem_barrier, o_phasebit);

    load(o_reg, o_smem[warpid]);
    load(og_reg, og_smem[warpid]);

    mul(og_reg, og_reg, o_reg);
    row_sum(d_reg, og_reg);
    
    store(d_smem[warpid], d_reg);

    __syncthreads(); 
    if (warpid == 0) {
        for (int w = 0; w < WORKERS; w++) {
            int tile_idx = (blockIdx.y * WORKERS * gridDim.x) + (blockIdx.x * WORKERS) + w; 
            tma::store_async(tma_d, (d_smem[w]), tile_idx); 
        }
        tma::store_commit_group();
    }

    tma::store_async_wait();
}

// check to see if this is correct version for warpgroup style reduction
template<int WORKERS, kittens::ducks::st::all ST, int N_TILES>
__device__ inline void tile_reduce(ST (&dst)[N_TILES]) {
    constexpr int STRIDE = WORKERS*kittens::WARP_THREADS;
    constexpr int RESPONSIBLE_ELEMENTS = (ST::num_elements+STRIDE-1) / STRIDE; // we know in advance this divides evenly.
    float acc[RESPONSIBLE_ELEMENTS];
    #pragma unroll
    for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
        int idx = (threadIdx.x % STRIDE) + j*STRIDE;
        if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) acc[j] = __bfloat162float(dst[0].data[idx]); // start
    }
    // then propagate accumulation through
    for(int i = 1; i < N_TILES; i++) {
        #pragma unroll
        for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
            int idx = (threadIdx.x % STRIDE) + j*STRIDE;
            if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) acc[j] += __bfloat162float(dst[i].data[idx]); // accumulate
        }
    }
    #pragma unroll
    for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
        int idx = (threadIdx.x % STRIDE) + j*STRIDE;
        if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) dst[0].data[idx] = acc[j]; // set
    }
}

constexpr int WORKERS_BWD    = 8; 
constexpr int WORKERS_BWD_QO = 8; 

constexpr int NUM_WARPGROUPS_BWD    = (WORKERS_BWD/(kittens::WARPGROUP_WARPS));
constexpr int NUM_WARPGROUPS_BWD_QO = (WORKERS_BWD_QO/(kittens::WARPGROUP_WARPS));

// static assert that the bigger of the two is a multiple of the smaller
static_assert(NUM_WARPGROUPS_BWD % NUM_WARPGROUPS_BWD_QO == 0 || NUM_WARPGROUPS_BWD_QO % NUM_WARPGROUPS_BWD == 0, "NUM_WARPGROUPS_BWD and NUM_WARPGROUPS_BWD_QO must be multiples of each other");

constexpr int tile_h = 4;    // should be 1
constexpr int tile_h_qo = 4; // should be 8

static_assert(tile_h_qo % 4 == 0, "tile_h_qo must be a multiple of 4");
static_assert(tile_h % 4 == 0, "tile_h must be a multiple of 4");

constexpr int tile_w = 64/16;

using layout_wgmma     = ducks::st_layout::wgmma_swizzle;
using layout_wgmma_itl = ducks::st_layout::wgmma_interleave;
using layout_tma_swi   = ducks::st_layout::swizzle; 

#define k_smem_tile  st_bf<tile_h, tile_w, layout_wgmma_itl>
#define v_smem_tile  st_bf<tile_h, tile_w, layout_wgmma>

#define q_smem_tile  st_bf<tile_h_qo, tile_w, layout_wgmma_itl>
#define og_smem_tile st_bf<tile_h_qo, tile_w, layout_wgmma_itl>
#define qg_smem_tile st_bf<tile_h_qo, tile_w, layout_tma_swi>
#define l_smem_tile  st_bf<tile_h_qo, tile_w, layout_tma_swi>::col_vec
#define d_smem_tile  st_bf<tile_h_qo, tile_w, layout_tma_swi>::col_vec

template<int N> __global__ __launch_bounds__(WORKERS_BWD*kittens::WARP_THREADS, 1)
void attend_ker_bwd_train(CUtensorMap* tma_q, CUtensorMap* tma_k, CUtensorMap* tma_v, 
                            CUtensorMap* tma_l_vec, CUtensorMap* tma_d_vec, 
                            CUtensorMap* tma_og, CUtensorMap* tma_qg, CUtensorMap* tma_kg, CUtensorMap* tma_vg, 
                            const bf16* __restrict__ __l__, const bf16* __restrict__ __d__)
{
    extern __shared__ int __shm[]; // this is the CUDA shared memory
    tma_swizzle_allocator al((int*)&__shm[0]);

    const bf16 *_l  = __l__ + (blockIdx.y * N);
    const bf16 *_d  = __d__ + (blockIdx.y * N);

    k_smem_tile  (&k_smem) [NUM_WARPGROUPS_BWD] = al.allocate<k_smem_tile, NUM_WARPGROUPS_BWD>();
    v_smem_tile  (&v_smem) [NUM_WARPGROUPS_BWD] = al.allocate<v_smem_tile, NUM_WARPGROUPS_BWD>();

    q_smem_tile  (&q_smem) [2][NUM_WARPGROUPS_BWD_QO]                         = al.allocate<q_smem_tile,  2, NUM_WARPGROUPS_BWD_QO>();
    og_smem_tile (&og_smem)[2][NUM_WARPGROUPS_BWD_QO]                         = al.allocate<og_smem_tile, 2, NUM_WARPGROUPS_BWD_QO>();
    qg_smem_tile (&qg_smem)[2][NUM_WARPGROUPS_BWD_QO][NUM_WARPGROUPS_BWD + 1] = al.allocate<qg_smem_tile, 2, NUM_WARPGROUPS_BWD_QO, NUM_WARPGROUPS_BWD + 1>();
    l_smem_tile  (&l_smem) [NUM_WARPGROUPS_BWD_QO]                            = al.allocate<l_smem_tile,  NUM_WARPGROUPS_BWD_QO>();
    d_smem_tile  (&d_smem) [NUM_WARPGROUPS_BWD_QO]                            = al.allocate<d_smem_tile,  NUM_WARPGROUPS_BWD_QO>();

    rt_fl<tile_h/kittens::WARPGROUP_WARPS, tile_w> kg_reg;
    rt_fl<tile_h/kittens::WARPGROUP_WARPS, tile_w> vg_reg;

    rt_fl<tile_h_qo/kittens::WARPGROUP_WARPS, tile_w> qg_reg;

    rt_bf<tile_h_qo/kittens::WARPGROUP_WARPS, tile_h>::col_vec l_reg_bf; 
    rt_bf<tile_h_qo/kittens::WARPGROUP_WARPS, tile_h>::col_vec d_reg_bf;
    rt_fl<tile_h_qo/kittens::WARPGROUP_WARPS, tile_h>::col_vec l_reg_fl; 
    rt_fl<tile_h_qo/kittens::WARPGROUP_WARPS, tile_h>::col_vec d_reg_fl;

    rt_fl<tile_h_qo/kittens::WARPGROUP_WARPS, tile_h> att_block; 
    rt_bf<tile_h_qo/kittens::WARPGROUP_WARPS, tile_h> att_block_mma;
    rt_fl<tile_h_qo/kittens::WARPGROUP_WARPS, tile_h> temp_block;

    int warpid = kittens::warpid();
    int warpgroupid = warpid/kittens::WARPGROUP_WARPS;

    constexpr int qo_blocks = N / (tile_h_qo * kittens::TILE_DIM * NUM_WARPGROUPS_BWD_QO);
    constexpr int kv_blocks = N / (tile_h    * kittens::TILE_DIM * NUM_WARPGROUPS_BWD);

    __shared__ uint64_t kv_b, qo_b, qg_b; 

    int tic = 0, toc = 1;

    int kv_phasebit = 0;
    int qo_phasebit = 0;
    int qg_phasebit = 0;

    if (threadIdx.x == 0) {
        tma::init_barrier<q_smem_tile,  NUM_WARPGROUPS_BWD_QO * 2>(qo_b, 1); // q, og
        tma::init_barrier<qg_smem_tile, NUM_WARPGROUPS_BWD_QO * 1>(qg_b, 1); // qg
        tma::init_barrier<k_smem_tile , NUM_WARPGROUPS_BWD    * 2>(kv_b, 1); // k, v
    } 

    if (warpid == 0) {
        for (int w = 0; w < NUM_WARPGROUPS_BWD_QO; w++) {
            int tile_idx = (blockIdx.y * NUM_WARPGROUPS_BWD_QO * qo_blocks) + (0 * NUM_WARPGROUPS_BWD_QO) + w;

            tma::load_async((q_smem [tic][w]),     tma_q,  qo_b, tile_idx); 
            tma::load_async((og_smem[tic][w]),    tma_og,  qo_b, tile_idx); 
            tma::load_async((qg_smem[tic][w][0]), tma_qg,  qg_b, tile_idx);
        } 
    }

    __syncthreads(); 

    for (int kv_idx = 0; kv_idx < kv_blocks; kv_idx++) {
        
        if (warpid == 0) {
            // load k and v
            for (int w = 0; w < NUM_WARPGROUPS_BWD; w++) {
                int tile_idx = (blockIdx.y * NUM_WARPGROUPS_BWD * kv_blocks) + (kv_idx * NUM_WARPGROUPS_BWD) + w; 
                tma::load_async((k_smem[w]), tma_k, kv_b, tile_idx); 
                tma::load_async((v_smem[w]), tma_v, kv_b, tile_idx); 
            }
        }
        
        tma::arrive_and_wait(kv_b, kv_phasebit);
        kv_phasebit ^= 1;

        if (threadIdx.x == 0) {
            tma::set_bytes(kv_b, NUM_WARPGROUPS_BWD * sizeof(bf16) * k_smem[0].num_elements * 2);
        }

        zero(kg_reg);
        zero(vg_reg);
        __syncthreads(); 

        for (int qo_idx = 0; qo_idx < qo_blocks; qo_idx++) {

            tma::arrive_and_wait(qo_b,  qo_phasebit);
            qo_phasebit ^= 1;

            if (qo_idx + 1 < qo_blocks) {
                if (threadIdx.x == 0) {
                    tma::set_bytes(qo_b, NUM_WARPGROUPS_BWD_QO * sizeof(bf16) * q_smem[0][0].num_elements * 2); 
                }

                if (warpid == 0) {
                    for (int w = 0; w < NUM_WARPGROUPS_BWD_QO; w++) {
                        int tile_idx = (blockIdx.y * NUM_WARPGROUPS_BWD_QO * qo_blocks) + ((qo_idx + 1) * NUM_WARPGROUPS_BWD_QO) + w;
                        tma::load_async((q_smem [toc][w]),     tma_q,  qo_b, tile_idx); 
                        tma::load_async((og_smem[toc][w]),    tma_og,  qo_b, tile_idx);
                    }
                }
            }
            else if (kv_idx + 1 < kv_blocks) {
                if (threadIdx.x == 0) {
                    tma::set_bytes(qo_b, NUM_WARPGROUPS_BWD_QO * sizeof(bf16) * q_smem[0][0].num_elements * 2); 
                }

                if (warpid == 0) {
                    for (int w = 0; w < NUM_WARPGROUPS_BWD_QO; w++) {
                        int tile_idx = (blockIdx.y * NUM_WARPGROUPS_BWD_QO * qo_blocks) + (0 * NUM_WARPGROUPS_BWD_QO) + w;
                        tma::load_async((q_smem [toc][w]),     tma_q,  qo_b, tile_idx); 
                        tma::load_async((og_smem[toc][w]),    tma_og,  qo_b, tile_idx); 
                    }
                }
            }

            if constexpr (NUM_WARPGROUPS_BWD_QO < NUM_WARPGROUPS_BWD) {
                if (warpgroupid < NUM_WARPGROUPS_BWD_QO) {
                    warpgroup::load(l_smem[warpgroupid], _l + (qo_idx * NUM_WARPGROUPS_BWD_QO + warpgroupid) * l_smem[0].length);
                    warpgroup::load(d_smem[warpgroupid], _d + (qo_idx * NUM_WARPGROUPS_BWD_QO + warpgroupid) * d_smem[0].length);
                }
            }
            else if constexpr (NUM_WARPGROUPS_BWD_QO > NUM_WARPGROUPS_BWD) {
                for (int w = warpgroupid; w < NUM_WARPGROUPS_BWD_QO; w += NUM_WARPGROUPS_BWD) {
                    warpgroup::load(l_smem[w], _l + (qo_idx * NUM_WARPGROUPS_BWD_QO + w) * l_smem[0].length);
                    warpgroup::load(d_smem[w], _d + (qo_idx * NUM_WARPGROUPS_BWD_QO + w) * d_smem[0].length);
                }
            }
            else if constexpr (NUM_WARPGROUPS_BWD_QO == NUM_WARPGROUPS_BWD) {
                warpgroup::load(l_smem[warpgroupid], _l + (qo_idx * NUM_WARPGROUPS_BWD_QO + warpgroupid) * l_smem[0].length);
                warpgroup::load(d_smem[warpgroupid], _d + (qo_idx * NUM_WARPGROUPS_BWD_QO + warpgroupid) * d_smem[0].length);
            }
            __syncthreads();

            for (int subtile = 0; subtile < NUM_WARPGROUPS_BWD_QO; subtile++) {
                warpgroup::mma_fence(att_block);
                warpgroup::mm_ABt(att_block, q_smem[tic][subtile], k_smem[warpgroupid]);
                warpgroup::mma_commit_group();

                warpgroup::load(l_reg_bf, l_smem[subtile]);
                copy(l_reg_fl, l_reg_bf);
                
                warpgroup::mma_async_wait();
                mul(att_block, att_block, __float2bfloat16(0.125f));
                sub_row(att_block, att_block, l_reg_fl);
                exp(att_block, att_block);
                copy(temp_block, att_block);
                copy(att_block_mma, att_block);

                auto (*att_smem)[NUM_WARPGROUPS_BWD_QO][NUM_WARPGROUPS_BWD + 1] = reinterpret_cast<st_bf<tile_h_qo, tile_w, layout_wgmma_itl> (*)[NUM_WARPGROUPS_BWD_QO][NUM_WARPGROUPS_BWD + 1]>(qg_smem); 

                __syncthreads(); 
                warpgroup::store(att_smem[tic][subtile][1 + warpgroupid], att_block_mma);
                __syncthreads(); 

                warpgroup::mma_fence(vg_reg);
                warpgroup::mma_AtB(vg_reg, att_smem[tic][subtile][1 + warpgroupid], og_smem[tic][subtile]);
                warpgroup::mma_commit_group();
        
                warpgroup::mma_fence(att_block);
                warpgroup::mm_ABt(att_block, og_smem[tic][subtile], v_smem[warpgroupid]);
                warpgroup::mma_commit_group();

                warpgroup::load(d_reg_bf, d_smem[subtile]);
                copy(d_reg_fl, d_reg_bf);

                warpgroup::mma_async_wait();
                sub_row(att_block, att_block, d_reg_fl);
                mul(temp_block, temp_block, att_block);
                mul(temp_block, temp_block, __float2bfloat16(0.125f));
                copy(att_block_mma, temp_block);

                __syncthreads(); 
                warpgroup::store(att_smem[tic][subtile][1 + warpgroupid], att_block_mma);
                __syncthreads(); 

                warpgroup::mma_fence(qg_reg);
                warpgroup::mm_AB(qg_reg, att_smem[tic][subtile][1 + warpgroupid], k_smem[warpgroupid]);
                warpgroup::mma_commit_group();

                warpgroup::mma_fence(kg_reg);
                warpgroup::mma_AtB(kg_reg, att_smem[tic][subtile][1 + warpgroupid], q_smem[tic][subtile]);
                warpgroup::mma_commit_group();
                warpgroup::mma_async_wait();
                warpgroup::store(qg_smem[tic][subtile][1 + warpgroupid], qg_reg);
            }

            tma::arrive_and_wait(qg_b, qg_phasebit);
            qg_phasebit ^= 1;

            if (qo_idx + 1 < qo_blocks) {
                if (threadIdx.x == 0) {
                    tma::set_bytes(qg_b, NUM_WARPGROUPS_BWD_QO * sizeof(bf16) * qg_smem[0][0][0].num_elements * 1); 
                }

                if (warpid == 0) {
                    for (int w = 0; w < NUM_WARPGROUPS_BWD_QO; w++) {
                        int tile_idx = (blockIdx.y * NUM_WARPGROUPS_BWD_QO * qo_blocks) + ((qo_idx + 1) * NUM_WARPGROUPS_BWD_QO) + w;
                        tma::load_async((qg_smem[toc][w][0]), tma_qg, qg_b, tile_idx);
                    }
                }
            }
            else if (kv_idx + 1 < kv_blocks) {
                if (threadIdx.x == 0) {
                    tma::set_bytes(qg_b, NUM_WARPGROUPS_BWD_QO * sizeof(bf16) * qg_smem[0][0][0].num_elements * 1); 
                }

                if (warpid == 0) {
                    for (int w = 0; w < NUM_WARPGROUPS_BWD_QO; w++) {
                        int tile_idx = (blockIdx.y * NUM_WARPGROUPS_BWD_QO * qo_blocks) + (0 * NUM_WARPGROUPS_BWD_QO) + w;
                        tma::load_async((qg_smem[toc][w][0]), tma_qg, qg_b, tile_idx);
                    }
                }
            }

            __syncthreads();
            if constexpr (NUM_WARPGROUPS_BWD_QO < NUM_WARPGROUPS_BWD) {
                if (warpgroupid < NUM_WARPGROUPS_BWD_QO) {
                    tile_reduce<kittens::WARPGROUP_WARPS, qg_smem_tile, NUM_WARPGROUPS_BWD + 1>(qg_smem[tic][warpgroupid]);
                }
            }
            else if constexpr (NUM_WARPGROUPS_BWD_QO > NUM_WARPGROUPS_BWD) {
                for (int w = warpgroupid; w < NUM_WARPGROUPS_BWD_QO; w += NUM_WARPGROUPS_BWD) {
                    tile_reduce<kittens::WARPGROUP_WARPS, qg_smem_tile, NUM_WARPGROUPS_BWD + 1>(qg_smem[tic][w]);
                }
            }
            else if constexpr (NUM_WARPGROUPS_BWD_QO == NUM_WARPGROUPS_BWD) {
                tile_reduce<kittens::WARPGROUP_WARPS, qg_smem_tile, NUM_WARPGROUPS_BWD + 1>(qg_smem[tic][warpgroupid]);
            }
            __syncthreads();

            if (warpid == 0) {
                for (int w = 0; w < NUM_WARPGROUPS_BWD_QO; w++) {
                    int tile_idx = (blockIdx.y * NUM_WARPGROUPS_BWD_QO * qo_blocks) + (qo_idx * NUM_WARPGROUPS_BWD_QO) + w; 
                    tma::store_async(tma_qg, (qg_smem[tic][w][0]), tile_idx);
                }
                tma::store_commit_group();
            }
            tma::store_async_wait();

            tic ^= 1;
            toc ^= 1;
        }

        warpgroup::store(k_smem[warpgroupid], kg_reg);
        warpgroup::store(v_smem[warpgroupid], vg_reg);
        __syncthreads();

        if (warpid % 4 == 0) {
            int tile_idx = (blockIdx.y * NUM_WARPGROUPS_BWD * kv_blocks) + (kv_idx * NUM_WARPGROUPS_BWD) + warpgroupid; 
            tma::store_async(tma_kg, (k_smem[warpgroupid]), tile_idx);
            tma::store_async(tma_vg, (v_smem[warpgroupid]), tile_idx);
            tma::store_commit_group();
        }
    }
    tma::store_async_wait();
}

#include "harness_h100_bwd_ideal.impl"