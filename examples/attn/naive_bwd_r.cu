#include "hip/hip_runtime.h"


#define KITTENS_HOPPER // we are on an H100
#include "../../src/kittens.cuh"
#include <hip/hip_cooperative_groups.h>

constexpr int NUM_WORKERS = 16;
constexpr int NUM_WARPGROUPS = (NUM_WORKERS/(kittens::WARPGROUP_WARPS));

constexpr int qo_height = 4, kv_height = 4;
constexpr int NUM_WORKERS_KV = 4;
constexpr int tile_width = 64/16;

using namespace kittens;

using layout_q = ducks::st_layout::wgmma_0b; // need to make this 128b
using layout_k = ducks::st_layout::wgmma_0b; // need to make this 128b
using layout_v = ducks::st_layout::wgmma_0b; // need to make this 128b
using layout_o = ducks::st_layout::xor_swizzle; 

template<int N> __global__  __launch_bounds__(NUM_WORKERS*kittens::WARP_THREADS, 1)
void attend_ker_fwd_train(CUtensorMap* tma_q, CUtensorMap* tma_k, CUtensorMap* tma_v, CUtensorMap* tma_o, CUtensorMap* tma_l) {
    extern __shared__ int __shm[]; // this is the CUDA shared memory
    tma_swizzle_allocator al((int*)&__shm[0]);

    st_bf<qo_height, tile_width, layout_q>           (&q_smem)   [NUM_WARPGROUPS] = al.allocate<st_bf<qo_height, tile_width, layout_q>,          NUM_WARPGROUPS>();
    st_bf<kv_height, tile_width, layout_k>           (&k_smem)[2][NUM_WORKERS_KV] = al.allocate<st_bf<kv_height, tile_width, layout_k>, 2,       NUM_WORKERS_KV>();
    st_bf<kv_height, tile_width, layout_v>           (&v_smem)[2][NUM_WORKERS_KV] = al.allocate<st_bf<kv_height, tile_width, layout_v>, 2,       NUM_WORKERS_KV>();
    st_bf<qo_height, tile_width, layout_q>::col_vec  (&l_smem)   [NUM_WARPGROUPS] = al.allocate<st_bf<qo_height, tile_width, layout_q>::col_vec, NUM_WARPGROUPS>();

    int tic = 0, toc = 1;
 
    rt_fl<1, kv_height> att_block;
    rt_bf<1, kv_height> att_block_mma;
    rt_fl<1, tile_width> o_prev;
    rt_fl<1, kv_height>::col_vec max_vec_last, max_vec;
    rt_fl<1, kv_height>::col_vec norm_vec_last, norm_vec;

    int warpid      = kittens::warpid();
    int warpgroupid = warpid/kittens::WARPGROUP_WARPS; 

    auto block = cooperative_groups::this_thread_block();

    constexpr int qo_tiles  = N / q_smem[0].rows; 
    constexpr int kv_blocks = N / (NUM_WORKERS_KV*k_smem[0][0].rows);

    __shared__ uint64_t qsmem_barrier, ksmem_barrier, vsmem_barrier;

    int q_phasebit = 0;
    int k_phasebit = 0;
    int v_phasebit = 0; 

    if (threadIdx.x == 0) {
        tma::init_barrier<st_bf<qo_height, tile_width, layout_q>, NUM_WARPGROUPS>(qsmem_barrier, 1);
        tma::init_barrier<st_bf<kv_height, tile_width, layout_k>, NUM_WORKERS_KV>(ksmem_barrier, 1); 
        tma::init_barrier<st_bf<kv_height, tile_width, layout_v>, NUM_WORKERS_KV>(vsmem_barrier, 1);
    }
    __syncthreads();

    if (warpid == 0) {
        for (int wg = 0; wg < NUM_WORKERS/kittens::WARPGROUP_WARPS; wg++) { // load q
            int tile_idx = (blockIdx.y * NUM_WARPGROUPS * gridDim.x) + (blockIdx.x * NUM_WARPGROUPS) + wg;
            tma::load_async((q_smem[wg]), tma_q, qsmem_barrier, tile_idx); 
        }
        for (int w = 0; w < NUM_WORKERS_KV; w++) { // load k, v      
            int tile_idx = (blockIdx.y * NUM_WORKERS_KV * kv_blocks) + (0 * NUM_WORKERS_KV) + w; 
            tma::load_async((k_smem[tic][w]), tma_k, ksmem_barrier, tile_idx); 
            tma::load_async((v_smem[tic][w]), tma_v, vsmem_barrier, tile_idx); 
        }
    }

    neg_infty(max_vec); // zero registers for the Q chunk
    zero(norm_vec);
    zero(o_prev);

    tma::arrive_and_wait(qsmem_barrier, q_phasebit);
    q_phasebit ^= 1;
    __syncthreads();

    warpgroup::mul(q_smem[warpgroupid], q_smem[warpgroupid], __float2bfloat16(0.125f));

    for(auto kv_idx = 0; kv_idx < kv_blocks; kv_idx++, tic ^= 1, toc ^= 1) {

        tma::arrive_and_wait(ksmem_barrier, k_phasebit);
        tma::arrive_and_wait(vsmem_barrier, v_phasebit);
        k_phasebit ^= 1;
        v_phasebit ^= 1;

        if ((threadIdx.x == 0)) {
            tma::set_bytes(ksmem_barrier, NUM_WORKERS_KV * sizeof(bf16) * k_smem[0][0].num_elements);
            tma::set_bytes(vsmem_barrier, NUM_WORKERS_KV * sizeof(bf16) * v_smem[0][0].num_elements);
        }
        __syncthreads();

        if ((kv_idx + 1 < kv_blocks) && (warpid == 0)) {
            for (int w = 0; w < NUM_WORKERS_KV; w++) {        
                int tile_idx = (blockIdx.y * NUM_WORKERS_KV * kv_blocks) + ((kv_idx + 1) * NUM_WORKERS_KV) + w; 
                tma::load_async((k_smem[toc][w]), tma_k, ksmem_barrier, tile_idx); 
                tma::load_async((v_smem[toc][w]), tma_v, vsmem_barrier, tile_idx); 
            }
        }

        for(int subtile = 0; subtile < NUM_WORKERS_KV; subtile++) {
            warpgroup::mma_fence(att_block);
            warpgroup::mm_ABt(att_block, q_smem[warpgroupid], k_smem[tic][subtile]);
            warpgroup::mma_commit_group();

            copy(norm_vec_last, norm_vec);
            copy(max_vec_last,  max_vec);

            warpgroup::mma_async_wait();

            row_max(max_vec, att_block, max_vec); // accumulate onto the max_vec
            sub_row(att_block, att_block, max_vec);
            exp(att_block, att_block);

            sub(max_vec_last, max_vec_last, max_vec);
            exp(max_vec_last, max_vec_last);
            mul(norm_vec, norm_vec, max_vec_last);

            row_sum(norm_vec, att_block, norm_vec); // accumulate onto the norm_vec
            div_row(att_block, att_block, norm_vec);

            mul(norm_vec_last, norm_vec_last, max_vec_last);
            div(norm_vec_last, norm_vec_last, norm_vec);

            copy(att_block_mma, att_block); // convert to bf16 for mma
            mul_row(o_prev, o_prev, norm_vec_last); // normalize o_prev in advance of mma'ing onto it

            warpgroup::mma_fence(o_prev);
            warpgroup::mma_AB(o_prev, att_block_mma, v_smem[tic][subtile]);
            warpgroup::mma_commit_group();
        }
    }

    auto *o_smem = reinterpret_cast<st_bf<qo_height, tile_width, layout_o>*>(&q_smem[0].data[0]); // reuse q memory
    warpgroup::store(o_smem[warpgroupid], o_prev); 
    __syncthreads();
    if (warpid % 4 == 0) { // store o
        int tile_idx = (blockIdx.y * NUM_WARPGROUPS * gridDim.x) + (blockIdx.x * NUM_WARPGROUPS) + warpgroupid; 
        tma::store_async(tma_o, (o_smem[warpgroupid]), tile_idx); 
        tma::store_commit_group(); 
    }

    log(norm_vec, norm_vec);
    add(norm_vec, norm_vec, max_vec);
    __syncthreads();

    warpgroup::store(l_smem[warpgroupid], norm_vec);
    __syncthreads();
    if (warpid % 4 == 0) { // store l
        int tile_idx = (blockIdx.y * NUM_WARPGROUPS * gridDim.x) + (blockIdx.x * NUM_WARPGROUPS) + warpgroupid; 
        tma::store_async(tma_l, (l_smem[warpgroupid]), tile_idx); 
        tma::store_commit_group(); 
    }

    tma::store_async_wait();
}

constexpr int WORKERS = 8;

constexpr int th = 4; 
constexpr int tw = 64/16;

using layout_nrow = ducks::st_layout::xor_swizzle;

template<int N> __global__  __launch_bounds__(WORKERS*kittens::WARP_THREADS, 1)
void attend_ker_prep_train(CUtensorMap* tma_o, CUtensorMap* tma_d, CUtensorMap* tma_o_grad) {
    extern __shared__ int __shm[]; // this is the CUDA shared memory
    tma_swizzle_allocator al((int*)&__shm[0]);

    int warpid = kittens::warpid();

    st_bf<th, tw, layout_nrow>          (&og_smem)[WORKERS] = al.allocate<st_bf<th, tw, layout_nrow>, WORKERS>();
    st_bf<th, tw, layout_nrow>          (&o_smem) [WORKERS] = al.allocate<st_bf<th, tw, layout_nrow>, WORKERS>();
    st_bf<th, tw, layout_nrow>::col_vec (&d_smem) [WORKERS] = al.allocate<st_bf<th, tw, layout_nrow>::col_vec, WORKERS>();

    rt_fl<th, tw> og_reg;
    rt_fl<th, tw> o_reg; 
    rt_fl<th, tw>::col_vec d_reg;

    __shared__ uint64_t ograd_smem_barrier, o_smem_barrier;
    int o_phasebit = 0; 
    int og_phasebit = 0;

    if (threadIdx.x == 0) {
        tma::init_barrier<st_bf<th, tw, layout_o>, WORKERS>(ograd_smem_barrier, 1);
        tma::init_barrier<st_bf<th, tw, layout_o>, WORKERS>(o_smem_barrier, 1);
    }
    __syncthreads();

    if (warpid == 0) {
        for (int w = 0; w < WORKERS; w++) { // load o, o_grad
            int tile_idx = (blockIdx.y * WORKERS * blockDim.x) + (blockIdx.x * WORKERS) + w; 
            tma::load_async((o_smem[w]), tma_o, o_smem_barrier, tile_idx); 
            tma::load_async((og_smem[w]), tma_o_grad, ograd_smem_barrier, tile_idx); 
        }
    }

    tma::arrive_and_wait(ograd_smem_barrier, og_phasebit);
    tma::arrive_and_wait(o_smem_barrier, o_phasebit);

    load(o_reg, o_smem[warpid]);
    load(og_reg, og_smem[warpid]);

    mul(og_reg, og_reg, o_reg);
    row_sum(d_reg, og_reg);
    
    store(d_smem[warpid], d_reg);

    __syncthreads(); 
    if (warpid == 0) {
        for (int w = 0; w < WORKERS; w++) {
            int tile_idx = (blockIdx.y * WORKERS * blockDim.x) + (blockIdx.x * WORKERS) + w; 
            tma::store_async(tma_d, (d_smem[w]), tile_idx); 
        }
        tma::store_commit_group();
    }

    tma::store_async_wait();
}

template<int WORKERS, kittens::ducks::st::all ST, int N_TILES>
__device__ inline void tile_reduce(ST (&dst)[N_TILES]) {
    constexpr int STRIDE = WORKERS*kittens::WARP_THREADS;
    constexpr int RESPONSIBLE_ELEMENTS = (ST::num_elements+STRIDE-1) / STRIDE; // we know in advance this divides evenly.
    float acc[RESPONSIBLE_ELEMENTS];
    #pragma unroll
    for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
        int idx = kittens::laneid() + j*STRIDE;
        if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) acc[j] = __bfloat162float(dst[0].data[idx]); // start
    }
    // then propagate accumulation through
    for(int i = 1; i < N_TILES; i++) {
        #pragma unroll
        for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
            int idx = kittens::laneid() + j*STRIDE;
            if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) acc[j] += __bfloat162float(dst[i].data[idx]); // accumulate
        }
    }
    #pragma unroll
    for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
        int idx = kittens::laneid() + j*STRIDE;
        if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) dst[0].data[idx] = acc[j]; // set
    }
}

constexpr int WORKERS_BWD = 4; 

constexpr int tile_h = 1;
constexpr int tile_w = 64/16;

using layout_nrow      = ducks::st_layout::xor_swizzle;
using layout_wgmma     = ducks::st_layout::wgmma_0b;
using layout_tma_swi   = ducks::st_layout::xor_swizzle; 

#define q_smem_tile      st_bf<tile_h, tile_w, layout_tma_swi>
#define og_smem_tile     st_bf<tile_h, tile_w, layout_tma_swi>
#define l_smem_tile      st_bf<tile_h, tile_w, layout_tma_swi>::col_vec
#define d_smem_tile      st_bf<tile_h, tile_w, layout_tma_swi>::col_vec
#define qg_smem_tile     st_bf<tile_h, tile_w, layout_tma_swi>
#define kg_pre_smem_tile st_bf<tile_h, tile_w, layout_tma_swi>
#define vg_pre_smem_tile st_bf<tile_h, tile_w, layout_tma_swi>

#define k_smem_tile  st_bf<tile_h, tile_w, layout_tma_swi>
#define v_smem_tile  st_bf<tile_h, tile_w, layout_tma_swi>

#define scratch_pad  st_bf<tile_h, tile_h, layout_tma_swi>

template<int N> __global__ __launch_bounds__(WORKERS_BWD*kittens::WARP_THREADS, 1)
void attend_ker_bwd_train(const bf16* __restrict__ __q__, const bf16* __restrict__ __k__, const bf16* __restrict__ __v__, 
                          const bf16* __restrict__ __l__, const bf16* __restrict__ __d__, 
                          const bf16* __restrict__ __og__, bf16* __qg__, bf16* __kg_pre__, bf16* __vg_pre__)
{
    extern __shared__ int __shm[]; // this is the CUDA shared memory
    
    // shared_allocator al((int*)&__shm[0]);
    shared_allocator al((int*)&__shm[0]);

    constexpr int qo_blocks = N / (tile_h * kittens::TILE_DIM * WORKERS_BWD);
    constexpr int kv_blocks = N / (tile_h * kittens::TILE_DIM * WORKERS_BWD);

    const bf16 *_q  = __q__  + (blockIdx.y * N * 64); 
    const bf16 *_k  = __k__  + (blockIdx.y * N * 64);
    const bf16 *_v  = __v__  + (blockIdx.y * N * 64);

    const bf16 *_l  = __l__  + (blockIdx.y * N);
    const bf16 *_d  = __d__  + (blockIdx.y * N);

    const bf16 *_og = __og__ + (blockIdx.y * N * 64);

    bf16 *_qg     = __qg__     + (blockIdx.y * N * 64);
    bf16 *_kg_pre = __kg_pre__ + (blockIdx.y * N * 64 * gridDim.x);
    bf16 *_vg_pre = __vg_pre__ + (blockIdx.y * N * 64 * gridDim.x);

    int warpid = kittens::warpid();

    q_smem_tile  (&kg_smem) [WORKERS_BWD][WORKERS_BWD] = al.allocate<kg_pre_smem_tile, WORKERS_BWD, WORKERS_BWD>();
    og_smem_tile (&vg_smem) [WORKERS_BWD][WORKERS_BWD] = al.allocate<vg_pre_smem_tile, WORKERS_BWD, WORKERS_BWD>();

    l_smem_tile  (&l_smem)  [WORKERS_BWD] = al.allocate<l_smem_tile, WORKERS_BWD>();
    d_smem_tile  (&d_smem)  [WORKERS_BWD] = al.allocate<d_smem_tile, WORKERS_BWD>();

    k_smem_tile  (&k_smem)  [WORKERS_BWD] = al.allocate<k_smem_tile, WORKERS_BWD>();
    v_smem_tile  (&v_smem)  [WORKERS_BWD] = al.allocate<v_smem_tile, WORKERS_BWD>();

    rt_bf<tile_h, tile_w> q_reg;
    rt_bf<tile_h, tile_w> og_reg; 
    rt_bf<tile_h, tile_w, ducks::rt_layout::col> og_reg_col; 

    rt_bf<tile_h, tile_w> k_reg;
    rt_bf<tile_h, tile_w> v_reg;

    rt_bf<tile_h, tile_w>::col_vec l_reg_bf;
    rt_fl<tile_h, tile_w>::col_vec l_reg_fl;

    rt_bf<tile_h, tile_w>::col_vec d_reg_bf; 
    rt_fl<tile_h, tile_w>::col_vec d_reg_fl;

    rt_fl<tile_h, tile_w> qg_reg;
    rt_fl<tile_h, tile_w> kg_pre_reg;
    rt_fl<tile_h, tile_w> vg_pre_reg;

    rt_fl<tile_h, tile_h> att_block; 
    rt_bf<tile_h, tile_h> att_block_mma;
    rt_fl<tile_h, tile_h> temp_block;

    load(q_reg, _q + (blockIdx.x * WORKERS_BWD + warpid) * q_reg.num_elements, q_reg.cols);
    load(og_reg, _og + (blockIdx.x * WORKERS_BWD + warpid) * og_reg.num_elements, og_reg.cols);
    load(l_reg_bf, _l + (blockIdx.x * WORKERS_BWD + warpid) * l_smem[0].length);
    load(d_reg_bf, _d + (blockIdx.x * WORKERS_BWD + warpid) * d_smem[0].length);
    __syncthreads(); 

    zero(qg_reg);
    copy(l_reg_fl, l_reg_bf);
    copy(d_reg_fl, d_reg_bf);
    swap_layout(og_reg_col, og_reg);

    for (int kv_idx = 0; kv_idx < kv_blocks; kv_idx++) {
        load(k_smem[warpid], _k + (kv_idx * WORKERS_BWD + warpid) * k_smem[0].num_elements, k_smem[0].cols);
        load(v_smem[warpid], _v + (kv_idx * WORKERS_BWD + warpid) * v_smem[0].num_elements, v_smem[0].cols);
        __syncthreads();

        for (int subtile = 0; subtile < WORKERS_BWD; subtile++) {
            load(k_reg, k_smem[subtile]);
            mul(k_reg, k_reg, __float2bfloat16(0.125f));

            zero(att_block);
            mma_ABt(att_block, q_reg, k_reg, att_block);

            sub_row(att_block, att_block, l_reg_fl);
            exp(att_block, att_block);
            copy(temp_block, att_block);
            copy(att_block_mma, att_block);

            transpose_inplace(att_block_mma); 
            zero(vg_pre_reg);
            mma_AB(vg_pre_reg, att_block_mma, og_reg_col, vg_pre_reg);
            store(vg_smem[subtile][warpid], vg_pre_reg); 

            load(v_reg, v_smem[subtile]);
            zero(att_block);
            mma_ABt(att_block, og_reg, v_reg, att_block);

            sub_row(att_block, att_block, d_reg_fl); 
            mul(temp_block, temp_block, att_block);
            copy(att_block_mma, temp_block);

            rt_bf<tile_h, tile_w, ducks::rt_layout::col> &k_reg_col = swap_layout_inplace(k_reg);
            mma_AB(qg_reg, att_block_mma, k_reg_col, qg_reg);

            transpose_inplace(att_block_mma);
            copy(v_reg, q_reg); 
            rt_bf<tile_h, tile_w, ducks::rt_layout::col> &q_reg_col = swap_layout_inplace(v_reg); 
            zero(kg_pre_reg);
            mma_AB(kg_pre_reg, att_block_mma, q_reg_col, kg_pre_reg);
            store(kg_smem[subtile][warpid], kg_pre_reg);
        }

        __syncthreads(); 
        tile_reduce<1, vg_pre_smem_tile, WORKERS_BWD>(vg_smem[warpid]);
        tile_reduce<1, kg_pre_smem_tile, WORKERS_BWD>(kg_smem[warpid]);

        // output tensor of shape
        store(_vg_pre + (blockIdx.x * kv_blocks * WORKERS_BWD * vg_pre_reg.num_elements) + (kv_idx * WORKERS_BWD * vg_pre_reg.num_elements) + (warpid * vg_pre_reg.num_elements), vg_smem[warpid][0], vg_smem[0][0].cols);
        store(_kg_pre + (blockIdx.x * kv_blocks * WORKERS_BWD * kg_pre_reg.num_elements) + (kv_idx * WORKERS_BWD * kg_pre_reg.num_elements) + (warpid * kg_pre_reg.num_elements), kg_smem[warpid][0], kg_smem[0][0].cols);
    }

    store(_qg + (blockIdx.x * WORKERS_BWD + warpid) * qg_reg.num_elements, qg_reg, qg_reg.cols);
    
}

#include "harness_naive_bwd_r.impl"