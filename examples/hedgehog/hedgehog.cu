#include "hip/hip_runtime.h"
# include "src/kittens.cuh"
#include <cuda/pipeline>

#define NUM_WORKERS (16) // hardcoded, don't change
#define NUM_THREADS (NUM_WORKERS*kittens::WARP_THREADS)
#define D_QK (256) // hardcoded, don't change
#define D_VO (64) // hardcoded but can be changed with some effort

using namespace kittens;

using layout = kittens::ducks::st_layout::xor_swizzle;

// sum of an array of tiles -- in fp32 to preserve maximal accuracy
template<int WORKERS, kittens::ducks::st::all ST, int N_TILES>
__device__ inline void tile_reduce(ST &dst, const ST (&src)[N_TILES]) {
    constexpr int STRIDE = WORKERS*kittens::WARP_THREADS;
    constexpr int RESPONSIBLE_ELEMENTS = (ST::num_elements+STRIDE-1) / STRIDE; // we know in advance this divides evenly.
    float acc[RESPONSIBLE_ELEMENTS];
    #pragma unroll
    for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
        int idx = threadIdx.x + j*STRIDE;
        if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) acc[j] = __bfloat162float(dst.data[idx]); // start
    }
    // then propagate accumulation through
    for(int i = 0; i < N_TILES; i++) {
        #pragma unroll
        for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
            int idx = threadIdx.x + j*STRIDE;
            if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) acc[j] += __bfloat162float(src[i].data[idx]); // accumulate
        }
    }
    #pragma unroll
    for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
        int idx = threadIdx.x + j*STRIDE;
        if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) dst.data[idx] = acc[j]; // set
    }
}
// alternatively, sum onto the FIRST tile -- needed by attention.
template<int WORKERS, kittens::ducks::st::all ST, int N_TILES>
__device__ inline void tile_reduce(ST (&dst)[N_TILES]) {
    constexpr int STRIDE = WORKERS*kittens::WARP_THREADS;
    constexpr int RESPONSIBLE_ELEMENTS = (ST::num_elements+STRIDE-1) / STRIDE; // we know in advance this divides evenly.
    float acc[RESPONSIBLE_ELEMENTS];
    #pragma unroll
    for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
        int idx = threadIdx.x + j*STRIDE;
        if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) acc[j] = __bfloat162float(dst[0].data[idx]); // start
    }
    // then propagate accumulation through
    for(int i = 1; i < N_TILES; i++) {
        #pragma unroll
        for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
            int idx = threadIdx.x + j*STRIDE;
            if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) acc[j] += __bfloat162float(dst[i].data[idx]); // accumulate
        }
    }
    #pragma unroll
    for(int j = 0; j < RESPONSIBLE_ELEMENTS; j++) {
        int idx = threadIdx.x + j*STRIDE;
        if(ST::num_elements%STRIDE == 0 || idx < ST::num_elements) dst[0].data[idx] = acc[j]; // set
    }
}

__global__ __launch_bounds__(NUM_THREADS, 1)
void hedgehog(int n, const bf16* __q, const bf16* __k, const bf16* __v, bf16* __o) {

    using G = kittens::group<NUM_WORKERS>;

    auto warpid = kittens::warpid();
    auto lane   = kittens::laneid();

    const bf16 *q_g   = reinterpret_cast<const bf16*>(__q)+blockIdx.x*(n*D_QK);
    const bf16 *k_g   = reinterpret_cast<const bf16*>(__k)+blockIdx.x*(n*D_QK);
    const bf16 *v_g   = reinterpret_cast<const bf16*>(__v)+blockIdx.x*(n*D_VO);
          bf16 *o_g   = reinterpret_cast<bf16*>      (__o)+blockIdx.x*(n*D_VO);

    extern __shared__ alignment_dummy __shm[];
    shared_allocator al((int*)&__shm[0]);
    using QK_BLOCK = st_bf_1x1<layout>;
    using VO_BLOCK = st_bf_1x4<layout>;
    QK_BLOCK (&q_s)[2][NUM_WORKERS] = al.allocate<QK_BLOCK, 2, NUM_WORKERS>(); // 2 * 8192 bytes -- 16x256
    QK_BLOCK (&k_s)[2][NUM_WORKERS] = al.allocate<QK_BLOCK, 2, NUM_WORKERS>(); // 2 * 8192 bytes -- 16x256
    VO_BLOCK (&v_s)[2]              = al.allocate<VO_BLOCK, 2>(); // 2 * 2048 bytes
    VO_BLOCK (&o_s)[2]              = al.allocate<VO_BLOCK, 2>(); // 2 * 2048 bytes

    // att_accumulate is not actually a QK block, even if it happens to be the same type here.
    st_bf_1x1<layout> (&att_accumulate)[NUM_WORKERS] = al.allocate<st_bf_1x1<layout>, NUM_WORKERS>(); // 8192 bytes -- 16x256
    VO_BLOCK          (&kv_accumulate) [NUM_WORKERS] = al.allocate<VO_BLOCK,          NUM_WORKERS>(); // 32768 bytes -- 16x(16x64)

    rt_fl_1x4 kv_state; // kv state gets propagated through here, split among all 16 workers.

    zero(kv_state); // everyone zeroes their part of the kv state.

    __shared__ cuda::barrier<cuda::thread_scope::thread_scope_block> qkv_barrier;
    if (threadIdx.x == 0) {init(&qkv_barrier, NUM_THREADS);}
    __syncthreads();
    load_async(q_s[0][warpid], q_g + warpid*QK_BLOCK::cols, D_QK, qkv_barrier);
    load_async(k_s[0][warpid], k_g + warpid*QK_BLOCK::cols, D_QK, qkv_barrier);
    G::load_async(v_s[0],      v_g, D_VO, qkv_barrier); // just collaboratively load v

    int n_blocks = n / kittens::TILE_DIM;

    int tic = 0, toc = 1;
    for(int block = 0; block < n_blocks; block++, tic^=1, toc^=1) {
        rt_bf_1x1<> q, k, local_attn_bf;
        rt_fl_1x1<> local_attn;
        rt_bf_1x4<> v;
        rt_fl_1x4<> o;

        // load new q, k, v into shared memory and zero o -- collaboratively, across the whole group
        // (the reason to do this is to fill larger transactions.)
        qkv_barrier.arrive_and_wait();
        if(block+1 < n_blocks) {
            load_async(q_s[toc][warpid], q_g + (block+1)*NUM_WORKERS*QK_BLOCK::num_elements + warpid*QK_BLOCK::cols, D_QK, qkv_barrier);
            load_async(k_s[toc][warpid], k_g + (block+1)*NUM_WORKERS*QK_BLOCK::num_elements + warpid*QK_BLOCK::cols, D_QK, qkv_barrier);
            G::load_async(v_s[toc],      v_g + (block+1)*VO_BLOCK::num_elements, D_VO, qkv_barrier); // just collaboratively load v
        }

        load(q, q_s[tic][warpid]);
        load(k, k_s[tic][warpid]);
        zero(local_attn);
        dot(local_attn, q, k, local_attn);
        store(att_accumulate[warpid], local_attn);
        // sum up local attention
        __syncthreads();
        tile_reduce<NUM_WORKERS>(att_accumulate); // now sum is in the first element.
        __syncthreads();
        load(v, v_s[tic]); // everyone needs v
        auto &v_col = swap_layout_inplace(v); // prepare for MMA
        if(warpid == 0) {
            load(local_attn_bf, att_accumulate[0]);
            make_causal(local_attn_bf, local_attn_bf);
            zero(o);
            mma(o, local_attn_bf, v_col, o); // causal bit.
            store(o_s[tic], o);
            // we have now taken care of the current attention block
        }

        // now we use the previous recurrent KV state to finish o_s[tic]
        rt_bf_1x4<> kv_bf;
        copy(kv_bf, kv_state);
        auto &kv_bf_col = swap_layout_inplace(kv_bf);
        zero(o);
        mma(o, q, kv_bf_col, o);
        store(kv_accumulate[warpid], o);
        __syncthreads();
        tile_reduce<NUM_WORKERS>(o_s[tic], kv_accumulate); // sum onto o_s.

        // we've now successfully compute o_s[tic] -- we can store it.
        __syncthreads();
        G::store(o_g + block*VO_BLOCK::num_elements, o_s[tic], D_VO);

        // finally we need to update the kv state for future iterations
        auto &kt = transpose_inplace(k); // k is now transposed! k has been invalidated; there is only kt.
        mma(kv_state, kt, v_col, kv_state);
    }
}

#include "harness.impl"