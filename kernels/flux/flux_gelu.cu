#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"

using namespace kittens;
template<kittens::ducks::sv::all SV> __device__ static inline void init_bias(rt_fl<16,SV::length> &acc, const SV &bias) {
    #pragma unroll
    for(int i = 0; i < SV::tiles; i++) {
        float2 tmp1 = __bfloat1622float2(*(bf16_2*)&bias.data[16*i + 0 + 2*(laneid()%4)]);
        acc.tiles[0][i].data[0].x = tmp1.x;
        acc.tiles[0][i].data[0].y = tmp1.y;
        acc.tiles[0][i].data[1].x = tmp1.x;
        acc.tiles[0][i].data[1].y = tmp1.y;
        float2 tmp2 = __bfloat1622float2(*(bf16_2*)&bias.data[16*i + 8 + 2*(laneid()%4)]);
        acc.tiles[0][i].data[2].x = tmp2.x;
        acc.tiles[0][i].data[2].y = tmp2.y;
        acc.tiles[0][i].data[3].x = tmp2.x;
        acc.tiles[0][i].data[3].y = tmp2.y;
    }
}
using namespace kittens::prototype;
template<int BLOCK_M, int BLOCK_N, int BLOCK_K, int transpose_lhs, int transpose_rhs>
struct flux_matmul_gelu_layout {
    using lhs_tile  = std::conditional_t<transpose_lhs, st_bf<BLOCK_K,      64>, st_bf<     64, BLOCK_K>>;
    using rhs_tile  = std::conditional_t<transpose_rhs, st_bf<BLOCK_N, BLOCK_K>, st_bf<BLOCK_K, BLOCK_N>>;
    using acc_tile  = st_bf<64, BLOCK_N>;
    using bias_vec  = sv_bf<acc_tile::cols>;
    struct globals { // global layout (here with TMA descriptors)
        gl<bf16, 1, 1, -1, -1, lhs_tile> lhs;
        gl<bf16, 1, 1, -1, -1, rhs_tile> rhs;
        gl<bf16, 1, 1,  1, -1, bias_vec> bias;
        gl<bf16, 1, 1, -1, -1, acc_tile> acc;
    };
    struct input_block {
        lhs_tile lhs[BLOCK_M/64];
        rhs_tile rhs;
    };
    struct scratch_block  { bias_vec bias; };
    struct consumer_state { rt_fl<16, BLOCK_N> acc;   };
    struct finish_block   { acc_tile           acc[BLOCK_M/64]; };
};
template<int BLOCK_M, int BLOCK_N, int BLOCK_K, int transpose_lhs=0, int transpose_rhs=0>
struct flux_matmul_gelu_template {
    using layout = flux_matmul_gelu_layout<BLOCK_M, BLOCK_N, BLOCK_K, transpose_lhs, transpose_rhs>;
    static constexpr int NUM_CONSUMER_WARPS = BLOCK_M/16, NUM_CONSUMER_WARPGROUPS = NUM_CONSUMER_WARPS / 4;
    __device__ static inline int iters(typename layout::globals &g) { return transpose_lhs ? g.lhs.rows / BLOCK_K : g.lhs.cols / BLOCK_K; }
    struct producer {
        __device__ static void setup(producer_setup_args<layout> args) { // setup and load the first iteration
            warpgroup::producer_registers(); // decrease registers for the producer warpgroup
        }
        __device__ static void load(producer_load_args<layout> args) { // barrier for the producer to load into
            if(warpgroup::warpid() == 0) {
                tma::expect_bytes(args.inputs_arrived, sizeof(layout::input_block));
                for(int i = 0; i < NUM_CONSUMER_WARPGROUPS; i++) {
                    if constexpr (transpose_lhs)
                        tma::load_async(args.input.lhs[i], args.globals.lhs, {args.iter, (int)blockIdx.x*NUM_CONSUMER_WARPGROUPS+i}, args.inputs_arrived);
                    else
                        tma::load_async(args.input.lhs[i], args.globals.lhs, {(int)blockIdx.x*NUM_CONSUMER_WARPGROUPS+i, args.iter}, args.inputs_arrived);
                }
                if constexpr (transpose_rhs)
                    tma::load_async(args.input.rhs, args.globals.rhs, {(int)blockIdx.y, args.iter}, args.inputs_arrived);
                else
                    tma::load_async(args.input.rhs, args.globals.rhs, {args.iter, (int)blockIdx.y}, args.inputs_arrived);
            }
            else arrive(args.inputs_arrived);
        }
    };
    struct consumer {
        __device__ static void setup(consumer_setup_args<layout> args) { // setup locals for before the first iteration
            warpgroup::consumer_registers<NUM_CONSUMER_WARPGROUPS>();
            group<NUM_CONSUMER_WARPS>::load(args.scratch.bias, args.globals.bias, {blockIdx.y});
            group<NUM_CONSUMER_WARPS>::sync();
            init_bias(args.state.acc, args.scratch.bias); // <std::remove_reference_t<decltype(args.scratch.bias)>>
        }
        __device__ static void work(consumer_work_args<layout> args) {
            if constexpr (transpose_lhs && transpose_rhs)
                warpgroup::mma_AtBt(args.state.acc, args.input.lhs[warpgroup::groupid()], args.input.rhs);
            else if constexpr (transpose_lhs)
                warpgroup::mma_AtB (args.state.acc, args.input.lhs[warpgroup::groupid()], args.input.rhs);
            else if constexpr (transpose_rhs)
                warpgroup::mma_ABt (args.state.acc, args.input.lhs[warpgroup::groupid()], args.input.rhs);
            else
                warpgroup::mma_AB  (args.state.acc, args.input.lhs[warpgroup::groupid()], args.input.rhs);
            warpgroup::mma_async_wait();
            arrive(args.inputs_finished);
        }
        __device__ static void finish(consumer_finish_args<layout> args) {
            #pragma unroll
            for(int i = 0; i < args.state.acc.width; i++) {
                #pragma unroll
                for(int j = 0; j < 4; j++) {
                    float f = args.state.acc.tiles[0][i].data[j].x, g = args.state.acc.tiles[0][i].data[j].y;
                    args.state.acc.tiles[0][i].data[j].x = f * 0.5f * (1.0f + tanh(f * 0.79788456f * (1 + f * f *0.044715f)));  
                    args.state.acc.tiles[0][i].data[j].y = g * 0.5f * (1.0f + tanh(g * 0.79788456f * (1 + g * g *0.044715f)));  
                } 
            }
            warpgroup::store(args.finish.acc[warpgroup::groupid()], args.state.acc);
            warpgroup::sync();
            if(warpgroup::warpid() == 0)
                tma::store_async(args.globals.acc, args.finish.acc[warpgroup::groupid()],
                {blockIdx.x * NUM_CONSUMER_WARPGROUPS + warpgroup::groupid(), blockIdx.y});
        }
    };
};

#include <iostream>
#include <random>
#include <math.h>
#include <hip/hip_bf16.h>

#include <omp.h>
template<int transpose_lhs, int transpose_rhs>
void cpu_gemm(float* a, float* b, float *bias, float* c, int M, int N, int K) {
    #pragma omp parallel for collapse(2) // otherwise the CPU version takes for everrrrrr
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++) {
                if constexpr (transpose_lhs && transpose_rhs) {
                    sum += a[k * M + i] * b[j * K + k];
                } else if constexpr (transpose_lhs) {
                    sum += a[k * M + i] * b[k * N + j];
                } else if constexpr (transpose_rhs) {
                    sum += a[i * K + k] * b[j * K + k];
                } else {
                    sum += a[i * K + k] * b[k * N + j];
                }
            }
            c[i * N + j] = sum + bias[j];
        }
    }
    for(int i = 0; i < M*N; i++) {
        c[i] = 0.5 * c[i] * (1.0 + tanh(0.79788456 * (c[i] + 0.044715 * c[i]*c[i]*c[i])));
    }
}

int main() {
    constexpr int transpose_lhs = 0, transpose_rhs = 1;
    // const int M = 3072, N = 12288, K = 3072; using fmt = flux_matmul_gelu_template<192, 192, 64>; // 760 TFLOPs
    // const int M = 3072, N = 3072, K = 12288; using fmt = flux_matmul_gelu_template<192, 192, 64>; // 813.5 TFLOPs
    // const int M = 256, N = 12288, K = 3072; using fmt = flux_matmul_gelu_template<128, 192, 64>; // 574.5 TFLOPs
    // const int M = 256, N = 3072, K = 12288; using fmt = flux_matmul_gelu_template<128, 64, 128>; // 433 TFLOPs
    // const int M = 3072, N = 3072, K = 3072; using fmt = flux_matmul_gelu_template<192, 192, 64>; // 740 TFLOPs
    const int M = 3072, N = 3072, K = 6144; using fmt = flux_matmul_gelu_template<192, 192, 64, transpose_lhs, transpose_rhs>; // 813.5 TFLOPs

    using lhs_tile   = typename std::remove_reference<decltype(std::declval<typename fmt::layout::input_block>().lhs[0])>::type;
    using rhs_tile   = typename std::remove_reference<decltype(std::declval<typename fmt::layout::input_block>().rhs)>::type;
    using acc_tile   = typename std::remove_reference<decltype(std::declval<typename fmt::layout::finish_block>().acc[0])>::type;
    using lhs_global = typename std::remove_reference<decltype(std::declval<typename fmt::layout::globals>().lhs)>::type;
    using rhs_global = typename std::remove_reference<decltype(std::declval<typename fmt::layout::globals>().rhs)>::type;
    using bias_global = typename std::remove_reference<decltype(std::declval<typename fmt::layout::globals>().bias)>::type;
    using acc_global = typename std::remove_reference<decltype(std::declval<typename fmt::layout::globals>().acc)>::type;
    using globals  = typename fmt::layout::globals;

    std::cout << "Has store: "  << (bool)kittens::prototype::detail::has_store<fmt>  << '\n';
    std::cout << "Has finish: " << (bool)kittens::prototype::detail::has_finish<fmt> << '\n';
    std::cout << "Transpose LHS: " << (bool)transpose_lhs << '\n';
    std::cout << "Transpose RHS: " << (bool)transpose_rhs << '\n';

    // Allocate host memory
    float *h_A = new float[M * K];
    float *h_B = new float[K * N];
    float *h_bias = new float[N];
    float *h_C = new float[M * N];
    float *h_C_ref = new float[M * N];

    std::cout << "Allocated host memory" << std::endl;

    // Initialize random number generator
    std::random_device rd;
    std::mt19937 gen(42);
    std::uniform_real_distribution<> dis(-1.0, 1.0);

    // Initialize matrices with random values
    for (int i = 0; i < M * K; ++i) h_A[i] = dis(gen);
    for (int i = 0; i < K * N; ++i) h_B[i] = dis(gen);
    for (int i = 0; i < N; ++i) h_bias[i] = dis(gen);
    std::cout << "Initialized matrices" << std::endl;

    // Perform CPU matrix multiplication for reference
    cpu_gemm<transpose_lhs, transpose_rhs>(h_A, h_B, h_bias, h_C_ref, M, N, K);

    std::cout << "Performed CPU matrix multiplication" << std::endl;

    // Allocate device memory
    __hip_bfloat16 *d_A, *d_B, *d_C, *d_bias;
    hipMalloc(&d_A, M*K*2);
    hipMalloc(&d_B, K*N*2);
    hipMalloc(&d_C, M*N*2);
    hipMalloc(&d_bias, N*2);

    std::cout << "Allocated device memory" << std::endl;

    std::cout << "lhs_tile::rows=" << lhs_tile::rows << " lhs_tile::cols=" << lhs_tile::cols << std::endl;
    std::cout << "rhs_tile::rows=" << rhs_tile::rows << " rhs_tile::cols=" << rhs_tile::cols << std::endl;
    std::cout << "acc_tile::rows=" << acc_tile::rows << " acc_tile::cols=" << acc_tile::cols << std::endl;
    lhs_global Ag{d_A, nullptr, nullptr, transpose_lhs ? K : M, transpose_lhs ? M : K};
    rhs_global Bg{d_B, nullptr, nullptr, transpose_rhs ? N : K, transpose_rhs ? K : N};
    acc_global Cg{d_C, nullptr, nullptr, M, N};
    bias_global Biasg{d_bias, nullptr, nullptr, nullptr, N};
    globals G{Ag, Bg, Biasg, Cg};

    std::cout << "Allocated memory" << std::endl;

    // Convert to __hip_bfloat16 and copy to device
    __hip_bfloat16 *h_A_bf16 = new __hip_bfloat16[M * K];
    __hip_bfloat16 *h_B_bf16 = new __hip_bfloat16[K * N];
    __hip_bfloat16 *h_bias_bf16 = new __hip_bfloat16[N];
    for (int i = 0; i < M * K; ++i) h_A_bf16[i] = __float2bfloat16(h_A[i]);
    for (int i = 0; i < K * N; ++i) h_B_bf16[i] = __float2bfloat16(h_B[i]);
    for (int i = 0; i < N; ++i) h_bias_bf16[i] = __float2bfloat16(h_bias[i]);

    hipMemcpy(d_A, h_A_bf16, M*K*2, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B_bf16, K*N*2, hipMemcpyHostToDevice);
    hipMemcpy(d_bias, h_bias_bf16, N*2, hipMemcpyHostToDevice);

    std::cout << "Copied matrices to device" << std::endl;

    unsigned long mem_size = MAX_SHARED_MEMORY; // need to launch two blocks if possible.
    
    hipFuncSetAttribute(reinterpret_cast<const void*>(prototype::pc<fmt>), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
    // Launch kernel
    dim3 grid(M / (acc_tile::rows*prototype::num_consumer_warpgroups<fmt>), N / acc_tile::cols); // rows, cols
    dim3 block(prototype::num_threads<fmt>);

    // Start timing
    hipDeviceSynchronize();
    std::cout << "Launching kernel with grid (" << grid.x << ", " << grid.y << "), block (" << block.x << "), and " << K/lhs_tile::cols << " reduction block dimension\n";
    std::cout << "Kernel has " << kittens::prototype::input_pipe_stages<fmt> << " input pipeline stages and " << kittens::prototype::output_pipe_stages<fmt> << " output pipeline stages\n";
    auto start = std::chrono::high_resolution_clock::now();

    constexpr int ITERS = 100;
    for(int i = 0; i < ITERS; i++) {
        prototype::pc<fmt><<<grid, block, mem_size>>>(G);
    }
    hipDeviceSynchronize();

    // End timing
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate duration
    std::chrono::duration<double> diff = end - start;
    double seconds = diff.count();

    // Calculate TFLOPs
    double flops = double(2.0) * M * N * K * ITERS; // 2 FLOPs per multiply-add
    double tflops = (flops / seconds) / 1e12;

    std::cout << "Kernel execution time: " << seconds << " seconds\n";
    std::cout << "Achieved performance: " << tflops << " TFLOPs\n";
    
    // Check for CUDA errors
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
        // Optionally, you might want to exit the program or handle the error in some way
        return -1;
    }

    // Copy result back to host
    __hip_bfloat16 *h_C_bf16 = new __hip_bfloat16[M * N];
    hipMemcpy(h_C_bf16, d_C, M*N*2, hipMemcpyDeviceToHost);

    std::cout << "Copied result back to host" << std::endl;

    // Convert result back to float for comparison
    for (int i = 0; i < M * N; ++i) h_C[i] = __bfloat162float(h_C_bf16[i]);

    std::cout << "Converted result back to float" << std::endl;

    // Check result
    float max_error = 0.0f;
    int error_count = 0;
    for (int i = 0; i < M * N; ++i) {
        float error = std::abs(h_C[i] - h_C_ref[i]);
        if(error > 1.0) { // large because of bf16 vs fp32 numerics
            if(error_count < 20) std::cout << "Error at row " << i / N << " col " << i % N << ": " << h_C[i] << " != " << h_C_ref[i] << " (ref)" << std::endl;
            else if(error_count == 21) std::cout << "Too many errors to show them all.\n";
            error_count++;
        }
        max_error = std::max(max_error, error);
    }

    std::cout << "Max error: " << max_error << std::endl;
    std::cout << "Error count: " << error_count << std::endl;

    // Clean up
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_ref;
    delete[] h_A_bf16;
    delete[] h_B_bf16;
    delete[] h_C_bf16;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
