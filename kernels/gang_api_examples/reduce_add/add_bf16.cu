#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include <random>
#include <hip/hip_bf16.h>  // Include bfloat16 support

constexpr int NUM_DEVICES = 2;
constexpr size_t N = 64;

using namespace kittens;

// Change the layout to use __hip_bfloat16 instead of float
using global_layout   =  gl<__hip_bfloat16, 1, 1, -1, -1>;
using pgl_m  =  pgl_manager<gl<__hip_bfloat16, 1, 1, -1, -1>, true>;
using kittens_pgl = kittens::pgl<global_layout>;

// kittens::atomic_add(p_o);
__global__ void all_reduce_int(kittens_pgl p_o) {
    if (threadIdx.x != 0 || threadIdx.y != 0 || threadIdx.z != 0 ||
        blockIdx.x != 0 || blockIdx.y != 0 || blockIdx.z != 0) {
        return;
    }
    bf16 *value = p_o.mc_ptr;
    unsigned int packed = (__bfloat16_as_ushort(value[0]) << 16) | 
                          __bfloat16_as_ushort(value[1]);
    asm volatile(
        "multimem.red.relaxed.sys.global.add.bf16x2 [%0], %1;"
        :
        : "l"(p_o.mc_ptr), "r"(packed)
        : "memory"
    );
    // bf16 *value = p_o.mc_ptr;
    // asm volatile(
    //     "multimem.red.relaxed.sys.global.add.bf16 [%0], %1;"
    //     :
    //     : "l"(p_o.mc_ptr), "h"(__bfloat16_as_ushort(value[0]))
    //     : "memory"
    // );
    
    // bf16 *value = p_o.mc_ptr;
    // unsigned int packed1 = (__bfloat16_as_ushort(value[0]) << 16) | 
    //                             __bfloat16_as_ushort(value[1]);
    // unsigned int packed2 = (__bfloat16_as_ushort(value[2]) << 16) | 
    //                         __bfloat16_as_ushort(value[3]);
    // unsigned int packed3 = (__bfloat16_as_ushort(value[4]) << 16) |
    //                         __bfloat16_as_ushort(value[5]);
    // unsigned int packed4 = (__bfloat16_as_ushort(value[6]) << 16) |
    //                         __bfloat16_as_ushort(value[7]);
    // asm volatile(
    //     "multimem.red.relaxed.sys.global.add.v4.bf16x2 [%0], {%1, %2, %3, %4};"
    //     :
    //     : "l"(p_o.mc_ptr), "r"(packed1), "r"(packed2), "r"(packed3), "r"(packed4)
    //     : "memory"
    // );
}

int main() {
    // Setup
    int nelem = N * N;
    size_t size = nelem * sizeof(__hip_bfloat16);  // Use bfloat16 size

    // Create and initialize host arrays with bfloat16 values
    __hip_bfloat16 *host_mat_1 = new __hip_bfloat16[nelem];
    for (int i = 0; i < nelem; ++i) host_mat_1[i] = __float2bfloat16(1.5f);

    __hip_bfloat16 *host_mat_2 = new __hip_bfloat16[nelem];
    for (int i = 0; i < nelem; ++i) host_mat_2[i] = __float2bfloat16(static_cast<float>(i));

    // Print data - convert bfloat16 to float for printing
    printf("Device 1: ");
    for (int i = 0; i < 10; ++i) {
        printf("%f ", __bfloat162float(host_mat_1[i]));
    }
    printf("... (%d elements)\n", nelem);

    printf("Device 2: ");
    for (int i = 0; i < 10; ++i) {
        printf("%f ", __bfloat162float(host_mat_2[i]));
    }
    printf("... (%d elements)\n", nelem);
    
    // Allocate and copy data to device
    __hip_bfloat16 **dev_mats = new __hip_bfloat16*[NUM_DEVICES];
    hipMemGenericAllocationHandle_t *dev_handles = new hipMemGenericAllocationHandle_t[NUM_DEVICES];

    int device_ids[NUM_DEVICES];
    for (int i = 0; i < NUM_DEVICES; ++i) device_ids[i] = i;
    
    hipSetDevice(0);
    pglCudaMalloc(NUM_DEVICES, device_ids, 0, &dev_mats[0], &dev_handles[0], size);
    hipMemcpy(dev_mats[0], host_mat_1, size, hipMemcpyHostToDevice);

    hipSetDevice(1);
    pglCudaMalloc(NUM_DEVICES, device_ids, 1, &dev_mats[1], &dev_handles[1], size);
    hipMemcpy(dev_mats[1], host_mat_2, size, hipMemcpyHostToDevice);

    // Initialize parallel global layout
    pgl_m dev_mat_pgl{device_ids, NUM_DEVICES, dev_mats, nullptr, nullptr, N, N};

    // Perform the reduction
    KittensClub club(device_ids, NUM_DEVICES);
    
    dim3 grid(1);
    dim3 block(32);
    hipSetDevice(0);
    all_reduce_int<<<grid, block>>>(dev_mat_pgl.get_pgl_obj(0));
    CHECK_CUDA_ERROR(hipDeviceSynchronize());

    // Bring back data
    hipMemcpy(host_mat_1, dev_mats[0], size, hipMemcpyDeviceToHost);
    hipSetDevice(1);
    hipMemcpy(host_mat_2, dev_mats[1], size, hipMemcpyDeviceToHost);
    
    // Print results - convert bfloat16 to float for printing
    printf("Device 1: ");
    for (int i = 0; i < 10; ++i) {
        printf("%f ", __bfloat162float(host_mat_1[i]));
    }
    printf("... (%d elements)\n", nelem);

    printf("Device 2: ");
    for (int i = 0; i < 10; ++i) {
        printf("%f ", __bfloat162float(host_mat_2[i]));
    }
    printf("... (%d elements)\n", nelem);

    // Check correctness, for Device 1, all elements should be 1.5 + 1.5 = 3.0
    for (int i = 0; i < nelem; ++i) {
        if (host_mat_1[i] != __float2bfloat16(3.0f)) {
            // printf("%d ", i);
            std::cerr << "Error: Device 1, index " << i << " expected " << 3.0f << " but got " << __bfloat162float(host_mat_1[i]) << std::endl;
            return -1;
        }
    }

    // Cleanup and exit
    delete[] dev_mats;
    delete[] dev_handles;
    delete[] host_mat_1;
    delete[] host_mat_2;

    std::cout << "Done!" << std::endl;
    return 0;
}