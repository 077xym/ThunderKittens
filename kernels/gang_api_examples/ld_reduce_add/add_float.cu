#include "hip/hip_runtime.h"
#include "kittens.cuh"

#include <random>

constexpr int NUM_DEVICES = 8;
constexpr size_t N = 4096;

constexpr int ITER_PER_THREAD = 32;
constexpr int MAX_VEC_SIZE = 16;

using namespace kittens;

using base_tile       =  st_fl<64, 64>;
using global_layout   =  gl<float, 1, 1, -1, -1, base_tile>;
using pglobal_layout  =  pgl<gl<float, 1, 1, -1, -1, base_tile>, true>;
using kittens_pgl = kittens::PglObj<global_layout>;

__global__ void all_reduce_float(kittens_pgl p_o) {
    kittens::all_reduce_add(p_o);
}

int main() {
    std::random_device rd;
    std::mt19937 gen(32);
    std::uniform_real_distribution<> dis(-0.5, 0.5);

    // Setup
    int nelem = N * N;
    size_t size = nelem * sizeof(float);

    // Allocate and initialize host memory
    float **host_mats = new float*[NUM_DEVICES];
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        host_mats[dev_idx] = new float[nelem];
        for (int i = 0; i < nelem; ++i) host_mats[dev_idx][i] = dis(gen);
    }
    
    float *expected = new float[nelem];
    for (int i = 0; i < nelem; ++i) {
        expected[i] = 0.0f;
        for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx)
            expected[i] += host_mats[dev_idx][i];
    }

    // Print data
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        std::cout << "Device " << dev_idx << ": ";
        for (int i = 0; i < std::min(nelem, 10); ++i) {
            std::cout << host_mats[dev_idx][i] << " ";
        }
        std::cout << "... (" << nelem << " elements)" << std::endl;
    }
    std::cout << "Expected: ";
    for (int i = 0; i < std::min(nelem, 10); ++i) {
        std::cout << expected[i] << " ";
    }
    std::cout << "... (" << nelem << " elements)" << std::endl;

    // Allocate and copy data to device
    float **dev_mats = new float*[NUM_DEVICES];
    hipMemGenericAllocationHandle_t *dev_handles = new hipMemGenericAllocationHandle_t[NUM_DEVICES];
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        hipSetDevice(dev_idx);
        pglCudaMalloc(dev_idx, &dev_mats[dev_idx], &dev_handles[dev_idx], size);
        hipMemcpy(dev_mats[dev_idx], host_mats[dev_idx], size, hipMemcpyHostToDevice);
    }

    // Initialize parallel global layout
    int device_ids[NUM_DEVICES];
    for (int i = 0; i < NUM_DEVICES; ++i) device_ids[i] = i;
    pglobal_layout dev_mat_pgl{device_ids, NUM_DEVICES, dev_mats, nullptr, nullptr, N, N};

    // Perform the reduction
    KittensClub club(device_ids, NUM_DEVICES);

    int nelem_per_dev = nelem / NUM_DEVICES;
    constexpr int nelem_per_block = 256 * ITER_PER_THREAD * (MAX_VEC_SIZE / sizeof(float));

    dim3 grid((nelem_per_dev + nelem_per_block - 1) / nelem_per_block);
    dim3 block(64, 4);
    club.execute([&](int worker_id) {
        all_reduce_float<<<grid, block>>>(dev_mat_pgl.get_pgl_obj(worker_id));
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
    });

    // Bring back data
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        hipSetDevice(dev_idx);
        hipMemcpy(host_mats[dev_idx], dev_mats[dev_idx], size, hipMemcpyDeviceToHost);
    }

    // Print results
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        std::cout << "Device " << dev_idx << ": ";
        for (int i = 0; i < std::min(nelem, 10); ++i) {
            std::cout << host_mats[dev_idx][i] << " ";
        }
        std::cout << "... (" << nelem << " elements)" << std::endl;
    }

    // Verify the results
    float TOL = 1e-5; // Can use tighter tolerance since we're using full precision floats
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        for (int i = 0; i < nelem; ++i) {
            if (fabs(expected[i] - host_mats[dev_idx][i]) > TOL) {
                std::cerr << "Mismatch at device " << dev_idx << 
                             ", index " << i << 
                             ": expected " << expected[i] << 
                             ", got " << host_mats[dev_idx][i] << std::endl;
                exit(EXIT_FAILURE);
            }
        }
    }

    // Cleanup and exit
    for (int dev_idx = 0; dev_idx < NUM_DEVICES; ++dev_idx) {
        delete[] host_mats[dev_idx];
        pglCudaFree(dev_idx, dev_mats[dev_idx], dev_handles[dev_idx], size);
    }
    delete[] host_mats;
    delete[] expected;
    delete[] dev_mats;
    delete[] dev_handles;

    std::cout << "Done!" << std::endl;
    return 0;
}