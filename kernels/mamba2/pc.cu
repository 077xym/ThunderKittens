#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"

#ifdef TORCH_COMPILE
#define TK_COMPILE_MAMBA2
#endif

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::lcsf;
struct mamba2_fwd_layout {
	using q_tile   = st_bf<64, 64>;
	using k_tile   = st_bf<64, 64>;
	using v_tile   = st_bf<64, 64>;
	using o_tile   = st_bf<64, 64>;
    using a_vec    = sv_fl<64>; // decays
	using q_global = kittens::gl<bf16, -1, -1, -1, 64, q_tile>; // B, H, N, S
	using k_global = kittens::gl<bf16, -1, -1, -1, 64, k_tile>;
	using v_global = kittens::gl<bf16, -1, -1, -1, 64, v_tile>;
	using o_global = kittens::gl<bf16, -1, -1, -1, 64, o_tile>;
    using a_global = kittens::gl<float, -1, -1,  1, -1, a_vec>;
	struct globals { q_global Q; k_global K; v_global V; o_global O; a_global A; };
	struct input_block    { 
        q_tile q;
        k_tile k;
        v_tile v;
        a_vec  a;
        a_vec  padding[7];
    };
    struct output_block {
        o_tile o;
    };
	struct scratch_block  { 
        st_bf<64, 64> kv, k;
        a_vec         a_cumsum;
        a_vec         padding[7];
    };
    struct common_state {
        int batch, head;
    };
	struct consumer_state {
		rt_fl<16, 64> o_reg;
		rt_fl<16, 64> att_block;
		rt_bf<16, 64> att_block_mma;
        rt_fl<16, 64> local_decay;
        rt_bf<16, 64> q_reg, k_reg;
        rt_fl<16, 64> kv;
	};
};
struct mamba2_fwd_template {
	static constexpr int NUM_CONSUMER_WARPS = 4, NUM_BLOCKS=2,
        OUTPUT_PIPE_STAGES=2, INPUT_PIPE_STAGES=2;
	using layout = mamba2_fwd_layout;
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        int task_id = args.task_iter*gridDim.x + blockIdx.x;
		args.common.batch = task_id/args.globals.Q.depth; // batch = id / heads.
		task_id -= args.common.batch*args.globals.Q.depth;
		args.common.head = task_id;
		args.num_iters = args.common.batch < args.globals.Q.batch ? args.globals.K.rows/layout::k_tile::rows : -1;
    }
	struct producer {
		__device__ static void setup(producer_setup_args<layout> args) {
			warpgroup::producer_registers();
		}
		__device__ static void load(producer_load_args<layout> args) {
			if(warpgroup::warpid() == 0) {
                tma::expect(args.inputs_arrived, args.input.q, args.input.k, args.input.v, args.input.a);
                tma::load_async(args.input.q, args.globals.Q, {args.common.batch,                 0, args.iter, 0}, args.inputs_arrived);
                tma::load_async(args.input.k, args.globals.K, {args.common.batch,                 0, args.iter, 0}, args.inputs_arrived);
                tma::load_async(args.input.v, args.globals.V, {args.common.batch,  args.common.head, args.iter, 0}, args.inputs_arrived);
                tma::load_async(args.input.a, args.globals.A, {args.common.batch,  args.common.head, 0, args.iter}, args.inputs_arrived);
                if(laneid() == 0) arrive(args.inputs_arrived, 3);
            }
		}
        __device__ static void store(producer_store_args<layout> args) {
            if(warpgroup::warpid() == 0) {;
                tma::store_async(args.globals.O, args.output.o, {args.common.batch, args.common.head, args.iter, 0});
                tma::store_async_read_wait();
                __syncwarp();
                if(laneid() == 0) arrive(args.outputs_finished, 4);
            }
        }
	};
	struct consumer {
		__device__ static void setup(consumer_setup_args<layout> args) {
			warpgroup::increase_registers<224>();
            zero(args.state.kv);
		}
		__device__ static bool compute(consumer_compute_args<layout> args) {
            int warpgroupid = warpgroup::warpid()/kittens::WARPGROUP_WARPS;

            // Start by doing cumsum into shared memory
            warpgroup::sync(warpgroupid + 4);
            warpgroup::copy(args.scratch.a_cumsum, args.input.a);
            warpgroup::sync(warpgroupid + 4);
            if(warpid() <= 1) {
                // Perform the prefix sum (Hillis-Steele scan)
                for (int offset = 1; offset < 64; offset *= 2) {
                    float temp = (threadIdx.x >= offset) ? args.scratch.a_cumsum[threadIdx.x - offset] : 0.0f;
                    group<2>::sync(14);
                    args.scratch.a_cumsum[threadIdx.x] += temp;
                    group<2>::sync(14);
                }
            }
            warpgroup::sync(warpgroupid + 4); // cumulative sum done
            // Calculate decays
            #pragma unroll
            for(int i = 0; i < 4; i++) {
                int base_row = warpgroup::warpid()*16 + laneid()/4;
                int base_col = i*16 + (laneid()%4)*2;
                args.state.local_decay.tiles[0][i].data[0].x = args.scratch.a_cumsum[base_row + 0] - args.scratch.a_cumsum[base_col + 0];
                args.state.local_decay.tiles[0][i].data[0].y = args.scratch.a_cumsum[base_row + 0] - args.scratch.a_cumsum[base_col + 1];
                args.state.local_decay.tiles[0][i].data[1].x = args.scratch.a_cumsum[base_row + 8] - args.scratch.a_cumsum[base_col + 0];
                args.state.local_decay.tiles[0][i].data[1].y = args.scratch.a_cumsum[base_row + 8] - args.scratch.a_cumsum[base_col + 1];
                args.state.local_decay.tiles[0][i].data[2].x = args.scratch.a_cumsum[base_row + 0] - args.scratch.a_cumsum[base_col + 8];
                args.state.local_decay.tiles[0][i].data[2].y = args.scratch.a_cumsum[base_row + 0] - args.scratch.a_cumsum[base_col + 9];
                args.state.local_decay.tiles[0][i].data[3].x = args.scratch.a_cumsum[base_row + 8] - args.scratch.a_cumsum[base_col + 8];
                args.state.local_decay.tiles[0][i].data[3].y = args.scratch.a_cumsum[base_row + 8] - args.scratch.a_cumsum[base_col + 9];
            }
            exp(args.state.local_decay, args.state.local_decay);
            // causal mask
            #pragma unroll
            for(int i = 0; i < 4; i++) { // causal mask
                auto &decay_subtile = reinterpret_cast<rt_fl<16,16>&>(args.state.local_decay.tiles[0][i]);
                if      (i >  warpgroup::warpid()) { zero       (decay_subtile); }
                else if (i == warpgroup::warpid()) { make_causal(decay_subtile, decay_subtile, kittens::base_types::constants<float>::zero()); }
            }
      		// A = Q @ K.T
            warpgroup::load(args.state.q_reg, args.input.q); // we need this later, anyways
			warpgroup::mm_ABt(args.state.att_block, args.state.q_reg, args.input.k);
			warpgroup::mma_async_wait();
            mul(args.state.att_block, args.state.att_block, args.state.local_decay);
            copy(args.state.att_block_mma, args.state.att_block);
            warpgroup::mm_AB(args.state.o_reg, args.state.att_block_mma, args.input.v);
            warpgroup::mma_async_wait();
            // // multiply q by decays
            {
                int base_row = warpgroup::warpid()*16 + laneid()/4;
                bf16 top = __float2bfloat16(expf(args.scratch.a_cumsum[base_row + 0]));
                bf16 bottom = __float2bfloat16(expf(args.scratch.a_cumsum[base_row +8]));
                #pragma unroll
                for(int i = 0; i < 4; i++) {
                    args.state.q_reg.tiles[0][i].data[0].x *= top;
                    args.state.q_reg.tiles[0][i].data[0].y *= top;
                    args.state.q_reg.tiles[0][i].data[1].x *= bottom;
                    args.state.q_reg.tiles[0][i].data[1].y *= bottom;
                    args.state.q_reg.tiles[0][i].data[2].x *= top;
                    args.state.q_reg.tiles[0][i].data[2].y *= top;
                    args.state.q_reg.tiles[0][i].data[3].x *= bottom;
                    args.state.q_reg.tiles[0][i].data[3].y *= bottom;
                }
            }
            warpgroup::store(args.scratch.kv, args.state.kv);
            warpgroup::sync(warpgroupid + 4);
            warpgroup::mma_AB(args.state.o_reg, args.state.q_reg, args.scratch.kv);
            warpgroup::mma_async_wait();
            warpgroup::store(args.output.o, args.state.o_reg);
            warpgroup::sync(warpgroupid + 4);
            float last_decay = args.scratch.a_cumsum[args.scratch.a_cumsum.length-1]; // last element
            float total_decay = expf(last_decay);
            mul(args.state.kv, args.state.kv, total_decay); // decay kv
            warpgroup::load(args.state.k_reg, args.input.k); // multiply k's by decays
            {
                int base_row = warpgroup::warpid()*16 + laneid()/4;
                bf16 top = __float2bfloat16(expf(last_decay - args.scratch.a_cumsum[base_row + 0]));
                bf16 bottom = __float2bfloat16(expf(last_decay - args.scratch.a_cumsum[base_row +8]));
                #pragma unroll
                for(int i = 0; i < 4; i++) {
                    args.state.k_reg.tiles[0][i].data[0].x *= top;
                    args.state.k_reg.tiles[0][i].data[0].y *= top;
                    args.state.k_reg.tiles[0][i].data[1].x *= bottom;
                    args.state.k_reg.tiles[0][i].data[1].y *= bottom;
                    args.state.k_reg.tiles[0][i].data[2].x *= top;
                    args.state.k_reg.tiles[0][i].data[2].y *= top;
                    args.state.k_reg.tiles[0][i].data[3].x *= bottom;
                    args.state.k_reg.tiles[0][i].data[3].y *= bottom;
                }
            }
            warpgroup::store(args.scratch.k, args.state.k_reg); // using as dummy memory
            warpgroup::sync(warpgroupid + 4);
            warpgroup::mma_AtB(args.state.kv, args.scratch.k, args.input.v);
            warpgroup::mma_async_wait();
            if(laneid() == 0) arrive(args.outputs_arrived);
            if(laneid() == 0) arrive(args.inputs_finished);
		}
        __device__ static void finish(consumer_finish_args<layout> args) {
            if(laneid() == 0) arrive(args.finish_finished);
        }
	};
};

#ifdef TK_COMPILE_MAMBA2
#include "common/pyutils/torch_helpers.cuh"
#include <iostream>
void dispatch_mamba2(
    bf16 *d_q, bf16 *d_k, bf16 *d_v, 
    bf16 *d_o, float *d_a,
    int B, int H, int N
){

    mamba2_fwd_template::layout::q_global Qg(d_q, B, 1, N, nullptr);
    mamba2_fwd_template::layout::k_global Kg(d_k, B, 1, N, nullptr);
    mamba2_fwd_template::layout::a_global Ag(d_a, B, H, nullptr, N);
    mamba2_fwd_template::layout::v_global Vg(d_v, B, H, N, nullptr);
    mamba2_fwd_template::layout::o_global Og(d_o, B, H, N, nullptr);

    mamba2_fwd_template::layout::globals globals = {Qg, Kg, Vg, Og, Ag};
    
    // launch setup
    unsigned long mem_size = (kittens::MAX_SHARED_MEMORY/2)-2048;
    
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        prototype::lcsf::kernel<mamba2_fwd_template>),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        mem_size
    );

    dim3 grid(264, 1, 1);

    constexpr int BLOCK_SIZE = prototype::detail::NUM_THREADS_v<mamba2_fwd_template>;
    prototype::lcsf::kernel<mamba2_fwd_template><<<grid, BLOCK_SIZE, mem_size>>>(globals);
}


torch::Tensor mamba2(
    const torch::Tensor q,
    const torch::Tensor k,
    const torch::Tensor v,
    const torch::Tensor a
) {
    CHECK_INPUT(q);
    CHECK_INPUT(k);
    CHECK_INPUT(v);
    CHECK_INPUT(a);

    int B = v.size(0);
    int H = v.size(1);
    int N = v.size(2);
    int D = v.size(3);
    
    // checks
    TORCH_CHECK(q.size(0) == B, "q has incompatible batch");
    // TORCH_CHECK(q.size(1) == H, "q has incompatible heads");
    TORCH_CHECK(q.size(2) == N, "q has incompatible sequence shape");

    TORCH_CHECK(k.size(0) == B, "k has incompatible batch");
    // TORCH_CHECK(k.size(1) == H, "k has incompatible heads");
    TORCH_CHECK(k.size(2) == N, "k has incompatible sequence");

    TORCH_CHECK(v.size(0) == B, "v has incompatible dim");
    // TORCH_CHECK(v.size(1) == H, "v has incompatible heads");
    TORCH_CHECK(v.size(2) == N, "v has incompatible sequence");

    torch::Tensor out = torch::empty({B, H, N, D}, q.options());

    // convert to bf16
    c10::BFloat16 *q_bf16 = q.data_ptr<c10::BFloat16>();
    c10::BFloat16 *k_bf16 = k.data_ptr<c10::BFloat16>();
    c10::BFloat16 *v_bf16 = v.data_ptr<c10::BFloat16>();
    // float for a
    float *a_bf16 = a.data_ptr<float>();

    bf16 *d_q = reinterpret_cast<bf16*>(q_bf16);
    bf16 *d_k = reinterpret_cast<bf16*>(k_bf16);
    bf16 *d_v = reinterpret_cast<bf16*>(v_bf16);
    float *d_a = reinterpret_cast<float*>(a_bf16);
    bf16 *d_o = reinterpret_cast<bf16*>(out.data_ptr<c10::BFloat16>());
    
    dispatch_mamba2(
        d_q, d_k, d_v, d_o, d_a,
        B, H, N
    );

    CHECK_CUDA_ERROR(hipGetLastError());
    return out;
}
#else
#include "harness.impl"
#endif

