#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"

using namespace kittens;
using namespace kittens::prototype;
template<int M_BLOCK=2, int N_BLOCK=4>
struct matmul_layout {
	using  base_tile      = st_bf<64, 64>;
	using  global_layout  = gl<bf16, 1, 1, -1, -1, base_tile>;
	struct globals        { global_layout A, B, C; };
	struct input_block    { base_tile a[M_BLOCK], b[N_BLOCK]; };
	struct finish_block   { base_tile c[M_BLOCK][N_BLOCK]; };
	struct producer_state { kittens::coord coords; };
	struct consumer_state { kittens::coord coords;
									        rt_fl<16, N_BLOCK*base_tile::cols> accum; };
};
template<int _M_BLOCK=2, int _N_BLOCK=4, int _SUPER_M=12>
struct matmul_template {
  static constexpr int M_BLOCK = _M_BLOCK, N_BLOCK = _N_BLOCK, SUPER_M = _SUPER_M;
	using layout    = matmul_layout<M_BLOCK, N_BLOCK>;
	using wide_tile = st_bf<64, 64*N_BLOCK>;
	static constexpr int NUM_CONSUMER_WARPS = M_BLOCK*4;
  	// Helper functions
	__host__ static inline dim3 grid(int M, int N, int K) {
		return dim3(M*N/(M_BLOCK*N_BLOCK*layout::base_tile::num_elements));
	}
	__device__ static inline void get_coords(kittens::coord &coords, const typename layout::globals &g, int id) {
		int Rblocks = g.C.rows / (M_BLOCK*64), Cblocks = g.C.cols / (N_BLOCK*64);
		int super_rows = (Rblocks/SUPER_M)*SUPER_M,
		final_rows = Rblocks - super_rows,
		super_repeat = SUPER_M*Cblocks;
		if (blockIdx.x < super_rows * Cblocks)
			coords = { SUPER_M*(blockIdx.x/super_repeat) + blockIdx.x%SUPER_M,
						   (blockIdx.x%super_repeat)/SUPER_M };
		else {
			int remainder_id = blockIdx.x - super_rows*Cblocks;
			coords = { super_rows + (remainder_id%final_rows), remainder_id/final_rows };
		}
		coords = { iters(g), coords.r*M_BLOCK + id, coords.c*N_BLOCK };
	}
  // ThunderKittens template functions
	__device__ static inline int iters(const typename layout::globals &g) { return g.A.cols/64; }
	struct producer {
		__device__ static void setup(producer_setup_args<layout> args) {
			warpgroup::producer_registers(); // decrease registers for producers
			get_coords(args.state.coords, args.globals, 0);
		}
		__device__ static void load(producer_load_args<layout> args) {
			if(warpgroup::warpid() == 0) {
				tma::expect(args.inputs_arrived, args.input);
				for(int i = 0; i < M_BLOCK; i++)
					tma::load_async(args.input.a[i], args.globals.A,
						              {args.state.coords.r+i, args.iter}, args.inputs_arrived);
				for(int i = 0; i < N_BLOCK; i++)
					tma::load_async(args.input.b[i], args.globals.B,
											    {args.iter, args.state.coords.c+i}, args.inputs_arrived);
				arrive(args.inputs_arrived, 3);
			}
		}
	};
	struct consumer {
		__device__ static void setup(consumer_setup_args<layout> args) {
			warpgroup::consumer_registers<NUM_CONSUMER_WARPS/4>(); // increase registers for consumers
			get_coords(args.state.coords, args.globals, warpgroup::groupid());
			zero(args.state.accum);
		}
		__device__ static void work(consumer_work_args<layout> args) {
			warpgroup::mma_AB(
				args.state.accum, // dest registers
				args.input.a[warpgroup::groupid()], // A matrix
				reinterpret_cast<wide_tile&>(args.input.b) // B matrix
			);
			warpgroup::mma_async_wait();
			if(warpgroup::laneid() == 0) arrive(args.inputs_finished, 4);
		}
		__device__ static void finish(consumer_finish_args<layout> args) {
			warpgroup::store(reinterpret_cast<wide_tile&>(args.finish.c[warpgroup::groupid()]), args.state.accum);
			warpgroup::sync();
			if(warpgroup::warpid() == 0) for(int i = 0; i < N_BLOCK; i++)
				tma::store_async(args.globals.C, args.finish.c[warpgroup::groupid()][i],
										     {args.state.coords.r, args.state.coords.c+i});
		}
	};
};


constexpr bool NCU = false;
#include <iostream>
#include <random>
#include <hip/hip_bf16.h>
#include <omp.h>

void cpu_gemm(float* a, float* b, float* c, int M, int N, int K) {
	#pragma omp parallel for collapse(2) // otherwise the CPU version takes for everrrrrr
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			float sum = 0.0f;
			for (int k = 0; k < K; k++) {
				sum += a[i * K + k] * b[k * N + j];
			}
			c[i * N + j] = sum;
		}
	}
}

template<typename mmt>
void inner_run(bf16 *d_A, bf16 *d_B, bf16 *d_C, int M, int N, int K, dim3 grid, dim3 block) {
	using global_layout = typename mmt::layout::global_layout;
	using globals  = typename mmt::layout::globals;
	global_layout Ag{d_A, nullptr, nullptr, M, K};
	global_layout Bg{d_B, nullptr, nullptr, K, N};
	global_layout Cg{d_C, nullptr, nullptr, M, N};
	globals G{Ag, Bg, Cg};
	prototype::pc<mmt><<<grid, block, MAX_SHARED_MEMORY-1024>>>(G);
}

template<typename mmt>
int run_benchmark(size_t M, size_t N, size_t K) {
	hipError_t cudaStatus;

	std::cout << "--------------------  M=" << M << " N=" << N << " K=" << K << "  --------------------\n";
	std::cout << "Block size: " << mmt::M_BLOCK*64 << "x" << mmt::N_BLOCK*64 << "\n";

	// Allocate host memory
	float *h_A = new float[M * K];
	float *h_B = new float[K * N];
	float *h_C = new float[M * N];
	float *h_C_ref = new float[M * N];

	std::cout << "Allocated host memory" << std::endl;

	// Initialize random number generator
	std::random_device rd;
	std::mt19937 gen(42);
	std::uniform_real_distribution<> dis(-0.5, 0.5);

	// Initialize matrices with random values
	for (int i = 0; i < M * K; ++i) h_A[i] = dis(gen);
	for (int i = 0; i < K * N; ++i) h_B[i] = dis(gen);

	std::cout << "Initialized matrices" << std::endl;

	// Perform CPU matrix multiplication for reference
	if(M < 8192) cpu_gemm(h_A, h_B, h_C_ref, M, N, K);

	std::cout << "Performed CPU matrix multiplication" << std::endl;

	// Allocate device memory
	__hip_bfloat16 *d_A, *d_B, *d_C;
	hipMalloc(&d_A, M*K*sizeof(__hip_bfloat16));
	hipMalloc(&d_B, K*N*sizeof(__hip_bfloat16));
	hipMalloc(&d_C, M*N*sizeof(__hip_bfloat16));

	// Check for CUDA errors
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
		// Optionally, you might want to exit the program or handle the error in some way
		return -1;
	}

	std::cout << "Allocated device memory" << std::endl;

	// Convert to __hip_bfloat16 and copy to device
	__hip_bfloat16 *h_A_bf16 = new __hip_bfloat16[M * K];
	__hip_bfloat16 *h_B_bf16 = new __hip_bfloat16[K * N];
	for (int i = 0; i < M * K; ++i) h_A_bf16[i] = __float2bfloat16(h_A[i]);
	for (int i = 0; i < K * N; ++i) h_B_bf16[i] = __float2bfloat16(h_B[i]);

	hipMemcpy(d_A, h_A_bf16, M*K*2, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B_bf16, K*N*2, hipMemcpyHostToDevice);

	std::cout << "Copied matrices to device" << std::endl;

	unsigned long mem_size = MAX_SHARED_MEMORY - 1024;
	hipFuncSetAttribute(reinterpret_cast<const void*>(prototype::pc<mmt>), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);

	// Launch kernel
	dim3 grid(mmt::grid(M, N, K));
	dim3 block(prototype::num_threads<mmt>);
	std::cout << "Launching warmup kernel with grid (" << grid.x << ", " << grid.y << "), block (" << block.x << ")\n";
	for(int i = 0; i < (NCU ? 0 : 2); i++) { // warmup
		inner_run<mmt>(d_A, d_B, d_C, M, N, K, grid, block);
	}

	// Start timing
	hipDeviceSynchronize();
	std::cout << "Launching kernel with grid (" << grid.x << ", " << grid.y << "), block (" << block.x << ")\n";
	auto start = std::chrono::high_resolution_clock::now();

	constexpr int ITERS = (NCU ? 1 : 10);
	for(int i = 0; i < ITERS; i++) {
		inner_run<mmt>(d_A, d_B, d_C, M, N, K, grid, block);
	}
	hipDeviceSynchronize();

	// End timing
	auto end = std::chrono::high_resolution_clock::now();

	// Calculate duration
	std::chrono::duration<double> diff = end - start;
	double useconds = diff.count() * 1e6 / ITERS;

	// Calculate TFLOPs
	double flops = double(2.0) * M * N * K; // 2 FLOPs per multiply-add
	double tflops = (flops / useconds) / 1e6;

	std::cout << "Avg Kernel execution time: " << useconds << " us\n";
	std::cout << "Achieved performance: " << tflops << " TFLOPs\n";
	
	// Check for CUDA errors
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
		// Optionally, you might want to exit the program or handle the error in some way
		return -1;
	}

	// Copy result back to host
	__hip_bfloat16 *h_C_bf16 = new __hip_bfloat16[M * N];
	hipMemcpy(h_C_bf16, d_C, M*N*2, hipMemcpyDeviceToHost);

	std::cout << "Copied result back to host" << std::endl;

	// Convert result back to float for comparison
	for (int i = 0; i < M * N; ++i) h_C[i] = __bfloat162float(h_C_bf16[i]);

	std::cout << "Converted result back to float" << std::endl;

	// Check result
	float max_error = 0.0f;
	int error_count = 0;
	for (int i = 0; i < M * N; ++i) {
		float error = std::abs(h_C[i] - h_C_ref[i]);
		if(error > 1.0) { // large because of bf16 vs fp32 numerics
			if(error_count < 20) std::cout << "Error at row " << i / N << " col " << i % N << ": " << h_C[i] << " != " << h_C_ref[i] << " (ref)" << std::endl;
			else if(error_count == 21) std::cout << "Too many errors to show them all.\n";
			error_count++;
		}
		max_error = std::max(max_error, error);
	}

	std::cout << "Max error: " << max_error << std::endl;
	std::cout << "Error count: " << error_count << std::endl;

	// Clean up
	delete[] h_A;
	delete[] h_B;
	delete[] h_C;
	delete[] h_C_ref;
	delete[] h_A_bf16;
	delete[] h_B_bf16;
	delete[] h_C_bf16;
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}

int main() {
	// int Cblocks = 22, Rblocks = 24;
	// int Cblocks192 = 20, Rblocks192 = 16;
	// run_benchmark<matmul_template<4>>(4096, 4096, 4096, Rblocks, Cblocks, Rblocks192, Cblocks192);
	// run_benchmark<matmul_template<8>>(4096, 4096, 4096, Rblocks, Cblocks, Rblocks192, Cblocks192);
	// run_benchmark<matmul_template<12>>(4096, 4096, 4096, Rblocks, Cblocks, Rblocks192, Cblocks192);
	int N;
	// N = 2048;
	// run_benchmark<matmul_template<2,4,8>>(N, N, N);
	// N = 3072;
	// run_benchmark<matmul_template<2,4,8>>(N, N, N);
	// run_benchmark<matmul_template<3,3,8>>(N, N, N);
	// N = 4096;
	// run_benchmark<matmul_template<2,4,8>>(N, N, N);
	// N = 6144;
	// run_benchmark<matmul_template<2,4,8>>(N, N, N);
	// run_benchmark<matmul_template<3,3,8>>(N, N, N);
	// N = 8192;
	// run_benchmark<matmul_template<2,4,8>>(N, N, N);
	// N = 12288;
	// run_benchmark<matmul_template<2,4,8>>(N, N, N);
	// run_benchmark<matmul_template<3,3,8>>(N, N, N);
	// N = 16384;
	// run_benchmark<matmul_template<2,4,8>>(N, N, N);
	// run_benchmark<matmul_template<2,4,12>>(N, N, N);
	// run_benchmark<matmul_template<3,3,12>>(192*12, 192*11, 8192);
	// run_benchmark<matmul_template<2,4,11>>(128*22, 256* 6, 8192);
	run_benchmark<matmul_template<3,3,12>>(192*22, 192*6*2, 4096);
	run_benchmark<matmul_template<3,3,12>>(192*22, 192*6*2, 8192);
	run_benchmark<matmul_template<3,3,12>>(192*22, 192*6*2, 16384);
	// run_benchmark<matmul_template<2,4,11>>(128*22*2, 256* 6*2, 8192);
	// run_benchmark<matmul_template<3,3,12>>(192*12*2, 192*11*2, 8192*2);
	// run_benchmark<matmul_template<2,4,11>>(128*22*2, 256* 6*2, 8192*2);
	return 0;
}