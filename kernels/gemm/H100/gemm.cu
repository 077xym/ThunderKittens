#include "hip/hip_runtime.h"
#include "kittens.cuh"
using namespace kittens;

using a_tile = st_bf<4,4>;
using b_tile = st_bf<4,16>;
using c_tile = st_bf<4,16>;

template<int _NUM_CONSUMER_WARPGROUPS>
struct producer_consumer_parameters {
    static constexpr int NUM_CONSUMER_WARPGROUPS = _NUM_CONSUMER_WARPGROUPS;
    static_assert(NUM_CONSUMER_WARPGROUPS >= 2 && NUM_CONSUMER_WARPGROUPS <= 6); // The register alloc is only set up for this range.
    static constexpr int NUM_CONSUMER_WARPS      = NUM_CONSUMER_WARPGROUPS * WARPGROUP_WARPS;
    static constexpr int NUM_WARPS               = NUM_CONSUMER_WARPS + WARPGROUP_WARPS; // producers, too
    static constexpr int NUM_THREADS             = NUM_WARPS * WARP_THREADS;
    static constexpr int NUM_PRODUCER_REG        = NUM_CONSUMER_WARPGROUPS == 2 ? 32 : 24;
    static constexpr int NUM_CONSUMER_REG        = 480/NUM_CONSUMER_WARPGROUPS-8; // valid up to 6 consumer warpgroups
};

struct globals {
    int n_blocks;
    const CUtensorMap* A_tma;
    const CUtensorMap* B_tma;
    CUtensorMap* C_tma;
    __host__ __device__ inline globals(int n_blocks, const CUtensorMap* A_tma, const CUtensorMap* B_tma, CUtensorMap* C_tma) :
        n_blocks(n_blocks), A_tma(A_tma), B_tma(B_tma), C_tma(C_tma) {}
};

template<int _NUM_CONSUMER_WARPGROUPS>
struct block { // the chunk of data that the producer and consumer are working on
    a_tile (&a_block)[_NUM_CONSUMER_WARPGROUPS];
    b_tile (&b_block);
    __device__ inline block(a_tile (&a_block)[_NUM_CONSUMER_WARPGROUPS], b_tile (&b_block)) : a_block(a_block), b_block(b_block) {}
};

struct producer_consumer {
    static constexpr int NUM_CONSUMER_WARPGROUPS = 2;
    using params = producer_consumer_parameters<NUM_CONSUMER_WARPGROUPS>;
    using block = block<NUM_CONSUMER_WARPGROUPS>;

    struct producer {
        struct state {
            int row_idx, col_idx; // persistent registers
        };
        __device__ static void setup(state &s, globals &g) { // setup and load the first iteration
            warpgroup::decrease_registers<params::NUM_PRODUCER_REG>(); // decrease registers for the producer warpgroup
            s.row_idx = blockIdx.x * NUM_CONSUMER_WARPGROUPS; // tiles vertical per block
            s.col_idx = blockIdx.y; // just 1 tile horizontal per block
        }
        __device__ static void load(state &s, block &b, globals &g, kittens::barrier &bar, int iter) { // barrier for the producer to load into
            if(warpgroup::warpid() == 0) {
                tma::expect_bytes(bar, size_bytes<a_tile>*NUM_CONSUMER_WARPGROUPS + size_bytes<b_tile>);
                #pragma unroll
                for(int i = 0; i < NUM_CONSUMER_WARPGROUPS; i++) {
                    tma::load_async(b.a_block[i], g.A_tma, bar, s.row_idx+i, iter);
                }
                tma::load_async(b.b_block, g.B_tma, bar, iter, s.col_idx);
            }
        }
        __device__ static void finish(state &s, globals &g) {}
    };

    struct consumer {
        struct state {
            rt_fl<1,c_tile::width> acc;
            c_tile &out_block;
            __host__ __device__ inline state(c_tile &out_block) : out_block(out_block) {}
        }; // persistent registers; none needed for this kernel.
        __device__ static void setup(state &s, globals &g) { // setup locals for before the first iteration
            warpgroup::increase_registers<params::NUM_CONSUMER_REG>();
            zero(s.acc);
        }
        __device__ static void compute(state &s, block &b, globals &g, int iter) {
            warpgroup::mma_fence(s.acc);
            warpgroup::mma_AB(s.acc, b.a_block[warpgroup::groupid()], b.b_block);
            warpgroup::mma_commit_group();
            warpgroup::mma_async_wait();
        }
        __device__ static void finish(state &s, globals &g) {
            warpgroup::store(s.out_block, s.acc);
            warpgroup::sync(); // writes to shared memory are now visible
            if(warpgroup::warpid() == 0) { // first warp stores
                tma::store_async(g.C_tma, s.out_block, blockIdx.x * NUM_CONSUMER_WARPGROUPS + warpgroup::groupid(), blockIdx.y);
                tma::store_commit_group();
            }
            tma::store_async_read_wait(); // this isn't really necessary, but it illustrates the principle.
            warpgroup::sync();
        }
    };
};

constexpr int PIPE_STAGES = 4;
__device__ inline int advance(int ring) { return (ring + 1) % PIPE_STAGES; }
__device__ inline int retreat(int ring) { return (ring + PIPE_STAGES-1) % PIPE_STAGES; }

// This is a producer+consumer copy kernel that demonstrates the use of TMA to implement a two-stage pipeline.
__global__ __launch_bounds__(producer_consumer::params::NUM_THREADS, 1)
void gpu_gemm(globals g) {
    using pc = producer_consumer;

    extern __shared__ int __shm[];
    shared_allocator alloc(&__shm[0]); // allocate shared memory
    a_tile (&a_smem) [PIPE_STAGES][producer_consumer::params::NUM_CONSUMER_WARPGROUPS] = alloc.allocate<a_tile, PIPE_STAGES, producer_consumer::params::NUM_CONSUMER_WARPGROUPS>();
    b_tile (&b_smem) [PIPE_STAGES]                                                     = alloc.allocate<b_tile, PIPE_STAGES>();
    c_tile (&c_smem) [producer_consumer::params::NUM_CONSUMER_WARPGROUPS]              = reinterpret_cast<c_tile(&)[producer_consumer::params::NUM_CONSUMER_WARPGROUPS]>(a_smem); // ovewrwrite at the end
    block<producer_consumer::params::NUM_CONSUMER_WARPGROUPS> blocks[] = {
        block(a_smem[0], b_smem[0]),
        block(a_smem[1], b_smem[1]),
        block(a_smem[2], b_smem[2]),
        block(a_smem[3], b_smem[3])
    };

    // Initialize barriers. This is constant for all two-stage producer-consumer kernels.
    __shared__ kittens::barrier producer_arrived[PIPE_STAGES], consumer_arrived[PIPE_STAGES];
    int ring = 0; // these are used to track the two-stage pipeline.
    if (warpid() < PIPE_STAGES) { // a single warp (in fact a single thread) does these.
        init_barrier(producer_arrived[warpid()], 0, 1); // needs to wait on just one memory transaction, each
        init_barrier(consumer_arrived[warpid()], pc::params::NUM_CONSUMER_WARPS, 0); // needs to wait on one thread from each consumer warp
    }

    __syncthreads(); // all warps must arrive here, confirming barrier initialization is visible to all threads.

    if(warpgroup::groupid() == pc::params::NUM_CONSUMER_WARPGROUPS) { // last warpgroup is a producer
        typename pc::producer::state s;
        pc::producer::setup(s, g);
        pc::producer::load(s, blocks[ring], g, producer_arrived[ring], 0); // load initial block
        if constexpr (PIPE_STAGES>2) pc::producer::load(s, blocks[advance(ring)], g, producer_arrived[advance(ring)], 1); // load second block for pipeline
        if constexpr (PIPE_STAGES>3) pc::producer::load(s, blocks[advance(advance(ring))], g, producer_arrived[advance(advance(ring))], 2); // load third block for pipeline
        for (int block_idx = PIPE_STAGES-1; block_idx < g.n_blocks; block_idx++, ring=advance(ring)) {
            int ring_load = retreat(ring); // maximally advanced, pipe_stages-1 times
            pc::producer::load(s, blocks[ring_load], g, producer_arrived[ring_load], block_idx);
            wait(consumer_arrived[ring], ((block_idx-(PIPE_STAGES-1))/PIPE_STAGES)%2); // phase changes at half the rate of the tic/toc
        }
        pc::producer::finish(s, g);
    }
    else { // other warpgroups are consumers
        typename pc::consumer::state s(c_smem[warpgroup::groupid()]);
        pc::consumer::setup(s, g);
        // Option 1: simple PC
        // for (int block_idx = 0; block_idx < g.n_blocks; block_idx++, ring=advance(ring)) {
        //     wait(producer_arrived[ring], (block_idx/PIPE_STAGES)%2); // wait for memory to arrive
        //     pc::consumer::compute(s, blocks[ring], g, block_idx);
        //     if(laneid() == 0) arrive(consumer_arrived[ring]); // overlap arrival for previous with this matmul
        // }
        // Option 2: hide barrier stuff during the wgmma's, which gives another ~20 TFLOPs
        wait(producer_arrived[ring], 0); // wait for initial memory to arrive
        warpgroup::mma_fence(s.acc);
        warpgroup::mma_AB(s.acc, blocks[ring].a_block[warpgroup::groupid()], blocks[ring].b_block); // launch first one, don't wait.
        warpgroup::mma_commit_group();
        ring = advance(ring);
        for (int block_idx = 1; block_idx < g.n_blocks; block_idx++, ring=advance(ring)) {
            wait(producer_arrived[ring], (block_idx/PIPE_STAGES)%2); // wait for next memory to arrive while we wait for tensor cores
            warpgroup::mma_async_wait(); // previous is finished
            warpgroup::mma_fence(s.acc);
            warpgroup::mma_AB(s.acc, blocks[ring].a_block[warpgroup::groupid()], blocks[ring].b_block);
            warpgroup::mma_commit_group();
            if(laneid() == 0) arrive(consumer_arrived[retreat(ring)]); // overlap arrival for previous with this matmul
        }
        warpgroup::mma_async_wait();
        if(laneid() == 0) arrive(consumer_arrived[retreat(ring)]); // final one finished
        // Common writeout
        pc::consumer::finish(s, g);
    }
}

#include <iostream>
#include <random>
#include <hip/hip_bf16.h>

#include <omp.h>
void cpu_gemm(float* a, float* b, float* c, int M, int N, int K) {
    #pragma omp parallel for collapse(2) // otherwise the CPU version takes for everrrrrr
    for (int i = 0; i < M; i++) {
        for (int j = 0; j < N; j++) {
            float sum = 0.0f;
            for (int k = 0; k < K; k++) {
                sum += a[i * K + k] * b[k * N + j];
            }
            c[i * N + j] = sum;
        }
    }
}

int main() {
    const int M = 4096, N = 4096, K = 4096;
    const size_t size_bytes = M * N * sizeof(float);
    const size_t size_bytes_bf16 = M * N * sizeof(__hip_bfloat16);

    // Allocate host memory
    float *h_A = new float[M * K];
    float *h_B = new float[K * N];
    float *h_C = new float[M * N];
    float *h_C_ref = new float[M * N];

    std::cout << "Allocated host memory" << std::endl;

    // Initialize random number generator
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<> dis(-1.0, 1.0);

    // Initialize matrices with random values
    for (int i = 0; i < M * K; ++i) h_A[i] = dis(gen);
    for (int i = 0; i < K * N; ++i) h_B[i] = dis(gen);

    std::cout << "Initialized matrices" << std::endl;

    // Perform CPU matrix multiplication for reference
    cpu_gemm(h_A, h_B, h_C_ref, M, N, K);

    std::cout << "Performed CPU matrix multiplication" << std::endl;

    // Allocate device memory
    __hip_bfloat16 *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size_bytes_bf16);
    hipMalloc(&d_B, size_bytes_bf16);
    hipMalloc(&d_C, size_bytes_bf16);

    std::cout << "Allocated device memory" << std::endl;

    std::cout << "a_tile::rows=" << a_tile::rows << " a_tile::cols=" << a_tile::cols << std::endl;
    std::cout << "b_tile::rows=" << b_tile::rows << " b_tile::cols=" << b_tile::cols << std::endl;
    std::cout << "c_tile::rows=" << c_tile::rows << " c_tile::cols=" << c_tile::cols << std::endl;
    CUtensorMap* tma_A_d = tma::allocate_and_create_tensor_map<a_tile>(d_A, M/a_tile::rows, K/a_tile::cols);
    CUtensorMap* tma_B_d = tma::allocate_and_create_tensor_map<b_tile>(d_B, K/b_tile::rows, N/b_tile::cols);
    CUtensorMap* tma_C_d = tma::allocate_and_create_tensor_map<c_tile>(d_C, M/c_tile::rows, N/c_tile::cols);

    std::cout << "Allocated TMA memory" << std::endl;

    // Convert to __hip_bfloat16 and copy to device
    __hip_bfloat16 *h_A_bf16 = new __hip_bfloat16[M * K];
    __hip_bfloat16 *h_B_bf16 = new __hip_bfloat16[K * N];
    for (int i = 0; i < M * K; ++i) h_A_bf16[i] = __float2bfloat16(h_A[i]);
    for (int i = 0; i < K * N; ++i) h_B_bf16[i] = __float2bfloat16(h_B[i]);

    hipMemcpy(d_A, h_A_bf16, size_bytes_bf16, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B_bf16, size_bytes_bf16, hipMemcpyHostToDevice);

    std::cout << "Copied matrices to device" << std::endl;

    unsigned long mem_size = 200000; // need to launch two blocks if possible.
    
    hipFuncSetAttribute(reinterpret_cast<const void*>(gpu_gemm), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);
    // Launch kernel
    dim3 grid(M / (c_tile::rows*producer_consumer::params::NUM_CONSUMER_WARPGROUPS), N / c_tile::cols); // rows, cols
    dim3 block(producer_consumer::params::NUM_THREADS);

    // Start timing
    hipDeviceSynchronize();
    std::cout << "Launching kernel" << std::endl;
    auto start = std::chrono::high_resolution_clock::now();

    constexpr int ITERS = 100;
    for(int i = 0; i < ITERS; i++) {
        gpu_gemm<<<grid, block, mem_size>>>(globals(K/a_tile::cols, tma_A_d, tma_B_d, tma_C_d));
    }
    hipDeviceSynchronize();

    // End timing
    auto end = std::chrono::high_resolution_clock::now();

    // Calculate duration
    std::chrono::duration<double> diff = end - start;
    double seconds = diff.count();

    // Calculate TFLOPs
    double flops = double(2.0) * M * N * K * ITERS; // 2 FLOPs per multiply-add
    double tflops = (flops / seconds) / 1e12;

    std::cout << "Launched kernel with grid (" << grid.x << ", " << grid.y << "), block (" << block.x << "), and " << K/a_tile::cols << " reduction block dimension\n";
    std::cout << "Kernel execution time: " << seconds << " seconds\n";
    std::cout << "Achieved performance: " << tflops << " TFLOPs\n";
    
    // Check for CUDA errors
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
        // Optionally, you might want to exit the program or handle the error in some way
        return -1;
    }

    // Copy result back to host
    __hip_bfloat16 *h_C_bf16 = new __hip_bfloat16[M * N];
    hipMemcpy(h_C_bf16, d_C, size_bytes_bf16, hipMemcpyDeviceToHost);

    std::cout << "Copied result back to host" << std::endl;

    // Convert result back to float for comparison
    for (int i = 0; i < M * N; ++i) h_C[i] = __bfloat162float(h_C_bf16[i]);

    std::cout << "Converted result back to float" << std::endl;

    // Check result
    float max_error = 0.0f;
    int error_count = 0;
    for (int i = 0; i < M * N; ++i) {
        float error = std::abs(h_C[i] - h_C_ref[i]);
        if(error > 1.0) { // large because of bf16 vs fp32 numerics
            if(error_count < 20) std::cout << "Error at row " << i / N << " col " << i % N << ": " << h_C[i] << " != " << h_C_ref[i] << " (ref)" << std::endl;
            else if(error_count == 21) std::cout << "Too many errors to show them all.\n";
            error_count++;
        }
        max_error = std::max(max_error, error);
    }

    std::cout << "Max error: " << max_error << std::endl;
    std::cout << "Error count: " << error_count << std::endl;

    // Clean up
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    delete[] h_C_ref;
    delete[] h_A_bf16;
    delete[] h_B_bf16;
    delete[] h_C_bf16;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return 0;
}
