#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"

using namespace kittens;
using namespace kittens::prototype;
template<int M_BLOCK=2, int N_BLOCK=4>
struct matmul_layout {
	using  base_tile      = st_bf<64, 64>;
	using  global_layout  = gl<bf16, 1, 1, -1, -1, base_tile>;
	struct globals        { global_layout A, B, C; };
	struct input_block    { base_tile a[M_BLOCK], b[N_BLOCK]; };
	struct finish_block   { base_tile c[M_BLOCK][N_BLOCK]; };
	struct producer_state { kittens::index coords; };
	struct consumer_state { kittens::index coords;
							            rt_fl<16, N_BLOCK*base_tile::cols> accum; };
};
template<int M_BLOCK=2, int N_BLOCK=4, int SUPER_M=12>
struct matmul_template {
	using layout    = matmul_layout<M_BLOCK, N_BLOCK>;
	using wide_tile = st_bf<64, 64*N_BLOCK>;
	static constexpr int NUM_CONSUMER_WARPS = M_BLOCK*4;
  // Helper functions
	__host__ static inline dim3 grid(int M, int N, int K) {
		return dim3(M*N/(M_BLOCK*N_BLOCK*layout::base_tile::num_elements));
	}
	__device__ static inline void get_coords(kittens::index &coords, typename layout::globals &g, int id) {
		int Rblocks = g.A.rows / (M_BLOCK*64), Cblocks = g.A.cols / (N_BLOCK*64);
		int super_rows = (Rblocks/SUPER_M)*SUPER_M,
        final_rows = Rblocks - super_rows,
        super_repeat = SUPER_M*Cblocks;
		if (blockIdx.x < super_rows * Cblocks)
			coords = { SUPER_M*(blockIdx.x/super_repeat) + blockIdx.x%SUPER_M,
					       (blockIdx.x%super_repeat)/SUPER_M };
		else {
			int remainder_id = blockIdx.x - super_rows*Cblocks;
			coords = { super_rows + (remainder_id%final_rows), remainder_id/final_rows };
		}
		coords = { iters(g), coords.r*M_BLOCK + id, coords.c*N_BLOCK };
	}
  // ThunderKittens template functions
	__device__ static inline int iters(typename layout::globals &g) { return g.A.cols/64; }
	struct producer {
		__device__ static void setup(producer_setup_args<layout> args) {
			warpgroup::producer_registers(); // decrease registers for producers
			get_coords(args.state.coords, args.globals, 0);
		}
		__device__ static void load(producer_load_args<layout> args) {
			if(warpgroup::warpid() == 0) {
				tma::expect(args.inputs_arrived, args.input);
				for(int i = 0; i < M_BLOCK; i++)
					tma::load_async(args.input.a[i], args.globals.A,
									        {args.state.coords.r+i, args.iter}, args.inputs_arrived);
				for(int i = 0; i < N_BLOCK; i++)
					tma::load_async(args.input.b[i], args.globals.B,
									        {args.iter, args.state.coords.c+i}, args.inputs_arrived);
				arrive(args.inputs_arrived, 3);
			}
		}
	};
	struct consumer {
		__device__ static void setup(consumer_setup_args<layout> args) {
			warpgroup::consumer_registers<NUM_CONSUMER_WARPS/4>(); // increase registers for consumers
			get_coords(args.state.coords, args.globals, warpgroup::groupid());
			zero(args.state.accum);
		}
		__device__ static void work(consumer_work_args<layout> args) {
			warpgroup::mma_AB(
				args.state.accum, // dest registers
				args.input.a[warpgroup::groupid()], // A matrix
				reinterpret_cast<wide_tile&>(args.input.b) // B matrix
			);
			warpgroup::mma_async_wait();
			if(warpgroup::laneid() == 0) arrive(args.inputs_finished, 4);
		}
		__device__ static void finish(consumer_finish_args<layout> args) {
			warpgroup::store(reinterpret_cast<wide_tile&>(args.finish.c[warpgroup::groupid()]), args.state.accum);
			warpgroup::sync();
			if(warpgroup::warpid() == 0) for(int i = 0; i < N_BLOCK; i++)
				tma::store_async(args.globals.C, args.finish.c[warpgroup::groupid()][i],
								         {args.state.coords.r, args.state.coords.c+i});
		}
	};
};


constexpr bool NCU = false;
#include <iostream>
#include <random>
#include <hip/hip_bf16.h>
#include <omp.h>

void cpu_gemm(float* a, float* b, float* c, int M, int N, int K) {
	#pragma omp parallel for collapse(2) // otherwise the CPU version takes for everrrrrr
	for (int i = 0; i < M; i++) {
		for (int j = 0; j < N; j++) {
			float sum = 0.0f;
			for (int k = 0; k < K; k++) {
				sum += a[i * K + k] * b[k * N + j];
			}
			c[i * N + j] = sum;
		}
	}
}

template<typename mmt, bool use_cache>
void inner_run(bf16 *d_A, bf16 *d_B, bf16 *d_C, int M, int N, int K, dim3 grid, dim3 block, kittens::tma::tma_cache & tma_cache) {
	using global_layout = typename mmt::layout::global_layout;
	using globals  = typename mmt::layout::globals;
	if constexpr (use_cache) {
		global_layout Ag{d_A, nullptr, nullptr, M, K, tma_cache};
		global_layout Bg{d_B, nullptr, nullptr, K, N, tma_cache};
		global_layout Cg{d_C, nullptr, nullptr, M, N, tma_cache};
		globals G{Ag, Bg, Cg};
		prototype::pc<mmt><<<grid, block, MAX_SHARED_MEMORY-1024>>>(G);
	} else {
		global_layout Ag{d_A, nullptr, nullptr, M, K};
		global_layout Bg{d_B, nullptr, nullptr, K, N};
		global_layout Cg{d_C, nullptr, nullptr, M, N};
		globals G{Ag, Bg, Cg};
		prototype::pc<mmt><<<grid, block, MAX_SHARED_MEMORY-1024>>>(G);
	}
	// hipDeviceSynchronize();
}

template<typename mmt, bool use_cache>
int run_benchmark(size_t M, size_t N, size_t K) {
	hipError_t cudaStatus;

	std::cout << "--------------------  M=" << M << " N=" << N << " K=" << K << "  --------------------  ";
	std::cout << (use_cache ? "USING TMA CACHE" : "NOT USING TMA CACHE") << std::endl;

	// Allocate host memory
	float *h_A = new float[M * K];
	float *h_B = new float[K * N];
	float *h_C = new float[M * N];
	float *h_C_ref = new float[M * N];

	std::cout << "Allocated host memory" << std::endl;

	// Initialize random number generator
	std::random_device rd;
	std::mt19937 gen(42);
	std::uniform_real_distribution<> dis(-0.5, 0.5);

	// Initialize matrices with random values
	for (int i = 0; i < M * K; ++i) h_A[i] = dis(gen);
	for (int i = 0; i < K * N; ++i) h_B[i] = dis(gen);

	std::cout << "Initialized matrices" << std::endl;

	// Perform CPU matrix multiplication for reference
	if(M < 8192) cpu_gemm(h_A, h_B, h_C_ref, M, N, K);

	std::cout << "Performed CPU matrix multiplication" << std::endl;

	// Allocate device memory
	__hip_bfloat16 *d_A, *d_B, *d_C;
	hipMalloc(&d_A, M*K*sizeof(__hip_bfloat16));
	hipMalloc(&d_B, K*N*sizeof(__hip_bfloat16));
	hipMalloc(&d_C, M*N*sizeof(__hip_bfloat16));

	// Check for CUDA errors
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
		// Optionally, you might want to exit the program or handle the error in some way
		return -1;
	}

	std::cout << "Allocated device memory" << std::endl;

	// Convert to __hip_bfloat16 and copy to device
	__hip_bfloat16 *h_A_bf16 = new __hip_bfloat16[M * K];
	__hip_bfloat16 *h_B_bf16 = new __hip_bfloat16[K * N];
	for (int i = 0; i < M * K; ++i) h_A_bf16[i] = __float2bfloat16(h_A[i]);
	for (int i = 0; i < K * N; ++i) h_B_bf16[i] = __float2bfloat16(h_B[i]);

	hipMemcpy(d_A, h_A_bf16, M*K*2, hipMemcpyHostToDevice);
	hipMemcpy(d_B, h_B_bf16, K*N*2, hipMemcpyHostToDevice);

	std::cout << "Copied matrices to device" << std::endl;

	unsigned long mem_size = MAX_SHARED_MEMORY - 1024;
	hipFuncSetAttribute(reinterpret_cast<const void*>(prototype::pc<mmt>), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);

	kittens::tma::tma_cache tma_cache;

	// Launch kernel
	dim3 grid(mmt::grid(M, N, K));
	dim3 block(prototype::num_threads<mmt>);
	std::cout << "Launching warmup kernel with grid (" << grid.x << ", " << grid.y << "), block (" << block.x << ")\n";
	for(int i = 0; i < (NCU ? 0 : 2); i++) { // warmup
		inner_run<mmt, use_cache>(d_A, d_B, d_C, M, N, K, grid, block, tma_cache);
	}

	// Start timing
	hipDeviceSynchronize();
	std::cout << "Launching kernel with grid (" << grid.x << ", " << grid.y << "), block (" << block.x << ")\n";
	auto start = std::chrono::high_resolution_clock::now();

	constexpr int ITERS = (NCU ? 1 : 10);
	for(int i = 0; i < ITERS; i++) {
		inner_run<mmt, use_cache>(d_A, d_B, d_C, M, N, K, grid, block, tma_cache);
	}
	hipDeviceSynchronize();

	// End timing
	auto end = std::chrono::high_resolution_clock::now();

	// Calculate duration
	std::chrono::duration<double> diff = end - start;
	double useconds = diff.count() * 1e6 / ITERS;

	// Calculate TFLOPs
	double flops = double(2.0) * M * N * K; // 2 FLOPs per multiply-add
	double tflops = (flops / useconds) / 1e6;

	std::cout << "Avg Kernel execution time: " << useconds << " us\n";
	std::cout << "Achieved performance: " << tflops << " TFLOPs\n";
	
	// Check for CUDA errors
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		std::cerr << "CUDA error: " << hipGetErrorString(cudaStatus) << std::endl;
		// Optionally, you might want to exit the program or handle the error in some way
		return -1;
	}

	// Copy result back to host
	__hip_bfloat16 *h_C_bf16 = new __hip_bfloat16[M * N];
	hipMemcpy(h_C_bf16, d_C, M*N*2, hipMemcpyDeviceToHost);

	std::cout << "Copied result back to host" << std::endl;

	// Convert result back to float for comparison
	for (int i = 0; i < M * N; ++i) h_C[i] = __bfloat162float(h_C_bf16[i]);

	std::cout << "Converted result back to float" << std::endl;

	// Check result
	float max_error = 0.0f;
	int error_count = 0;
	for (int i = 0; i < M * N; ++i) {
		float error = std::abs(h_C[i] - h_C_ref[i]);
		if(error > 1.0) { // large because of bf16 vs fp32 numerics
			if(error_count < 20) std::cout << "Error at row " << i / N << " col " << i % N << ": " << h_C[i] << " != " << h_C_ref[i] << " (ref)" << std::endl;
			else if(error_count == 21) std::cout << "Too many errors to show them all.\n";
			error_count++;
		}
		max_error = std::max(max_error, error);
	}

	std::cout << "Max error: " << max_error << std::endl;
	std::cout << "Error count: " << error_count << std::endl;

	// Clean up
	delete[] h_A;
	delete[] h_B;
	delete[] h_C;
	delete[] h_C_ref;
	delete[] h_A_bf16;
	delete[] h_B_bf16;
	delete[] h_C_bf16;
	// Ag.cleanup();
	// Bg.cleanup();
	// Cg.cleanup();
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);

	return 0;
}

int main() {
	// int Cblocks = 22, Rblocks = 24;
	// int Cblocks192 = 20, Rblocks192 = 16;
	// run_benchmark<matmul_template<4>>(4096, 4096, 4096, Rblocks, Cblocks, Rblocks192, Cblocks192);
	// run_benchmark<matmul_template<8>>(4096, 4096, 4096, Rblocks, Cblocks, Rblocks192, Cblocks192);
	// run_benchmark<matmul_template<12>>(4096, 4096, 4096, Rblocks, Cblocks, Rblocks192, Cblocks192);
	int N;
	// N = 2048;
	// run_benchmark<matmul_template_128_256<8>>(N, N, N, N/128, N/256, N/128, N/256, dim3(N*N/(128*256)));
	// N = 3072;
	// run_benchmark<matmul_template_192_192<8>>(N, N, N, N/192, N/192, N/192, N/192, dim3(N*N/(192*192)));
	// run_benchmark<matmul_template_128_256<8>>(N, N, N, N/128, N/256, N/128, N/256, dim3(N*N/(128*256)));
	N = 4096;
	// run_benchmark<matmul_template_192_192<8>>(N, N, N, N/192, N/192, N/192, N/192, dim3(N*N/(192*192)));
	run_benchmark<matmul_template<2,4,8>, false>(N, N, N);
	run_benchmark<matmul_template<2,4,8>, true>(N, N, N);
	// N = 6144;
	// run_benchmark<matmul_template_192_192<8>>(N, N, N, N/192, N/192, N/192, N/192, dim3(N*N/(192*192)));
	// run_benchmark<matmul_template_128_256<8>>(N, N, N, N/128, N/256, N/128, N/256, dim3(N*N/(128*256)));
	// N = 8192;
	// run_benchmark<matmul_template_128_256<8>>(N, N, N, N/128, N/256, N/128, N/256, dim3(N*N/(128*256)));
	// N = 9216;
	// run_benchmark<matmul_template_192_192<8>>(N, N, N, N/192, N/192, N/192, N/192, dim3(N*N/(192*192)));
	// run_benchmark<matmul_template_128_256<8>>(N, N, N, N/128, N/256, N/128, N/256, dim3(N*N/(128*256)));
	// N = 12288;
	// run_benchmark<matmul_template_192_192<8>>(N, N, N, N/192, N/192, N/192, N/192, dim3(N*N/(192*192)));
	// run_benchmark<matmul_template_128_256<8>>(N, N, N, N/128, N/256, N/128, N/256, dim3(N*N/(128*256)));
	N = 16384;
	run_benchmark<matmul_template<2,4,12>, false>(N, N, N);
	run_benchmark<matmul_template<2,4,12>, true>(N, N, N);
	return 0;
}