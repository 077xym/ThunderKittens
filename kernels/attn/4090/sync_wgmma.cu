#include "hip/hip_runtime.h"
#include "kittens.cuh"

 // this kernel is more of an example kernel to show some TK programming models, rather than a kernel we think you should put into production, though it is pretty fast!

#define NUM_WORKERS 1 // This kernel uses 16 workers in parallel per block, to help issue instructions more quickly.
#define NUM_WARPS   (NUM_WORKERS*4) // This kernel uses 16 workers in parallel per block, to help issue instructions more quickly.

using namespace kittens;

using q_tile = st_bf<4,4>;
using kv_tile = st_bf<4,4>;
using global_layout = gl<bf16, -1, -1, -1, 64>; // B, H, N specified at runtime, D=64 known at compile time for this kernel

__launch_bounds__(NUM_WARPS*32, 1)
__global__ void attend_ker64(global_layout Qg, global_layout Kg, global_layout Vg, global_layout Og) {

    const int N = Qg.rows; // sequence length
    auto workerid = warpgroup::groupid(); // which worker am I?
    const int batch = blockIdx.z;
    const int head  = blockIdx.y;
    const int q_seq = (blockIdx.x * NUM_WORKERS + workerid);

    extern __shared__ alignment_dummy __shm[]; // this is the CUDA shared memory
    shared_allocator al((int*)&__shm[0]);
    
    // K and V live in shared memory -- this is about all that will fit.

    q_tile  (&q_smem)[NUM_WORKERS] = al.allocate<q_tile, NUM_WORKERS>();
    kv_tile (&k_smem)[NUM_WORKERS] = al.allocate<kv_tile, NUM_WORKERS>();
    kv_tile (&v_smem)[NUM_WORKERS] = al.allocate<kv_tile, NUM_WORKERS>();

    // Initialize all of the register tiles.
    rt_fl<1,kv_tile::height> att_block;
    rt_bf<1,kv_tile::height> att_block_mma;
    rt_fl<1,4> o_reg;
    rt_fl<1,kv_tile::height>::col_vec max_vec_last, max_vec; // these are column vectors for the attention block
    rt_fl<1,kv_tile::height>::col_vec norm_vec_last, norm_vec; // these are column vectors for the attention block

    // each warp loads its own Q tile of 16x64, and then multiplies by 1/sqrt(d)
    if (q_seq*q_tile::rows < N) warpgroup::load(q_smem[workerid], Qg, {batch, head, q_seq, 0});

    // zero flash attention L, M, and O registers.
    neg_infty(max_vec); // zero registers for the Q chunk
    zero(norm_vec);
    zero(o_reg);
    __syncthreads();

    warpgroup::mul(q_smem[workerid], q_smem[workerid], __float2bfloat16(0.125f)); // temperature adjustment
    
    const int kv_blocks = (N + NUM_WORKERS*kv_tile::rows - 1) / (NUM_WORKERS*kv_tile::rows);

    // iterate over k, v for these q's that have been loaded
    for(auto kv_idx = 0; kv_idx < kv_blocks; kv_idx++) {

        // each warp loads its own chunk of k, v into shared memory
        int load_idx = kv_idx*NUM_WORKERS + workerid;
        if (load_idx*kv_tile::rows < N) {
            warpgroup::load(k_smem[workerid], Kg, {batch, head, load_idx, 0});
            warpgroup::load(v_smem[workerid], Vg, {batch, head, load_idx, 0});
        }
        __syncthreads(); // we need to make sure all memory is loaded before we can begin the compute phase

        // now each warp goes through all of the subtiles, loads them, and then does the flash attention internal alg.
        int max_subtile = min(NUM_WORKERS, N/kv_tile::rows - kv_idx*NUM_WORKERS);
        for(int subtile = 0; subtile < max_subtile; subtile++) {

            warpgroup::mm_ABt(att_block, q_smem[workerid], k_smem[subtile]);
            warpgroup::mma_async_wait();

            copy(norm_vec_last, norm_vec);
            copy(max_vec_last,  max_vec);
            
            row_max(max_vec, att_block, max_vec); // accumulate onto the max_vec
            sub_row(att_block, att_block, max_vec); // subtract max from attention -- now all <=0
            exp(att_block, att_block); // exponentiate the block in-place.

            sub(max_vec_last, max_vec_last, max_vec); // subtract new max from old max to find the new normalization.
            exp(max_vec_last, max_vec_last); // exponentiate this vector -- this is what we need to normalize by.
            mul(norm_vec, norm_vec, max_vec_last); // and the norm vec is now normalized.

            row_sum(norm_vec, att_block, norm_vec); // accumulate the new attention block onto the now-rescaled norm_vec
            div_row(att_block, att_block, norm_vec); // now the attention block is correctly normalized

            mul(norm_vec_last, norm_vec_last, max_vec_last); // normalize the previous norm vec according to the new max
            div(norm_vec_last, norm_vec_last, norm_vec); // normalize the previous norm vec according to the new norm

            copy(att_block_mma, att_block); // convert to bf16 for mma_AB
            mul_row(o_reg, o_reg, norm_vec_last); // normalize o_reg in advance of mma_AB'ing onto it

            warpgroup::mma_AB(o_reg, att_block_mma, v_smem[subtile]); // mfma onto o_reg with the local attention@V matmul.
            warpgroup::mma_async_wait();
        }
        __syncthreads(); // we need to make sure all warps are done before we can start loading the next kv chunk
    }

    if (q_seq*q_tile::rows < N) warpgroup::store(Og, o_reg, {batch, head, q_seq, 0}); // write out o.
}

#include "harness.impl"
