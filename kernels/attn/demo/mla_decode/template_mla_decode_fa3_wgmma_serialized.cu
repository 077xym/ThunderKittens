#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"
#include "pyutils/pyutils.cuh"

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::interpreter;

static constexpr int QK_D = 576, VO_D = 512, VO_Dd2 = VO_D/2, NUM_ROWS = 32, PAGE_SIZE = 256, ITERS_PER_PAGE = PAGE_SIZE / NUM_ROWS;
using q_tile              = st_bf<64, QK_D>;
using q_global            = kittens::gl<bf16, -1, -1, -1, QK_D, q_tile>; // B * R * H * D_QK
using cache_tile          = st_bf<NUM_ROWS, QK_D>; using v_tile = st_bf<NUM_ROWS, VO_Dd2>; // we need the v_tile for later
using cache_global        = kittens::gl<bf16, 1, -1, PAGE_SIZE, QK_D, cache_tile>; // 1 * #page * pagesize * QK_D
using instructions_global = kittens::gl<int, 1, -1, -1, 16>;
using table_global        = kittens::gl<int, 1, 1, -1, -1>; // B * (max # pages)
using o_tile              = st_bf<64, VO_D>;
using o_tile_fl           = st_fl<16, VO_D>;
using o_global            = kittens::gl<bf16, -1, -1, -1, VO_D, st_bf<16, VO_Dd2>, st_bf<16, VO_D/8>>; // B * NEWTOKENS * H * D_VO

using o_scratch_global    = kittens::gl<float, -1, -1, 16, VO_D, st_fl<16, VO_D/8>, st_fl<16,256>>; // For partial O's
using lvec_scratch_global = kittens::gl<float,  1, -1, -1, 16, sv_fl<16>>; // For partial O's
using semaphore_global    = kittens::gl<int,   1,  1,  -1, -1>;            // 1 * 1 * uid * NEWTOKENS

struct config {
    struct globals {
        instructions_global instructions;
        q_global Q;
        cache_global Cache;
        table_global Table;
        o_global O;
        o_scratch_global O_scratch;
        lvec_scratch_global Lvec_scratch;
        semaphore_global semaphore;
        const float Softmax_scale;
        int dynamic_shared_memory() { return 226000; }
        dim3 grid()  { return dim3(132); } //dim3(Q.batch * ((Q.depth + 3) / 4)); }
        dim3 block() { return dim3((8+4)*WARP_THREADS); }
    };
};

struct location {
    int batch_idx; // batch_idx >=0, otherwise it's the negative index, minus one, into scratch
    int seq_idx;
};
struct partial_layout {
    using globals = config::globals;
    struct input_block { cache_tile c; };
    struct scratch_block { q_tile q; st_bf<64, cache_tile::rows> att_block; sv_fl<64> max_vec, norm_vec; };
    struct finish_block { st_fl<16, 256> o[4][2]; sv_fl<16> lvec[4]; };
    struct common_state {
        int uid;
        location dst;
        int q_batch_idx;
        int q_seq_idx;
        int length;
    };
    struct consumer_state {
        col_vec<rt_fl<16, cache_tile::rows>> max_vec, norm_vec;
        rt_fl<16, VO_Dd2> o;
    };
};
struct partial_template {
    using config = config;
    using layout = partial_layout;
    static constexpr int opcode = 1;
    static constexpr int INPUT_PIPE_STAGES = 3;
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        args.common.uid         =  args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 1}];
        args.common.dst         = {args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 2}],
                                   args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 3}]};
        args.common.q_batch_idx =  args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 4}];
        args.common.q_seq_idx   =  args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 5}];
        args.common.length      =  args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 6}];
        args.num_iters          = (args.common.length + NUM_ROWS - 1) / NUM_ROWS;
        args.common.length -= (args.globals.Q.depth - (args.common.q_seq_idx + warpgroup::warpid()) - 1); // adjust for the causal mask
    }
    struct producer {
        __device__ static inline void setup(producer_setup_args<layout> args) {}
        __device__ static inline void load(producer_load_args<layout> args) {
            if(warpgroup::warpid() == 0) {
                int global_load_idx = args.iter % ITERS_PER_PAGE;
                int next_page_id = args.globals.Table[coord<>{args.common.uid, args.iter / ITERS_PER_PAGE}];
                // next page we need to load?
                tma::expect(args.inputs_arrived, args.input.c);
                // cache shape is 1, # pages, page size, QK_D
                tma::load_async(args.input.c, args.globals.Cache, {0, next_page_id, global_load_idx, 0}, args.inputs_arrived);
            }
            else if(laneid() == 0) arrive(args.inputs_arrived);
            warpgroup::sync(5);
        }
    };
    struct consumer {
        __device__ static inline void setup(consumer_setup_args<layout> args) {
            zero(args.state.norm_vec);
            neg_infty(args.state.max_vec);
            zero(args.state.o);
            auto q_st = subtile_inplace<16, QK_D/2>(args.scratch.q, {warpgroup::warpid(), warpgroup::groupid()});
            load_async(q_st, args.globals.Q, {args.common.q_batch_idx, args.common.q_seq_idx + warpgroup::warpid(), 0, warpgroup::groupid()});
            load_async_wait();
            group<8>::sync(10);
        }
        template<bool do_right_fill> __device__ static inline void internal_compute(consumer_compute_args<layout> args) {
            // 1.44269504089f is from exp2
            const float SOFTMAX_TEMPERATURE = args.globals.Softmax_scale * 1.44269504089f;

            col_vec<rt_fl<16, cache_tile::rows>> local_max_vec, local_norm_vec;
            col_vec<rt_fl<16, cache_tile::rows>> max_vec_last_scaled, max_vec_scaled;

            copy(local_max_vec,  args.state.max_vec);
            copy(local_norm_vec, args.state.norm_vec);
 
            // if warpgroupid is 0, is responsible for QK matmuls and softmax, too. 
            if(warpgroup::groupid() == 0) {
                // launch the QK matmul.
                // A = Q @ K.T
                rt_fl<16, cache_tile::rows> att_block_fp32;
                warpgroup::mm_ABt(att_block_fp32, args.scratch.q, args.input.c);

                if(args.iter > 0) {
                    group<8>::sync(0);

                    warpgroup::load(max_vec_last_scaled, args.scratch.max_vec);
                    mul_row(args.state.o, args.state.o, max_vec_last_scaled); // normalize o_reg before mma

                    // O += A @ V
                    auto (&v_smem)[2] = reinterpret_cast<v_tile(&)[2]>(args.prev_input.c);
                    warpgroup::mma_AB(args.state.o, args.scratch.att_block, v_smem[warpgroup::groupid()]);
                }

                mul(max_vec_last_scaled, local_max_vec, SOFTMAX_TEMPERATURE);

                // wait on QK matmul, and do the softmax

                warpgroup::mma_async_wait<1>();
                // softmax
                if constexpr (do_right_fill) { // need to mask out a bunch of entries in the last page
                    const int length = args.common.length - args.iter*NUM_ROWS;
                    right_fill(att_block_fp32, att_block_fp32, length, base_types::constants<float>::neg_infty());
                }

                row_max(local_max_vec, att_block_fp32, local_max_vec);
                mul(max_vec_scaled, local_max_vec, SOFTMAX_TEMPERATURE);

                mul(att_block_fp32, att_block_fp32, SOFTMAX_TEMPERATURE);
                sub_row(att_block_fp32, att_block_fp32, max_vec_scaled);
                
                exp2(att_block_fp32, att_block_fp32);
                
                sub(max_vec_last_scaled, max_vec_last_scaled, max_vec_scaled);
                exp2(max_vec_last_scaled, max_vec_last_scaled);
                
                mul(local_norm_vec, local_norm_vec, max_vec_last_scaled);
                row_sum(local_norm_vec, att_block_fp32, local_norm_vec);

                // store attention up
                warpgroup::store(args.scratch.max_vec, max_vec_last_scaled);
                warpgroup::store(args.scratch.att_block, att_block_fp32);

                warpgroup::mma_async_wait(); // Need to make sure O is ready to use at this point
                if(warpgroup::laneid() == 0) arrive(args.prev_inputs_finished, WARPGROUP_WARPS); // done!
            }
            // Otherwise, just do the O matmul
            else if(args.iter > 0) {
                group<8>::sync(0);

                warpgroup::load(max_vec_last_scaled, args.scratch.max_vec);
                mul_row(args.state.o, args.state.o, max_vec_last_scaled); // normalize o_reg before mma

                // O += A @ V
                auto (&v_smem)[2] = reinterpret_cast<v_tile(&)[2]>(args.prev_input.c);
                warpgroup::mma_AB(args.state.o, args.scratch.att_block, v_smem[warpgroup::groupid()]);
                warpgroup::mma_async_wait(); // Need to make sure O is ready to use at this point
                if(warpgroup::laneid() == 0) arrive(args.prev_inputs_finished, WARPGROUP_WARPS); // done!
            }

            // If last iter, need to do more.
            if(args.iter == args.num_iters-1) {
                group<8>::sync(0);

                warpgroup::load(max_vec_last_scaled, args.scratch.max_vec);
                mul_row(args.state.o, args.state.o, max_vec_last_scaled); // normalize o_reg before mma

                // O += A @ V
                auto (&v_smem)[2] = reinterpret_cast<v_tile(&)[2]>(args.input.c);
                warpgroup::mma_AB(args.state.o, args.scratch.att_block, v_smem[warpgroup::groupid()]);
                warpgroup::mma_async_wait(); // Need to make sure O is ready to use at this point
                if(warpgroup::laneid() == 0) arrive(args.inputs_finished, WARPGROUP_WARPS); // done!
            }

            copy(args.state.max_vec, local_max_vec);
            copy(args.state.norm_vec, local_norm_vec);
        }
        __device__ static inline void compute(consumer_compute_args<layout> args) {
            if(args.iter >= args.num_iters-2) internal_compute<true>(args);
            else internal_compute<false>(args);
        }
        __device__ static inline void finish(consumer_finish_args<layout> args) {
            col_vec<rt_fl<16, cache_tile::rows>> local_max_vec, local_norm_vec;

            copy(local_norm_vec, args.state.norm_vec);
            copy(local_max_vec, args.state.max_vec);

            if (warpgroup::groupid() == 0) warpgroup::store(args.scratch.norm_vec, local_norm_vec);
            group<8>::sync(10);
            if(warpgroup::groupid() == 1) warpgroup::load(local_norm_vec, args.scratch.norm_vec);
            div_row(args.state.o, args.state.o, local_norm_vec);

            if(args.common.dst.batch_idx >= 0) { // batch is meaningful
                auto &o_smem = reinterpret_cast<st_bf<16, VO_Dd2>&>(args.finish.o[warpgroup::warpid()][warpgroup::groupid()]);
                store(o_smem, args.state.o);
                __syncwarp();
                tma::store_async(args.globals.O, o_smem, {args.common.dst.batch_idx, args.common.dst.seq_idx+warpgroup::warpid(), 0, warpgroup::groupid()});
            }
            else { // write out directly to O scratch, without going through smem
                if(warpgroup::groupid() == 0) {
                    mul(local_max_vec, local_max_vec, args.globals.Softmax_scale * 1.44269504089f);
                    log2(local_norm_vec, local_norm_vec);
                    add(local_norm_vec, local_norm_vec, local_max_vec); // l_vec = log2(norm_vec) + max_vec
                    store(args.finish.lvec[warpgroup::warpid()], local_norm_vec);
                    __syncwarp();
                    tma::store_async(args.globals.Lvec_scratch, args.finish.lvec[warpgroup::warpid()], {-args.common.dst.batch_idx-1, args.common.dst.seq_idx+warpgroup::warpid(), 0});
                }

                store(args.finish.o[warpgroup::warpid()][warpgroup::groupid()], args.state.o);
                __syncwarp();
                tma::store_async(args.globals.O_scratch, args.finish.o[warpgroup::warpid()][warpgroup::groupid()], {-args.common.dst.batch_idx-1, args.common.dst.seq_idx+warpgroup::warpid(), 0, warpgroup::groupid()});
            }
            tma::store_async_wait(); // not just read wait
            group<8>::sync(10);
            if(args.common.dst.batch_idx < 0) {
                asm volatile("fence.sc.sys;");
                if(group<8>::laneid() < 4 && args.common.dst.seq_idx + group<8>::laneid() < args.globals.O_scratch.depth) {
                    args.globals.semaphore[{-args.common.dst.batch_idx-1, args.common.dst.seq_idx + group<8>::laneid()}] = 1;
                }
            }
            if(warpgroup::laneid() == 0) arrive(args.finish_finished, WARPGROUP_WARPS); // done!
        }
    };
};
struct reduction_layout {
    using globals = config::globals;
    struct input_block   { st_fl<16, VO_D/8> o[8]; sv_fl<16> lvec; sv_fl<16> padding[15]; };
    struct scratch_block { st_fl<16, VO_D/8> o[8]; sv_fl<16> lvec; }; // used both for setup load and finish store
    struct common_state {
        int uid;
        // int num_iters; // same as the number of active load_uid's, marked here for instruction clarity but we just use args.num_iters instead.
        location dst; // again, negative batch means we're writing to O scratch, seq_idx is consistent
        int src_uid;
    };
    struct producer_state {
        int load_uid[10];
    };
    struct consumer_state {
        rt_fl<16, VO_D/8> o;
        col_vec<rt_fl<16, cache_tile::rows>> lvec;
    };
};
struct reduction_template {
    using config = config;
    using layout = reduction_layout;
    static constexpr int opcode = 2;
    static constexpr int INPUT_PIPE_STAGES = 4;
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        args.common.uid     =  args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 1}];
        args.num_iters      =  args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 2}];
        args.common.dst     = {args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 3}],
                               args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 4}]};
        args.common.src_uid =  args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 5}];
    }
    struct producer {
        __device__ static inline void setup(producer_setup_args<layout> args) {
            #pragma unroll
            for(int i = 0; i < 10; i++) {
                args.state.load_uid[i] = args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 5+i}];
            }
        }
        __device__ static inline void load(producer_load_args<layout> args) {
            if(warpgroup::warpid() == args.iter%4) {
                // spinloop until we're ready
                int load_uid = args.state.load_uid[args.iter];
                if(laneid() == 0) while(*(volatile int*)&args.globals.semaphore[{load_uid}] == 0) {}
                __syncwarp();
                // next page we need to load?
                tma::expect(args.inputs_arrived, args.input.o, args.input.lvec);
                #pragma unroll
                for(int i = 0; i < 8; i++) {
                    tma::load_async(args.input.o[i], args.globals.O_scratch, {load_uid, args.common.dst.seq_idx, 0, i}, args.inputs_arrived);
                }
                tma::load_async(args.input.lvec, args.globals.Lvec_scratch, {load_uid, args.common.dst.seq_idx, 0}, args.inputs_arrived);
                if(laneid() == 0) arrive(args.inputs_arrived, 3);
            }
        }
    };
    struct consumer {
        __device__ static inline void setup(consumer_setup_args<layout> args) {
            // If we are doing a reduction, we need to spinloop until we have confirmation that all the partial results have been written out.
            if(threadIdx.x == 0) { // easier to have a single thread spin
                while(*(volatile int*)&args.globals.semaphore[{args.common.src_uid}] == 0) {} // note volatile, L1 is not guaranteed to be coherent.
            }
            group<8>::sync(11); // all warps must sync here.
            load_async(args.scratch.o[group<8>::warpid()], args.globals.O_scratch, {args.common.src_uid, args.common.dst.seq_idx, 0, group<8>::warpid()});
            if(warpid() == 0) {
                load_async(args.scratch.lvec, args.globals.Lvec_scratch, {args.common.src_uid, args.common.dst.seq_idx, 0});
            }
            load_async_wait();
            __syncwarp();
            load(args.state.o, args.scratch.o[group<8>::warpid()]);
            group<8>::sync(11); // all warps must sync here.
            load(args.state.lvec, args.scratch.lvec);
        }
        __device__ static inline void compute(consumer_compute_args<layout> args) {
            col_vec<rt_fl<16, cache_tile::rows>> lvec, max_lvec, sum_lvec;
            rt_fl<16, VO_D / 8> o;
            load(o, args.input.o[group<8>::warpid()]);
            load(lvec, args.input.lvec);
            __syncwarp();
            if(laneid() == 0) arrive(args.inputs_finished); // done!
            max(max_lvec, args.state.lvec, lvec);
            sub(args.state.lvec, args.state.lvec, max_lvec);
            sub(lvec, lvec, max_lvec);
            exp2(args.state.lvec, args.state.lvec);
            exp2(lvec, lvec);
            add(sum_lvec, args.state.lvec, lvec);
            div(args.state.lvec, args.state.lvec, sum_lvec);
            div(lvec, lvec, sum_lvec);
            mul_row(args.state.o, args.state.o, args.state.lvec);
            mul_row(o, o, lvec);
            add(args.state.o, args.state.o, o);
            log2(sum_lvec, sum_lvec);
            add(args.state.lvec, sum_lvec, max_lvec);
        }
        __device__ static inline void finish(consumer_finish_args<layout> args) {
            if(args.common.dst.batch_idx >= 0) {
                auto &o_smem = reinterpret_cast<st_bf<16, VO_D/8>&>(args.scratch.o[group<8>::warpid()]);
                store(o_smem, args.state.o);
                __syncwarp();
                tma::store_async(args.globals.O, o_smem, {args.common.dst.batch_idx, args.common.dst.seq_idx, 0, group<8>::warpid()});
            }
            else {
                store(args.scratch.o[group<8>::warpid()], args.state.o);
                if(group<8>::warpid() == 0) store(args.scratch.lvec, args.state.lvec);
                __syncwarp();
                tma::store_async(args.globals.O_scratch, args.scratch.o[group<8>::warpid()], {-args.common.dst.batch_idx-1, args.common.dst.seq_idx, 0, group<8>::warpid()});
                if(group<8>::warpid() == 0) tma::store_async(args.globals.Lvec_scratch, args.scratch.lvec, {-args.common.dst.batch_idx-1, args.common.dst.seq_idx, 0});
            }
            tma::store_async_wait();
            group<8>::sync(11);
            // Increment the semaphore for the next stage, if this is not the last one.
            if(args.common.dst.batch_idx < 0) {
                asm volatile("fence.sc.sys;");
                if(group<8>::laneid() == 0) {
                    args.globals.semaphore[{-args.common.dst.batch_idx-1, args.common.dst.seq_idx}] = 1;
                }
            }
            if(warpgroup::laneid() == 0) arrive(args.finish_finished, WARPGROUP_WARPS); // done!
        }
    };
};
struct dummy_layout {
    using globals = config::globals;
    struct input_block {};
};
struct dummy_template {
    using config = config;
    using layout = dummy_layout;
    static constexpr int opcode = 3;
    static constexpr int INPUT_PIPE_STAGES = 1;
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        // if(blockIdx.x == 0 && threadIdx.x == 0) {
        //     printf("task_iter: %d, num_iters: %d\n", args.task_iter, args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 1}]);
        // }
        args.num_iters = args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 1}];
    }
    struct producer {
        __device__ static inline void setup(producer_setup_args<layout> args) {}
        __device__ static inline void load(producer_load_args<layout> args) {}
    };
    struct consumer {
        __device__ static inline void setup(consumer_setup_args<layout> args) {}
        __device__ static inline void compute(consumer_compute_args<layout> args) {}
        __device__ static inline void finish(consumer_finish_args<layout> args) {}
    };
};

struct dummy_globals {
    dim3 grid() { return dim3(132); }
    dim3 block() { return dim3(1024); }
    float f;
};
__global__ void dummy_kernel(__grid_constant__ const dummy_globals globals) {}

PYBIND11_MODULE(mla_decode, m) {
    m.doc() = "mla_decode python module";
    py::bind_kernel<interpreter::kernel<config, partial_template, reduction_template, dummy_template>>(m, "mla_decode",
        &config::globals::instructions,
        &config::globals::Q,
        &config::globals::Cache,
        &config::globals::Table,
        &config::globals::O,
        &config::globals::O_scratch,
        &config::globals::Lvec_scratch,
        &config::globals::semaphore,
        &config::globals::Softmax_scale
    );
    py::bind_kernel<dummy_kernel>(m, "dummy_kernel", &dummy_globals::f);
}