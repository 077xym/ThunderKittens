#include "hip/hip_runtime.h"
#include "kittens.cuh"
#include "prototype.cuh"
#include "pyutils/pyutils.cuh"

using namespace kittens;
using namespace kittens::prototype;
using namespace kittens::prototype::interpreter;

static constexpr int QK_D = 576, VO_D = 512, VO_Dd2 = VO_D/2, NUM_ROWS = 32, PAGE_SIZE = 256, ITERS_PER_PAGE = PAGE_SIZE / NUM_ROWS;
using q_tile              = st_bf<64, QK_D>;
using q_global            = kittens::gl<bf16, -1, -1, -1, QK_D, q_tile>; // B * R * H * D_QK
using cache_tile          = st_bf<NUM_ROWS, QK_D>; using v_tile = st_bf<NUM_ROWS, VO_Dd2>; // we need the v_tile for later
using cache_global        = kittens::gl<bf16, 1, -1, PAGE_SIZE, QK_D, cache_tile>; // 1 * #page * pagesize * QK_D
using instructions_global = kittens::gl<int, 1, -1, -1, 8>;
using table_global        = kittens::gl<int, 1, 1, -1, -1>; // B * (max # pages)
using o_tile              = st_bf<64, VO_D>;
using o_tile_d2           = st_bf<64, VO_Dd2>;
using o_tile_fl           = st_fl<16, VO_D>;
using o_global            = kittens::gl<bf16, -1, -1, -1, VO_D, o_tile_d2, st_bf<16, VO_D>>; // B * R * H * D_VO

// using o_scratch_global    = kittens::gl<float, 1, -1, 64, VO_D, o_tile_fl>; // For partial O's
// using lvec_scratch_global = kittens::gl<float, 1,  1, -1,   64, sv_fl<16>>; // For partial O's
using o_scratch_global    = kittens::gl<float, -1, -1, 16, VO_D, o_tile_fl>; // For partial O's
using lvec_scratch_global = kittens::gl<float,  1, -1, -1, 16, sv_fl<16>>; // For partial O's

using semaphore_global    = kittens::gl<int,   1,  1,  1, -1>;              // 1 * 1 * 1 * uid

struct config {
    struct globals {
        instructions_global instructions;
        q_global Q;
        cache_global Cache;
        table_global Table;
        o_global O;
        o_scratch_global O_scratch;
        lvec_scratch_global Lvec_scratch;
        semaphore_global semaphore;
        const float Softmax_scale;
        int dynamic_shared_memory() { return 226000; }
        dim3 grid()  { return dim3(132); } //dim3(Q.batch * ((Q.depth + 3) / 4)); }
        dim3 block() { return dim3((8+4)*WARP_THREADS); }
    };
};

struct location {
    int batch_idx; // batch_idx >=0, otherwise it's the negative index into scratch
    int seq_idx;
};
struct partial_layout {
    using globals = config::globals;
    struct input_block { cache_tile c; };
    struct scratch_block { q_tile q; st_bf<64, cache_tile::rows> att_block; sv_fl<64> max_vec, norm_vec; };
    struct common_state {
        int uid;
        location dst;
        int q_batch_idx;
        int q_seq_idx;
        int length;
    };
    struct consumer_state {
        col_vec<rt_fl<16, cache_tile::rows>> max_vec, norm_vec;
        rt_fl<16, VO_Dd2> o;
    };
};
struct partial_template {
    using config = config;
    using layout = partial_layout;
    static constexpr int opcode = 1;
    static constexpr int INPUT_PIPE_STAGES = 2;
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        args.common.uid         =  args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 1}];
        args.common.dst         = {args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 2}],
                                   args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 3}]};
        args.common.q_batch_idx =  args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 4}];
        args.common.q_seq_idx   =  args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 5}];
        args.common.length      =  args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 6}];
        args.num_iters          = (args.common.length + NUM_ROWS - 1) / NUM_ROWS;
        args.common.length -= (args.globals.Q.depth - (args.common.q_seq_idx + warpgroup::warpid()) - 1); // adjust for the causal mask
    }
    struct producer {
        __device__ static inline void setup(producer_setup_args<layout> args) {}
        __device__ static inline void load(producer_load_args<layout> args) {
            if(warpgroup::warpid() == 0) {
                int global_load_idx = args.iter % ITERS_PER_PAGE;
                int next_page_id = args.globals.Table[coord<>{args.common.uid, args.iter / ITERS_PER_PAGE}];
                // next page we need to load?
                tma::expect(args.inputs_arrived, args.input.c);
                // cache shape is 1, # pages, page size, QK_D
                tma::load_async(args.input.c, args.globals.Cache, {0, next_page_id, global_load_idx, 0}, args.inputs_arrived);
            }
            else if(laneid() == 0) arrive(args.inputs_arrived);
            warpgroup::sync(5);
        }
    };
    struct consumer {
        __device__ static inline void setup(consumer_setup_args<layout> args) {
            zero(args.state.norm_vec);
            neg_infty(args.state.max_vec);
            zero(args.state.o);
            auto q_st = subtile_inplace<16, QK_D/2>(args.scratch.q, {warpgroup::warpid(), warpgroup::groupid()});
            load_async(q_st, args.globals.Q, {args.common.q_batch_idx, args.common.q_seq_idx + warpgroup::warpid(), 0, warpgroup::groupid()});
            load_async_wait();
            group<8>::sync(10);
        }
        template<bool do_right_fill> __device__ static inline void internal_compute(consumer_compute_args<layout> args) {
            // 1.44269504089f is from exp2
            const float SOFTMAX_TEMPERATURE = args.globals.Softmax_scale * 1.44269504089f;

            col_vec<rt_fl<16, cache_tile::rows>> local_max_vec, local_norm_vec;
            col_vec<rt_fl<16, cache_tile::rows>> max_vec_last_scaled, max_vec_scaled;

            copy(local_max_vec,  args.state.max_vec);
            copy(local_norm_vec, args.state.norm_vec);

            if(warpgroup::groupid() == 0) {
                // A = Q @ K.T
                rt_fl<16, cache_tile::rows> att_block_fp32;
                warpgroup::mm_ABt(att_block_fp32, args.scratch.q, args.input.c);

                mul(max_vec_last_scaled, local_max_vec, SOFTMAX_TEMPERATURE);

                warpgroup::mma_async_wait();
                // softmax
                if constexpr (do_right_fill) { // need to mask out a bunch of entries in the last page
                    const int length = args.common.length - args.iter*NUM_ROWS;
                    right_fill(att_block_fp32, att_block_fp32, length, base_types::constants<float>::neg_infty());
                }

                row_max(local_max_vec, att_block_fp32, local_max_vec);
                mul(max_vec_scaled, local_max_vec, SOFTMAX_TEMPERATURE);

                mul(att_block_fp32, att_block_fp32, SOFTMAX_TEMPERATURE);
                sub_row(att_block_fp32, att_block_fp32, max_vec_scaled);
                
                exp2(att_block_fp32, att_block_fp32);
                
                sub(max_vec_last_scaled, max_vec_last_scaled, max_vec_scaled);
                exp2(max_vec_last_scaled, max_vec_last_scaled);
                warpgroup::store(args.scratch.max_vec, max_vec_last_scaled);
                
                mul(local_norm_vec, local_norm_vec, max_vec_last_scaled);
                row_sum(local_norm_vec, att_block_fp32, local_norm_vec);
                warpgroup::store(args.scratch.att_block, att_block_fp32);
            }
            group<8>::sync(10);

            warpgroup::load(max_vec_last_scaled, args.scratch.max_vec);
            mul_row(args.state.o, args.state.o, max_vec_last_scaled); // normalize o_reg before mma

            // O += A @ V
            auto (&v_smem)[2] = reinterpret_cast<v_tile(&)[2]>(args.input.c);
            warpgroup::mma_AB(args.state.o, args.scratch.att_block, v_smem[warpgroup::groupid()]);

            copy(args.state.max_vec, local_max_vec);
            copy(args.state.norm_vec, local_norm_vec);

            warpgroup::mma_async_wait();
            if(warpgroup::laneid() == 0) arrive(args.inputs_finished, WARPGROUP_WARPS); // done!
        }
        __device__ static inline void compute(consumer_compute_args<layout> args) {
            if(args.iter >= args.num_iters-2) internal_compute<true>(args);
            else internal_compute<false>(args);
        }
        __device__ static inline void finish(consumer_finish_args<layout> args) {
            col_vec<rt_fl<16, cache_tile::rows>> local_max_vec, local_norm_vec;

            copy(local_norm_vec, args.state.norm_vec);
            copy(local_max_vec, args.state.max_vec);

            if (warpgroup::groupid() == 0) warpgroup::store(args.scratch.norm_vec, local_norm_vec);
            group<8>::sync(10);
            if(warpgroup::groupid() == 1) warpgroup::load(local_norm_vec, args.scratch.norm_vec);
            div_row(args.state.o, args.state.o, local_norm_vec);

            if(args.common.dst.batch_idx >= 0) { // batch is meaningful
                auto &o_smem = reinterpret_cast<o_tile&>(args.scratch.q);
                auto o_st = subtile_inplace<16, VO_Dd2>(o_smem, {warpgroup::warpid(), warpgroup::groupid()});
                store(o_st, args.state.o);
                __syncwarp();
                store(args.globals.O, o_st, {args.common.dst.batch_idx, args.common.dst.seq_idx+warpgroup::warpid(), 0, warpgroup::groupid()});
            }
            else { // write out directly to O scratch, without going through smem
                if(warpgroup::groupid() == 0) {
                    mul(local_max_vec, local_max_vec, args.globals.Softmax_scale * 1.44269504089f);
                    log2(local_norm_vec, local_norm_vec);
                    add(local_norm_vec, local_norm_vec, local_max_vec); // l_vec = log2(norm_vec) + max_vec
                    store(args.globals.Lvec_scratch, local_norm_vec, {args.common.dst.seq_idx, warpgroup::warpid(), 0});
                }

                store(args.globals.O_scratch, args.state.o, {args.common.dst.seq_idx, warpgroup::warpid(), 0, warpgroup::groupid()});
            }
            group<8>::sync(10);
            if(args.common.dst.batch_idx < 0) {
                asm volatile("fence.sc.sys;");
                if(group<8>::laneid() == 0) {
                    args.globals.semaphore[{args.common.dst.seq_idx}] = 1;
                }
            }
            if(warpgroup::laneid() == 0) arrive(args.finish_finished, WARPGROUP_WARPS); // done!
        }
    };
};
struct reduction_layout {
    using globals = config::globals;
    struct input_block  { o_tile_fl o[2]; sv_fl<16> lvec[2]; sv_fl<16> padding[14]; };
    struct output_block { o_tile_fl o; sv_fl<16> lvec; sv_fl<16> padding[15]; };
    struct common_state {
        int uid;
        location dst;
        int src_uid[2];
        int src_batch[2];
    };
    struct consumer_state {};
};
struct reduction_template {
    using config = config;
    using layout = reduction_layout;
    static constexpr int opcode = 2;
    static constexpr int INPUT_PIPE_STAGES = 2;
    __device__ static inline void common_setup(common_setup_args<layout> args) {
        args.common.uid           = args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 1}];
        args.common.dst           = {args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 2}], args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 3}]};
        args.common.src_uid[0] = args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 4}];
        args.common.src_uid[1] = args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 5}];
        args.common.src_batch[0] = args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 6}];
        args.common.src_batch[1] = args.globals.instructions[kittens::coord<>{0, (int)(blockIdx.x), args.task_iter, 7}];
        args.num_iters = 1;
        // If we are doing a reduction, we need to spinloop until we have confirmation that all the partial results have been written out.
        if(threadIdx.x == 0) { // easier to have a single thread spin
            while(*(volatile int*)&args.globals.semaphore[{args.common.src_uid[0]}] == 0) {} // note volatile, L1 is not guaranteed to be coherent.
            while(*(volatile int*)&args.globals.semaphore[{args.common.src_uid[1]}] == 0) {}
        }
        group<12>::sync(11); // all warps must sync here.
    }
    struct producer {
        __device__ static inline void setup(producer_setup_args<layout> args) {}
        __device__ static inline void load(producer_load_args<layout> args) {
            if(warpgroup::warpid() == args.iter%4) {
                // next page we need to load?
                tma::expect(args.inputs_arrived, args.input.o[0], args.input.o[1], args.input.lvec[0], args.input.lvec[1]);
                tma::load_async(args.input.o[0], args.globals.O_scratch, {args.common.src_batch[0], args.common.dst.seq_idx, 0, 0}, args.inputs_arrived);
                tma::load_async(args.input.o[1], args.globals.O_scratch, {args.common.src_batch[1], args.common.dst.seq_idx, 0, 0}, args.inputs_arrived);
                tma::load_async(args.input.lvec[0], args.globals.Lvec_scratch, {args.common.src_batch[0], args.common.dst.seq_idx, 0}, args.inputs_arrived);
                tma::load_async(args.input.lvec[1], args.globals.Lvec_scratch, {args.common.src_batch[1], args.common.dst.seq_idx, 0}, args.inputs_arrived);
                if(laneid() == 0) arrive(args.inputs_arrived, 3);
            }
        }
        __device__ static inline void store(producer_store_args<layout> args) {
            if(warpgroup::warpid() == args.iter%4) {
                if(args.common.dst.batch_idx >= 0) {
                    tma::store_async(args.globals.O, reinterpret_cast<st_bf<16, VO_D>&>(args.output.o),
                        {args.common.dst.batch_idx, args.common.dst.seq_idx, 0, group<8>::warpid()});
                }
                else {
                    tma::store_async(args.globals.O_scratch, args.output.o, {-args.common.dst.batch_idx, args.common.dst.seq_idx, 0});
                    tma::store_async(args.globals.Lvec_scratch, args.output.lvec, {-args.common.dst.batch_idx, args.common.dst.seq_idx});
                }
                tma::store_async_wait();
                __syncwarp();
                if(laneid() == 0) arrive(args.outputs_finished, 4);
            }
        }
    };
    struct consumer {
        __device__ static inline void setup(consumer_setup_args<layout> args) {}
        __device__ static inline void compute(consumer_compute_args<layout> args) {
            col_vec<rt_fl<16, cache_tile::rows>> lvec[2], max_lvec, sum_lvec;
            auto o1_st = subtile_inplace<16, VO_D/8>(args.input.o[0], {0, group<8>::warpid()});
            auto o2_st = subtile_inplace<16, VO_D/8>(args.input.o[1], {0, group<8>::warpid()});
            rt_fl<16, VO_D / 8> o[2];
            load(o[0], o1_st);
            load(o[1], o2_st);
            load(lvec[0], args.input.lvec[0]);
            load(lvec[1], args.input.lvec[1]);
            __syncwarp();
            if(laneid() == 0) arrive(args.inputs_finished); // done!
            max(max_lvec, lvec[0], lvec[1]);
            sub(lvec[0], lvec[0], max_lvec);
            sub(lvec[1], lvec[1], max_lvec);
            exp2(lvec[0], lvec[0]);
            exp2(lvec[1], lvec[1]);
            add(sum_lvec, lvec[0], lvec[1]);
            div(lvec[0], lvec[0], sum_lvec);
            div(lvec[1], lvec[1], sum_lvec);
            mul_row(o[0], o[0], lvec[0]);
            mul_row(o[1], o[1], lvec[1]);
            add(o[0], o[0], o[1]);
            log2(sum_lvec, sum_lvec);
            add(sum_lvec, sum_lvec, max_lvec);
            if(args.common.dst.batch_idx >= 0) {
                auto &o_smem = reinterpret_cast<st_bf<16, VO_D>&>(args.output.o);
                auto o_st = subtile_inplace<16, VO_D/8>(o_smem, {0, group<8>::warpid()});
                store(o_st, o[0]);
            }
            else {
                auto o_st = subtile_inplace<16, VO_D/8>(args.output.o, {0, group<8>::warpid()});
                store(o_st, o[0]);
                if(group<8>::warpid() == 0) {
                    store(args.output.lvec, sum_lvec);
                }
            }
            __syncwarp();
            if(laneid() == 0) arrive(args.outputs_arrived); // done!
        }
        __device__ static inline void finish(consumer_finish_args<layout> args) {
            // Increment the semaphore for the next stage, if this is not the last one.
            warpgroup::sync(warpgroup::groupid()); // Make sure memory has been flushed to global memory.
            if(args.common.dst.batch_idx < 0) {
                asm volatile("fence.sc.sys;");
                if(group<8>::laneid() == 0) {
                    args.globals.semaphore[{args.common.dst.seq_idx}] = 1;
                }
                __syncwarp();
            }
            if(warpgroup::laneid() == 0) arrive(args.finish_finished, WARPGROUP_WARPS); // done!
        }
    };
};

PYBIND11_MODULE(mla_decode, m) {
    m.doc() = "mla_decode python module";
    py::bind_kernel<interpreter::kernel<config, partial_template, reduction_template>>(m, "mla_decode",
        &config::globals::instructions,
        &config::globals::Q,
        &config::globals::Cache,
        &config::globals::Table,
        &config::globals::O,
        &config::globals::O_scratch,
        &config::globals::Lvec_scratch,
        &config::globals::semaphore,
        &config::globals::Softmax_scale
    );
}